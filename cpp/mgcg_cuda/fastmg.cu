#include "hip/hip_runtime.h"
#include <algorithm>
#include <cassert>
#include <cstring>
#include <fstream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsparse.h>
#include <hipsolver.h>
#include <iostream>
#include <string>
#include <sstream>
#include <cstdio>
#include <cmath>
#include <chrono>
#include <array>
#include <unordered_set>
#include <unordered_map>
#include <map>
#include <set>
#include <numeric>

#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/inner_product.h>
#include <thrust/random.h>

#include "kernels.cuh"
#include "cuda_utils.cuh"
#include "cusparse_wrappers.h"
#include "mglevel.h"
#include "smoother.h"
#include "fastmg.h"
#include "fastfill.h"
#include "vcycle.h"
#include "Vec.h"
#include "CSR.h"

using std::cout;
using std::endl;

#define USE_LESSMEM 1

namespace fastmg{


float sum(std::vector<float> &v)
{
    return std::accumulate(v.begin(), v.end(), 0.0);
}

float avg(std::vector<float> &v)
{
    return std::accumulate(v.begin(), v.end(), 0.0) / v.size();
}



/* -------------------------------------------------------------------------- */
/*                                   VCycle                                   */
/* -------------------------------------------------------------------------- */

    void FastMG::set_scale_RAP(float s, int lv)
    {
        levels.at(lv).scale_RAP = s;
        cout<<"Set scale_RAP: "<<levels.at(lv).scale_RAP<<"  at level "<<lv<<endl;
    }


    void  FastMG::create_levels(size_t numlvs) {
        if (levels.size() < numlvs) {
            levels.resize(numlvs);
        }
        nlvs = numlvs;

        smoother = std::make_shared<Smoother>(levels);
        vcycle = std::make_unique<VCycle>(levels, smoother,z,r,buff);

    }


    void  FastMG::set_P(size_t lv, float const *datap, size_t ndat, int const *indicesp, size_t nind, int const *indptrp, size_t nptr, size_t rows, size_t cols, size_t nnz) {
        levels.at(lv).P.assign(datap, ndat, indicesp, nind, indptrp, nptr, rows, cols, nnz);
    }

    void  FastMG::set_A0(float const *datap, size_t ndat, int const *indicesp, size_t nind, int const *indptrp, size_t nptr, size_t rows, size_t cols, size_t nnz) {
        levels.at(0).A.assign(datap, ndat, indicesp, nind, indptrp, nptr, rows, cols, nnz);
    }


    int  FastMG::get_nnz(int lv) {
        return levels.at(lv).A.numnonz;
    }

    int  FastMG::get_nrows(int lv) {
        return levels.at(lv).A.nrows;
    }

    // only update the data of A0
    void  FastMG::update_A0(float const *datap) {
        CHECK_CUDA(hipMemcpy(levels.at(0).A.data.data(), datap, levels.at(0).A.data.size() * sizeof(float), hipMemcpyHostToDevice));
    }

    void  FastMG::set_A0_from_fastFill(FastFillBase *ff)
    {
        if (levels.size() < 1) {
            levels.resize(1);
        }

        levels.at(0).A.numnonz = ( ff->num_nonz);
        levels.at(0).A.nrows = ( ff->nrows);

        //FIXME: As in python code, we need copy A, why?
        levels.at(0).A.data.resize(ff->A.data.size());
        CHECK_CUDA(hipMemcpy(levels.at(0).A.data.data(), (ff->A).data.data(), (ff->A).data.size() * sizeof(float), hipMemcpyDeviceToDevice));
        levels.at(0).A.indices.resize(ff->A.indices.size());
        CHECK_CUDA(hipMemcpy(levels.at(0).A.indices.data(), (ff->A).indices.data(), (ff->A).indices.size() * sizeof(int), hipMemcpyDeviceToDevice));
        levels.at(0).A.indptr.resize(ff->A.indptr.size());
        CHECK_CUDA(hipMemcpy(levels.at(0).A.indptr.data(), (ff->A).indptr.data(), (ff->A).indptr.size() * sizeof(int), hipMemcpyDeviceToDevice));
    }



    float  FastMG::calc_residual(CSR<float> const &A, Vec<float> &x, Vec<float> const &b, Vec<float> &r) {
        copy(r, b);
        spmv(r, -1, A, x, 1, buff); // residual = b - A@x
        return vnorm(r);
    }



    void  FastMG::set_outer_x(float const *x, size_t n) {
        outer_x.resize(n);
        CHECK_CUDA(hipMemcpy(outer_x.data(), x, n * sizeof(float), hipMemcpyHostToDevice));
        copy(x_new, outer_x);
    }

    void  FastMG::set_outer_b(float const *b, size_t n) {
        outer_b.resize(n);
        CHECK_CUDA(hipMemcpy(outer_b.data(), b, n * sizeof(float), hipMemcpyHostToDevice));
    }

    float  FastMG::init_cg_iter0(float *residuals) {
        float bnrm2 = vnorm(outer_b);
        // r = b - A@(x)
        copy(r, outer_b);
        spmv(outer_b, -1, levels.at(0).A, outer_x, 1, buff);
        float normr = vnorm(r);
        residuals[0] = normr;
        return bnrm2;
    }

    void  FastMG::do_cg_itern(float *residuals, size_t iteration) {
        float rho_cur = vdot(r, z);
        if (iteration > 0) {
            float beta = rho_cur / save_rho_prev;
            // p *= beta
            // p += z
            scal(save_p, beta);
            axpy(save_p, 1, z);
        } else {
            // p = move(z)
            save_p.swap(z);
        }
        // q = A@(p)
        save_q.resize(levels.at(0).A.nrows);
        spmv(save_q, 1, levels.at(0).A, save_p, 0, buff);
        save_rho_prev = rho_cur;
        float alpha = rho_cur / vdot(save_p, save_q);
        // x += alpha*p
        axpy(x_new, alpha, save_p);
        // r -= alpha*q
        axpy(r, -alpha, save_q);
        float normr = vnorm(r);
        residuals[iteration + 1] = normr;
    }

    void  FastMG::compute_RAP(size_t lv) {
            CSR<float> &A = levels.at(lv).A;
            CSR<float> &R = levels.at(lv).R;
            CSR<float> &P = levels.at(lv).P;
            CSR<float> AP;
            CSR<float> &RAP = levels.at(lv+1).A;
            R.resize(P.ncols, P.nrows, P.numnonz);
            transpose(P, R);            
            spgemm(A, P, AP) ;
            spgemm(R, AP, RAP);

            float s = levels.at(lv).scale_RAP;
            if (s!=0.0){
                // scale RAP by a scalar
                cout<<"scaling RAP by "<<s<<" at lv "<<lv<<endl;
                scal(RAP.data, s);
            }
    }

    void  FastMG::fetch_A_data(float *data) {
        CSR<float> &A = levels.at(0).A;
        CHECK_CUDA(hipMemcpy(data, A.data.data(), A.data.size() * sizeof(float), hipMemcpyDeviceToHost));
    }

    // In python end, before you call fetch A, you should call get_nnz and get_matsize first to determine the size of the csr matrix. 
    void  FastMG::fetch_A(size_t lv, float *data, int *indices, int *indptr) {
        CSR<float> &A = levels.at(lv).A;
        CHECK_CUDA(hipMemcpy(data, A.data.data(), A.data.size() * sizeof(float), hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(indices, A.indices.data(), A.indices.size() * sizeof(int), hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(indptr, A.indptr.data(), A.indptr.size() * sizeof(int), hipMemcpyDeviceToHost));
    }
    
    void  FastMG::set_data(const float* x, size_t nx, const float* b, size_t nb, float rtol_, size_t maxiter_)
    {
        set_outer_x(x, nx);
        set_outer_b(b, nb);
        rtol = rtol_;
        maxiter = maxiter_;
        residuals.resize(maxiter+1);
    }



    size_t  FastMG::get_data(float* x_out, float* r_out)
    {
        CHECK_CUDA(hipMemcpy(x_out, x_new.data(), x_new.size() * sizeof(float), hipMemcpyDeviceToHost));
        std::copy(residuals.begin(), residuals.end(), r_out);
        return niter;
    }


    void  FastMG::presolve()
    {
        // TODO: move fillA from python-end to here as well in the future refactoring
        for(int lv=0; lv<nlvs; lv++)
        {
            // for jacobi_v2 (use cusparse etc.)
            if(smoother->smoother_type == 2)
            {
                get_Aoff_and_Dinv(levels.at(lv).A, levels.at(lv).Dinv, levels.at(lv).Aoff);
            }
        }
        for (size_t lv = 0; lv < nlvs-1; lv++)
        {
            compute_RAP(lv);
        }
        
    }

    void  FastMG::solve()
    {
        presolve();
        float bnrm2 = init_cg_iter0(residuals.data());
        float atol = bnrm2 * rtol;
        for (size_t iter=0; iter<maxiter; iter++)
        {   
            if (residuals[iter] < atol)
            {
                niter = iter;
                break;
            }
            copy(z, outer_x);
            vcycle -> run();
            do_cg_itern(residuals.data(), iter); 
            niter = iter;
        }
    }

    void  FastMG::solve_only_jacobi()
    {
        timer1.start();
        get_Aoff_and_Dinv(levels.at(0).A, levels.at(0).Dinv, levels.at(0).Aoff);
        for (size_t iter=0; iter<maxiter; iter++)
            smoother->jacobi_v2(0, outer_x, outer_b);
        copy(x_new, outer_x);
        
        timer1.stop();
        elapsed1.push_back(timer1.elapsed());
        // if (verbose)
            cout<<" only iterative time: "<<(elapsed1[0])<<" ms"<<endl;
        elapsed1.clear();
    }

    void  FastMG::solve_only_directsolver()
    {
        timer1.start();

        spsolve(outer_x, levels.at(0).A, outer_b);
        copy(x_new, outer_x);
        
        timer1.stop();
        elapsed1.push_back(timer1.elapsed());
        // if (verbose)
            cout<<" only direct time: "<<(elapsed1[0])<<" ms"<<endl;
        elapsed1.clear();
    }

    void  FastMG::solve_only_smoother()
    {
        timer1.start();
        presolve();
        float bnrm2 = init_cg_iter0(residuals.data());
        float atol = bnrm2 * rtol;
        for (size_t iter=0; iter<maxiter; iter++)
        {   
            smoother->smooth(0, outer_x, outer_b);
            auto rnorm = calc_residual(levels.at(0).A, outer_x, outer_b, r);
            residuals[iter] = rnorm;
            if (residuals[iter] < atol)
            {
                niter = iter;
                break;
            }
            niter = iter;
        }
        copy(x_new, outer_x);

        timer1.stop();
        elapsed1.push_back(timer1.elapsed());
        cout<<elapsed1.size()<<" only smoother time: "<<(elapsed1[0])<<" ms"<<endl;
        elapsed1.clear();

    }


void FastMG::get_Aoff_and_Dinv(CSR<float> &A, CSR<float> &Dinv, CSR<float> &Aoff)
{
    int n = A.nrows;
    // get diagonal inverse of A, fill into a vector
    Vec<float> d_diag_inv;
    d_diag_inv.resize(n);
    calc_diag_inv_kernel<<<(n + 255) / 256, 256>>>(d_diag_inv.data(), A.data.data(), A.indices.data(), A.indptr.data(), n);
    hipDeviceSynchronize();
    LAUNCH_CHECK();

    // fill diag to a CSR matrix Dinv
    std::vector<int> seqence(n);
    for (int i = 0; i < n; i++)
        seqence[i] = i;
    // copy d_diag_inv to host
    std::vector<float> h_diag_inv(n);
    CHECK_CUDA(hipMemcpy(h_diag_inv.data(), d_diag_inv.data(), n * sizeof(float), hipMemcpyDeviceToHost));
    Dinv.assign_v2(h_diag_inv.data(), seqence.data(), seqence.data(), n, n, n);
    hipDeviceSynchronize();
    LAUNCH_CHECK();

    Aoff.resize(n, n, A.numnonz);
    CHECK_CUDA(hipMemcpy(Aoff.data.data(), A.data.data(), A.numnonz * sizeof(float), hipMemcpyDeviceToDevice));
    Aoff.assign(Aoff.data.data(), A.numnonz, A.indices.data(), A.numnonz, A.indptr.data(), n + 1, n, n, A.numnonz);
    // get Aoff by set diagonal of A to 0
    get_Aoff_kernel<<<(A.numnonz + 255) / 256, 256>>>(Aoff.data.data(), A.indices.data(), A.indptr.data(), n);
    hipDeviceSynchronize();
    LAUNCH_CHECK();
}


FastMG *fastmg = nullptr;


#if _WIN32
#define DLLEXPORT __declspec(dllexport)
#else
#define DLLEXPORT
#endif


extern "C" DLLEXPORT void fastmg_setup_smoothers(int type) {
    fastmg->smoother->setup_smoothers(type);
}


extern "C" DLLEXPORT void fastmg_set_smoother_niter(const size_t niter) {
    fastmg->smoother->set_smoother_niter(niter);
}

extern "C" DLLEXPORT void fastmg_set_colors(const int *c, int n, int color_num, int lv) {
    fastmg->smoother->set_colors(c, n, color_num, lv);
}

extern "C" DLLEXPORT void fastmg_use_radical_omega(int flag) {
    fastmg->smoother->use_radical_omega = bool(flag);
}


extern "C" DLLEXPORT void fastmg_set_coarse_solver_type(int t) {
    fastmg->vcycle->coarse_solver_type = t;
}


extern "C" DLLEXPORT void fastmg_set_A0_from_fastFillCloth() {
    fastmg->set_A0_from_fastFill(fastFillCloth);
}

extern "C" DLLEXPORT void fastmg_set_A0_from_fastFillSoft() {
    fastmg->set_A0_from_fastFill(fastFillSoft);
}



extern "C" DLLEXPORT void fastmg_new() {
    if (!fastmg)
        fastmg = new FastMG{};
}

extern "C" DLLEXPORT void fastmg_setup_nl(size_t numlvs) {
    fastmg->create_levels(numlvs);
}


extern "C" DLLEXPORT void fastmg_RAP(size_t lv) {
    fastmg->compute_RAP(lv);
}


extern "C" DLLEXPORT int fastmg_get_nnz(size_t lv) {
    int nnz = fastmg->get_nnz(lv);
    std::cout<<"nnz: "<<nnz<<std::endl;
    return nnz;
}

extern "C" DLLEXPORT int fastmg_get_matsize(size_t lv) {
    int n = fastmg->get_nrows(lv);
    std::cout<<"matsize: "<<n<<std::endl;
    return n;
}

extern "C" DLLEXPORT void fastmg_fetch_A(size_t lv, float* data, int* indices, int* indptr) {
    fastmg->fetch_A(lv, data, indices, indptr);
}

extern "C" DLLEXPORT void fastmg_fetch_A_data(float* data) {
    fastmg->fetch_A_data(data);
}

extern "C" DLLEXPORT void fastmg_solve() {
    fastmg->solve();
}

extern "C" DLLEXPORT void fastmg_set_data(const float* x, size_t nx, const float* b, size_t nb, float rtol, size_t maxiter) {
    fastmg->set_data(x, nx, b, nb, rtol, maxiter);
}

extern "C" DLLEXPORT size_t fastmg_get_data(float *x, float *r) {
    size_t niter = fastmg->get_data(x, r);
    return niter;
}

extern "C" DLLEXPORT void fastmg_set_A0(float* data, int* indices, int* indptr, int rows, int cols, int nnz)
{
    fastmg->set_A0(data, nnz, indices, nnz, indptr, rows + 1, rows, cols, nnz);
}

// only update the data of A0
extern "C" DLLEXPORT void fastmg_update_A0(const float* data_in)
{
    fastmg->update_A0(data_in);
}

extern "C" DLLEXPORT void fastmg_set_P(int lv, float* data, int* indices, int* indptr, int rows, int cols, int nnz)
{
    fastmg->set_P(lv, data, nnz, indices, nnz, indptr, rows + 1, rows, cols, nnz);
}



extern "C" DLLEXPORT void fastmg_scale_RAP(float s, int lv) {
    fastmg->set_scale_RAP(s, lv);
}


extern "C" DLLEXPORT void fastmg_solve_only_smoother() {
    fastmg->solve_only_smoother();
}


extern "C" DLLEXPORT void fastmg_solve_only_jacobi() {
    fastmg->solve_only_jacobi();
}

extern "C" DLLEXPORT void fastmg_solve_only_directsolver() {
    fastmg->solve_only_directsolver();
}







} // namespace
