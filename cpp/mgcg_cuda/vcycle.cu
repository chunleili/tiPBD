#include "hip/hip_runtime.h"
#include <algorithm>
#include <cassert>
#include <cstring>
#include <fstream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsparse.h>
#include <hipsolver.h>
#include <iostream>
#include <string>
#include <sstream>
#include <cstdio>
#include <cmath>
#include <chrono>
#include <filesystem>
#include <array>
#include <unordered_set>

#include "Eigen/Core"
#include "Eigen/Dense"
#include "Eigen/Sparse"
#include "unsupported/Eigen/SparseExtra"

using namespace std;
using EigenSpMat = Eigen::SparseMatrix<float, Eigen::RowMajor>;


#if __GNUC__ && __linux__
#include <sys/ptrace.h>

[[noreturn]] static void cuerr() {
    if (ptrace(PTRACE_TRACEME, 0, NULL, NULL) != 0)
        __builtin_trap();
    exit(EXIT_FAILURE);
}
#elif _WIN32 && _MSC_VER
#include <windows.h>

[[noreturn]] static void cuerr() {
    int debugger_present = 0;
    HANDLE process = GetCurrentProcess();
    CheckRemoteDebuggerPresent(process, &debugger_present);
    if (debugger_present) {
        __debugbreak();
    }
    exit(EXIT_FAILURE);
}
#else
[[noreturn]] static void cuerr() {
    exit(EXIT_FAILURE);
}
#endif

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("%s:%d: %s (%d): %s\n", __FILE__, __LINE__,                     \
               hipGetErrorString(status), status, #func);                     \
        cuerr();                                                               \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("%s:%d: %s (%d): %s\n", __FILE__, __LINE__,                     \
               hipsparseGetErrorString(status), status, #func);                 \
        cuerr();                                                               \
    }                                                                          \
}

#define CHECK_CUBLAS(func)                                                     \
{                                                                              \
    hipblasStatus_t status = (func);                                            \
    if (status != HIPBLAS_STATUS_SUCCESS) {                                     \
        printf("%s:%d: %s (%d): %s\n", __FILE__, __LINE__,                     \
               cublasGetStatusString(status), status, #func);                  \
        cuerr();                                                               \
    }                                                                          \
}


// https://github.com/NVIDIA/CUDALibrarySamples/blob/ed19a07b6dd0900b7547b274a6ed9d7c22a6d431/cuSOLVER/utils/cusolver_utils.h#L55
#define CHECK_CUSOLVER(err)                                                                        \
    do {                                                                                           \
        hipsolverStatus_t err_ = (err);                                                             \
        if (err_ != HIPSOLVER_STATUS_SUCCESS) {                                                     \
            printf("cusolver error %d at %s:%d\n", err_, __FILE__, __LINE__);                      \
            throw std::runtime_error("cusolver error");                                            \
        }                                                                                          \
    } while (0)


namespace {

struct Buffer {
    void *m_data;
    size_t m_cap;

    Buffer() noexcept : m_data(nullptr), m_cap(0) {
    }

    Buffer(Buffer &&that) noexcept : m_data(that.m_data), m_cap(that.m_cap) {
        that.m_data = nullptr;
        that.m_cap = 0;
    }

    Buffer &operator=(Buffer &&that) noexcept {
        if (this == &that) return *this;
        if (m_data)
            CHECK_CUDA(hipFree(m_data));
        m_data = nullptr;
        m_data = that.m_data;
        m_cap = that.m_cap;
        that.m_data = nullptr;
        that.m_cap = 0;
        return *this;
    }

    ~Buffer() noexcept {
        if (m_data)
            CHECK_CUDA(hipFree(m_data));
        m_data = nullptr;
    }

    void reserve(size_t new_cap) {
        if (m_cap < new_cap) {
            if (m_data)
                CHECK_CUDA(hipFree(m_data));
            m_data = nullptr;
            CHECK_CUDA(hipMalloc(&m_data, new_cap));
            m_cap = new_cap;
        }
    }

    size_t capacity() const noexcept {
        return m_cap;
    }

    void const *data() const noexcept {
        return m_data;
    }

    void *data() noexcept {
        return m_data;
    }
};

template <class T>
hipDataType cudaDataTypeFor();

template <>
hipDataType cudaDataTypeFor<int8_t>() {
    return HIP_R_8I;
}

template <>
hipDataType cudaDataTypeFor<uint8_t>() {
    return HIP_R_8U;
}

template <>
hipDataType cudaDataTypeFor<int16_t>() {
    return HIP_R_16I;
}

template <>
hipDataType cudaDataTypeFor<uint16_t>() {
    return HIP_R_16U;
}

template <>
hipDataType cudaDataTypeFor<int32_t>() {
    return HIP_R_32I;
}

template <>
hipDataType cudaDataTypeFor<uint32_t>() {
    return HIP_R_32U;
}

template <>
hipDataType cudaDataTypeFor<int64_t>() {
    return HIP_R_64I;
}

template <>
hipDataType cudaDataTypeFor<uint64_t>() {
    return HIP_R_64U;
}

template <>
hipDataType cudaDataTypeFor<nv_half>() {
    return HIP_R_16F;
}

template <>
hipDataType cudaDataTypeFor<hip_bfloat16>() {
    return HIP_R_16BF;
}

template <>
hipDataType cudaDataTypeFor<float>() {
    return HIP_R_32F;
}

template <>
hipDataType cudaDataTypeFor<double>() {
    return HIP_R_64F;
}

template <class T>
struct Vec {
    T *m_data;
    size_t m_size;
    size_t m_cap;

    Vec() noexcept : m_data(nullptr), m_size(0), m_cap(0) {
    }

    Vec(Vec &&that) noexcept : m_data(that.m_data), m_size(that.m_size), m_cap(that.m_cap) {
        that.m_data = nullptr;
        that.m_size = 0;
        that.m_cap = 0;
    }

    Vec &operator=(Vec &&that) noexcept {
        if (this == &that) return *this;
        if (m_data)
            CHECK_CUDA(hipFree(m_data));
        m_data = nullptr;
        m_data = that.m_data;
        m_size = that.m_size;
        m_cap = that.m_cap;
        that.m_data = nullptr;
        that.m_size = 0;
        that.m_cap = 0;
        return *this;
    }

    void swap(Vec &that) noexcept {
        std::swap(m_data, that.m_data);
        std::swap(m_size, that.m_size);
        std::swap(m_cap, that.m_cap);
    }

    ~Vec() noexcept {
        if (m_data)
            CHECK_CUDA(hipFree(m_data));
        m_data = nullptr;
    }

    void resize(size_t new_size) {
        bool change = m_cap < new_size;
        if (change) {
            if (m_data)
                CHECK_CUDA(hipFree(m_data));
            m_data = nullptr;
            CHECK_CUDA(hipMalloc(&m_data, sizeof(T) * new_size));
            m_cap = new_size;
        }
        if (m_size != new_size || change) {
            m_size = new_size;
        }
    }

    void assign(T const *data, size_t size) {
        resize(size);
        CHECK_CUDA(hipMemcpy(m_data, data, sizeof(T) * size, hipMemcpyHostToDevice));
    }

    void store(T *data) const {
        CHECK_CUDA(hipMemcpy(data, m_data, sizeof(T) * size(), hipMemcpyDeviceToHost));
    }

    size_t size() const noexcept {
        return m_size;
    }

    T const *data() const noexcept {
        return m_data;
    }

    T *data() noexcept {
        return m_data;
    }
};

struct DnVec {
    hipsparseDnVecDescr_t handle;

    operator hipsparseDnVecDescr_t() const noexcept {
        return handle;
    }

    DnVec() noexcept : handle(0) {}

    template <class T>
    DnVec(Vec<T> &v) {
        CHECK_CUSPARSE(hipsparseCreateDnVec(&handle, v.size(), v.data(), cudaDataTypeFor<T>()));
    }

    DnVec(DnVec &&that) noexcept : handle(that.handle) {
        that.handle = nullptr;
    }

    DnVec &operator=(DnVec &&that) noexcept {
        if (this == &that) return *this;
        if (handle)
            CHECK_CUSPARSE(hipsparseDestroyDnVec(handle));
        handle = that.handle;
        that.handle = nullptr;
        return *this;
    }

    ~DnVec() {
        if (handle)
            CHECK_CUSPARSE(hipsparseDestroyDnVec(handle));
    }
};

struct ConstDnVec {
    hipsparseConstDnVecDescr_t handle;

    operator hipsparseConstDnVecDescr_t() const noexcept {
        return handle;
    }

    ConstDnVec() noexcept : handle(0) {}

    template <class T>
    ConstDnVec(Vec<T> const &v) {
        CHECK_CUSPARSE(hipsparseCreateConstDnVec(&handle, v.size(), v.data(), cudaDataTypeFor<T>()));
    }

    ConstDnVec(ConstDnVec &&that) noexcept : handle(that.handle) {
        that.handle = nullptr;
    }

    ConstDnVec &operator=(ConstDnVec &&that) noexcept {
        if (this == &that) return *this;
        if (handle)
            CHECK_CUSPARSE(hipsparseDestroyDnVec(handle));
        handle = that.handle;
        that.handle = nullptr;
        return *this;
    }

    ~ConstDnVec() {
        if (handle)
            CHECK_CUSPARSE(hipsparseDestroyDnVec(handle));
    }
};

template <class T>
struct CSR {
    Vec<T> data;
    Vec<int> indices;
    Vec<int> indptr;
    int64_t nrows;
    int64_t ncols;
    int64_t numnonz;

    CSR() noexcept : nrows(0), ncols(0), numnonz(0) {}

    void assign(T const *datap, size_t ndat, int const *indicesp, size_t nind, int const *indptrp, size_t nptr, size_t rows, size_t cols, size_t nnz) {
        indices.resize(nind);
        indptr.resize(nptr);
        data.resize(ndat);
        CHECK_CUDA(hipMemcpy(data.data(), datap, data.size() * sizeof(T), hipMemcpyHostToDevice));
        CHECK_CUDA(hipMemcpy(indices.data(), indicesp, indices.size() * sizeof(int), hipMemcpyHostToDevice));
        CHECK_CUDA(hipMemcpy(indptr.data(), indptrp, indptr.size() * sizeof(int), hipMemcpyHostToDevice));
        nrows = rows;
        ncols = cols;
        numnonz = nnz;
    }

    void resize(size_t rows, size_t cols, size_t nnz) {
        nrows = rows;
        ncols = cols;
        numnonz = nnz;
        data.resize(nnz);
        indices.resize(nnz);
        indptr.resize(rows + 1);
    }
};

struct SpMat {
    hipsparseSpMatDescr_t handle;

    operator hipsparseSpMatDescr_t() const noexcept {
        return handle;
    }

    SpMat() noexcept : handle(0) {}

    template <class T>
    SpMat(CSR<T> &m) {
        CHECK_CUSPARSE(hipsparseCreateCsr(&handle, m.nrows, m.ncols, m.numnonz,
                                         m.indptr.data(), m.indices.data(), m.data.data(),
                                         HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                         HIPSPARSE_INDEX_BASE_ZERO, cudaDataTypeFor<T>()) );
    }

    SpMat(SpMat &&that) noexcept : handle(that.handle) {
        that.handle = nullptr;
    }

    SpMat &operator=(SpMat &&that) noexcept {
        if (this == &that) return *this;
        if (handle)
            CHECK_CUSPARSE(hipsparseDestroySpMat(handle));
        handle = that.handle;
        that.handle = nullptr;
        return *this;
    }

    ~SpMat() {
        if (handle)
            CHECK_CUSPARSE(hipsparseDestroySpMat(handle));
    }
};

struct ConstSpMat {
    hipsparseConstSpMatDescr_t handle;

    operator hipsparseConstSpMatDescr_t() const noexcept {
        return handle;
    }

    ConstSpMat() noexcept : handle(0) {}

    template <class T>
    ConstSpMat(CSR<T> const &m) {
        CHECK_CUSPARSE(hipsparseCreateConstCsr(&handle, m.nrows, m.ncols, m.numnonz,
                                              m.indptr.data(), m.indices.data(), m.data.data(),
                                              HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                              HIPSPARSE_INDEX_BASE_ZERO, cudaDataTypeFor<T>()) );
    }

    ConstSpMat(SpMat &&that) noexcept : handle(that.handle) {
        that.handle = nullptr;
    }

    ConstSpMat &operator=(ConstSpMat &&that) noexcept {
        if (this == &that) return *this;
        if (handle)
            CHECK_CUSPARSE(hipsparseDestroySpMat(handle));
        handle = that.handle;
        that.handle = nullptr;
        return *this;
    }

    ~ConstSpMat() {
        if (handle)
            CHECK_CUSPARSE(hipsparseDestroySpMat(handle));
    }
};

struct Kernels {
    hipblasHandle_t cublas;
    hipsparseHandle_t cusparse;
    hipsolverSpHandle_t cusolverH;

    Kernels() {
        CHECK_CUSPARSE(hipsparseCreate(&cusparse));
        CHECK_CUBLAS(hipblasCreate(&cublas));
        CHECK_CUSOLVER(hipsolverSpCreate(&cusolverH));
    }

    Kernels(Kernels &&) = delete;

    ~Kernels() {
        CHECK_CUSPARSE(hipsparseDestroy(cusparse));
        CHECK_CUBLAS(hipblasDestroy(cublas));
        CHECK_CUSOLVER(hipsolverSpDestroy(cusolverH));
    }

    // out = alpha * A@x + beta * out
    void spmv(Vec<float> &out, float const &alpha, CSR<float> const &A, Vec<float> const &x, float const &beta, Buffer &buffer) {
        assert(out.size() == A.nrows);
        size_t bufSize = 0;
        ConstSpMat dA(A);
        ConstDnVec dx(x);
        DnVec dout(out);
        CHECK_CUSPARSE(hipsparseSpMV_bufferSize(cusparse, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                               &alpha, dA, dx, &beta,
                                               dout, cudaDataTypeFor<float>(),
                                               HIPSPARSE_SPMV_ALG_DEFAULT, &bufSize));
        buffer.reserve(bufSize);
        CHECK_CUSPARSE(hipsparseSpMV(cusparse, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    &alpha, dA, dx, &beta,
                                    dout, cudaDataTypeFor<float>(),
                                    HIPSPARSE_SPMV_ALG_DEFAULT, buffer.data()));
    }

    // C = A * B
    void spgemm(CSR<float> const &matA_,  CSR<float> const &matB_, CSR<float> &matC_) 
    {
        ConstSpMat matA(matA_);
        ConstSpMat matB(matB_);
        matC_.resize(matA_.nrows, matB_.ncols, 0);
        SpMat matC(matC_);
        // https://github.com/NVIDIA/CUDALibrarySamples/blob/ade391a17672d26e55429035450bc44afd277d34/cuSPARSE/spgemm/spgemm_example.c#L161
        // https://docs.nvidia.com/cuda/cusparse/#cusparsespgemm
        //--------------------------------------------------------------------------
        float               alpha       = 1.0f;
        float               beta        = 0.0f;
        hipsparseOperation_t opA         = HIPSPARSE_OPERATION_NON_TRANSPOSE;
        hipsparseOperation_t opB         = HIPSPARSE_OPERATION_NON_TRANSPOSE;
        hipDataType        computeType = HIP_R_32F;
        //--------------------------------------------------------------------------
        // buffers
        size_t bufferSize1 = 0,    bufferSize2 = 0;
        Buffer dBuffer1, dBuffer2;
        //--------------------------------------------------------------------------
        // SpGEMM Computation
        hipsparseSpGEMMDescr_t spgemmDesc;
        CHECK_CUSPARSE( hipsparseSpGEMM_createDescr(&spgemmDesc) )

        // ask bufferSize1 bytes for external memory
        CHECK_CUSPARSE(
            hipsparseSpGEMM_workEstimation(cusparse, opA, opB,
                                        &alpha, matA, matB, &beta, matC,
                                        computeType, HIPSPARSE_SPGEMM_DEFAULT,
                                        spgemmDesc, &bufferSize1, NULL) )
        // CHECK_CUDA( hipMalloc((void**) &dBuffer1, bufferSize1) )
        dBuffer1.reserve(bufferSize1);

        // inspect the matrices A and B to understand the memory requirement for
        // the next step
        CHECK_CUSPARSE(
            hipsparseSpGEMM_workEstimation(cusparse, opA, opB,
                                        &alpha, matA, matB, &beta, matC,
                                        computeType, HIPSPARSE_SPGEMM_DEFAULT,
                                        spgemmDesc, &bufferSize1, dBuffer1.data()) )

        // ask bufferSize2 bytes for external memory
        CHECK_CUSPARSE(
            hipsparseSpGEMM_compute(cusparse, opA, opB,
                                &alpha, matA, matB, &beta, matC,
                                computeType, HIPSPARSE_SPGEMM_DEFAULT,
                                spgemmDesc, &bufferSize2, NULL) )
        // CHECK_CUDA( hipMalloc((void**) &dBuffer2, bufferSize2) )
        dBuffer2.reserve(bufferSize2);

        // compute the intermediate product of A * B
        CHECK_CUSPARSE( hipsparseSpGEMM_compute(cusparse, opA, opB,
                                            &alpha, matA, matB, &beta, matC,
                                            computeType, HIPSPARSE_SPGEMM_DEFAULT,
                                            spgemmDesc, &bufferSize2, dBuffer2.data()) )
        // --------------------------------------------------------------------------
        // get matrix C non-zero entries C_nnz1
        CHECK_CUSPARSE( hipsparseSpMatGetSize(matC, &matC_.nrows, &matC_.ncols, &matC_.numnonz) )
        // allocate matrix C
        matC_.resize(matC_.nrows, matC_.ncols, matC_.numnonz);
        // update matC with the new pointers
        CHECK_CUSPARSE(hipsparseCsrSetPointers(matC, matC_.indptr.data(), matC_.indices.data(), matC_.data.data()) )

        // copy the final products to the matrix C
        CHECK_CUSPARSE(
            hipsparseSpGEMM_copy(cusparse, opA, opB,
                                &alpha, matA, matB, &beta, matC,
                                computeType, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc) )
    }


    // dst = src + alpha * dst
    void axpy(Vec<float> &dst, float const &alpha, Vec<float> const &src) {
        assert(dst.size() == src.size());
        CHECK_CUBLAS(hipblasSaxpy(cublas, dst.size(), &alpha, src.data(), 1, dst.data(), 1));
    }

    void zero(Vec<float> &dst) {
        CHECK_CUDA(hipMemset(dst.data(), 0, dst.size() * sizeof(float)));
    }

    void copy(Vec<float> &dst, Vec<float> const &src) {
        dst.resize(src.size());
        CHECK_CUDA(hipMemcpy(dst.data(), src.data(), src.size() * sizeof(float), hipMemcpyDeviceToDevice));
    }

    // dst = alpha * x
    void scal2(Vec<float> &dst, float const &alpha, Vec<float> const &x) {
        copy(dst, x);
        CHECK_CUBLAS(hipblasSscal(cublas, dst.size(), &alpha, dst.data(), 1));
    }

    // dst = alpha * alpha
    void scal(Vec<float> &dst, float const &alpha) {
        CHECK_CUBLAS(hipblasSscal(cublas, dst.size(), &alpha, dst.data(), 1));
    }

    // x = A^{-1} b by cusolver cholesky
    void spsolve(Vec<float> &x, CSR<float> const &A, Vec<float> &b) {
        // https://docs.nvidia.com/cuda/cusolver/index.html#cusolversp-t-csrlsvchol
        hipsparseMatDescr_t descrA = NULL;
        CHECK_CUSPARSE(hipsparseCreateMatDescr(&descrA));
        CHECK_CUSPARSE(hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL));
        CHECK_CUSPARSE(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO)); 
        int singularity;

        CHECK_CUSOLVER( hipsolverSpScsrlsvchol(cusolverH, A.nrows, A.numnonz, descrA, A.data.data(), A.indptr.data(), A.indices.data(), b.data(), 1e-10, 0, x.data(), &singularity) );
    }

    float vdot(Vec<float> const &x, Vec<float> const &y) {
        float result;
        CHECK_CUBLAS(hipblasSdot(cublas, x.size(), x.data(), 1, y.data(), 1, &result));
        return result;
    }

    float vnorm(Vec<float> const &x) {
        float result;
        CHECK_CUBLAS(hipblasSnrm2(cublas, x.size(), x.data(), 1, &result));
        return result;
    }


    void transpose(CSR<float> const & A, CSR<float>& AT)
    {
        // https://docs.nvidia.com/cuda/cusparse/index.html?highlight=hipsparseCsr2cscEx2#cusparsecsr2cscex2

        // hipsparseHandle_t     handle = NULL;
        int m = A.nrows;
        int n = A.ncols;
        int nnz = A.numnonz;
        const float *csrVal  = A.data.data();
        const int *csrRowPtr = A.indptr.data();
        const int *csrColInd = A.indices.data();
        float *cscVal  = AT.data.data();
        int *cscColPtr = AT.indptr.data();
        int *cscRowInd = AT.indices.data();
        hipDataType  valType = HIP_R_32F;
        hipsparseAction_t copyValues = HIPSPARSE_ACTION_NUMERIC;
        hipsparseIndexBase_t idxBase = HIPSPARSE_INDEX_BASE_ZERO;
        hipsparseCsr2CscAlg_t    alg = HIPSPARSE_CSR2CSC_ALG_DEFAULT;
        hipsparseStatus_t status;
        size_t bufferSize = 0;
        Buffer buffer;

        CHECK_CUSPARSE( hipsparseCsr2cscEx2_bufferSize(cusparse, m, n, nnz, csrVal, csrRowPtr, csrColInd, cscVal, cscColPtr, cscRowInd, valType, copyValues, idxBase, alg, &bufferSize));
        buffer.reserve(bufferSize);
        CHECK_CUSPARSE( hipsparseCsr2cscEx2(           cusparse, m, n, nnz, csrVal, csrRowPtr, csrColInd, cscVal, cscColPtr, cscRowInd, valType, copyValues, idxBase, alg, buffer.data()));                
    }

    // https://stackoverflow.com/a/57382195/19253199
    void CuSparseToEigenSparse(CSR<float> const &A, EigenSpMat &mat) 
    {
        //EigenSpMat is RowMajor, i.e. CSR
        const int *indptr = A.indptr.data();
        const int *indices = A.indices.data();
        const float *data = A.data.data();
        const int nnz = A.numnonz;
        const int nrows = A.nrows;
        const int ncols = A.ncols;
        std::vector<int> inner(nnz);       // inner index is the column indices: indices
        std::vector<int> outer(nrows + 1); // outer index is the rowStart: indptr
        std::vector<float> value(nnz);    // value

        hipMemcpy(inner.data(), indices, sizeof(int) * nnz,         hipMemcpyDeviceToHost);
        hipMemcpy(outer.data(), indptr,  sizeof(int) * (nrows + 1), hipMemcpyDeviceToHost);
        hipMemcpy(value.data(), data,    sizeof(float) * nnz,       hipMemcpyDeviceToHost);

        Eigen::Map<EigenSpMat> mat_map(
            nrows, ncols, nnz, outer.data(), inner.data(), value.data());

        mat = mat_map.eval();
    }


};

struct MGLevel {
    CSR<float> A;
    CSR<float> R;
    CSR<float> P;
    Vec<float> residual;
    Vec<float> b;
    Vec<float> x;
    Vec<float> h;
    Vec<float> outh;
};

struct VCycle : Kernels {
    std::vector<MGLevel> levels;
    size_t nlvs;
    std::vector<float> coefficients;
    Vec<float> init_x;
    Vec<float> init_b;
    Vec<float> outer_x;
    Vec<float> alter_x;
    Vec<float> outer_b;
    float save_rho_prev;
    Vec<float> save_p;
    Vec<float> save_q;
    Buffer buff;
    float rtol;
    size_t maxiter;
    std::vector<float> residuals;
    size_t niter; //final number of iterations to break the loop

    void setup(size_t numlvs) {
        if (levels.size() < numlvs) {
            levels.resize(numlvs);
        }
        nlvs = numlvs;
        coefficients.clear();
    }

    void set_lv_csrmat(size_t lv, size_t which, float const *datap, size_t ndat, int const *indicesp, size_t nind, int const *indptrp, size_t nptr, size_t rows, size_t cols, size_t nnz) {
        CSR<float> *mat = nullptr;
        if (which == 1) mat = &levels.at(lv).A;
        if (which == 2) mat = &levels.at(lv).R;
        if (which == 3) mat = &levels.at(lv).P;
        if (mat) {
            mat->assign(datap, ndat, indicesp, nind, indptrp, nptr, rows, cols, nnz);
        }
    }

    void set_coeff(float const *coeff, size_t ncoeffs) {
        coefficients.assign(coeff, coeff + ncoeffs);
    }

    void _smooth(int lv, Vec<float> &x, Vec<float> const &b) {
        copy(levels.at(lv).residual, b);
        spmv(levels.at(lv).residual, -1, levels.at(lv).A, x, 1, buff); // residual = b - A@x
        scal2(levels.at(lv).h, coefficients.at(0), levels.at(lv).residual); // h = c0 * residual


        for (int i = 1; i < coefficients.size(); ++i) {
            // h' = ci * residual + A@h
            copy(levels.at(lv).outh, levels.at(lv).residual);
            spmv(levels.at(lv).outh, 1, levels.at(lv).A, levels.at(lv).h, coefficients.at(i), buff);

            // copy(levels.at(lv).h, levels.at(lv).outh);
            levels.at(lv).h.swap(levels.at(lv).outh);
        }

        axpy(x, 1, levels.at(lv).h); // x += h
    }

    void set_init_x(float const *x, size_t n) {
        init_x.resize(n);
        CHECK_CUDA(hipMemcpy(init_x.data(), x, n * sizeof(float), hipMemcpyHostToDevice));
    }

    void set_init_b(float const *b, size_t n) {
        init_b.resize(n);
        CHECK_CUDA(hipMemcpy(init_b.data(), b, n * sizeof(float), hipMemcpyHostToDevice));
    }

    void vcycle_down() {
        for (int lv = 0; lv < nlvs-1; ++lv) {
            Vec<float> &x = lv != 0 ? levels.at(lv - 1).x : init_x;
            Vec<float> &b = lv != 0 ? levels.at(lv - 1).b : init_b;
            _smooth(lv, x, b);

            copy(levels.at(lv).residual, b);
            spmv(levels.at(lv).residual, -1, levels.at(lv).A, x, 1, buff); // residual = b - A@x

            levels.at(lv).b.resize(levels.at(lv).R.nrows);
            spmv(levels.at(lv).b, 1, levels.at(lv).R, levels.at(lv).residual, 0, buff); // coarse_b = R@residual

            levels.at(lv).x.resize(levels.at(lv).b.size());
            zero(levels.at(lv).x);
        }
    }

    void vcycle_up() {
        for (int lv = nlvs-2; lv >= 0; --lv) {
            Vec<float> &x = lv != 0 ? levels.at(lv - 1).x : init_x;
            Vec<float> &b = lv != 0 ? levels.at(lv - 1).b : init_b;
            spmv(x, 1, levels.at(lv).P, levels.at(lv).x, 1, buff); // x += P@coarse_x
            _smooth(lv, x, b);
        }
    }

    void vcycle() {
        vcycle_down();
        coarse_solve();
        vcycle_up();
    }

    size_t get_coarsist_size() {
        auto const &this_b = levels.at(nlvs - 2).b;
        return this_b.size();
    }

    void get_coarsist_b(float *b) {
        auto const &this_b = levels.at(nlvs - 2).b;
        CHECK_CUDA(hipMemcpy(b, this_b.data(), this_b.size() * sizeof(float), hipMemcpyDeviceToHost));
    }

    void get_finest_x(float *x) {
        CHECK_CUDA(hipMemcpy(x, init_x.data(), init_x.size() * sizeof(float), hipMemcpyDeviceToHost));
    }

    void set_coarsist_x(float const *x) {
        auto const &this_b = levels.at(nlvs - 2).b;
        auto &this_x = levels.at(nlvs - 2).x;
        this_x.resize(this_b.size());
        CHECK_CUDA(hipMemcpy(this_x.data(), x, this_x.size() * sizeof(float), hipMemcpyHostToDevice));
    }

    void coarse_solve() {
        auto const &A = levels.at(nlvs - 1).A;
        auto &x = levels.at(nlvs - 2).x;
        auto &b = levels.at(nlvs - 2).b;
        spsolve(x, A, b);
    }

    void copy_outer2init_x() {
        copy(init_x, outer_x);
    }

    void set_outer_x(float const *x, size_t n) {
        outer_x.resize(n);
        CHECK_CUDA(hipMemcpy(outer_x.data(), x, n * sizeof(float), hipMemcpyHostToDevice));
        copy(alter_x, outer_x);
    }

    void set_outer_b(float const *b, size_t n) {
        outer_b.resize(n);
        CHECK_CUDA(hipMemcpy(outer_b.data(), b, n * sizeof(float), hipMemcpyHostToDevice));
    }

    float init_cg_iter0(float *residuals) {
        float bnrm2 = vnorm(outer_b);
        // r = b - A@(x)
        copy(init_b, outer_b);
        spmv(outer_b, -1, levels.at(0).A, outer_x, 1, buff);
        float normr = vnorm(init_b);
        residuals[0] = normr;
        return bnrm2;
    }

    void do_cg_itern(float *residuals, size_t iteration) {
        float rho_cur = vdot(init_b, init_x);
        if (iteration > 0) {
            float beta = rho_cur / save_rho_prev;
            // p *= beta
            // p += z
            scal(save_p, beta);
            axpy(save_p, 1, init_x);
        } else {
            // p = move(z)
            save_p.swap(init_x);
        }
        // q = A@(p)
        save_q.resize(levels.at(0).A.nrows);
        spmv(save_q, 1, levels.at(0).A, save_p, 0, buff);
        save_rho_prev = rho_cur;
        float alpha = rho_cur / vdot(save_p, save_q);
        // x += alpha*p
        axpy(alter_x, alpha, save_p);
        // r -= alpha*q
        axpy(init_b, -alpha, save_q);
        float normr = vnorm(init_b);
        residuals[iteration + 1] = normr;
    }

    void fetch_cg_final_x(float *x) {
        CHECK_CUDA(hipMemcpy(x, alter_x.data(), alter_x.size() * sizeof(float), hipMemcpyDeviceToHost));
    }

    void fetch_cg_final_r(float *r) {
        // CHECK_CUDA(hipMemcpy(r, residuals.data(), residuals.size() * sizeof(float), hipMemcpyDeviceToHost));
        std::copy(residuals.begin(), residuals.end(), r);
    }

    void compute_RAP(size_t lv) {
            CSR<float> &A = levels.at(lv).A;
            CSR<float> &R = levels.at(lv).R;
            CSR<float> &P = levels.at(lv).P;
            CSR<float> AP;
            CSR<float> &RAP = levels.at(lv+1).A;
            spgemm(A, P, AP) ;
            spgemm(R, AP, RAP);
    }

    void fetch_A(size_t lv, float *data, int *indices, int *indptr) {
        CSR<float> &A = levels.at(lv).A;
        CHECK_CUDA(hipMemcpy(data, A.data.data(), A.data.size() * sizeof(float), hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(indices, A.indices.data(), A.indices.size() * sizeof(int), hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(indptr, A.indptr.data(), A.indptr.size() * sizeof(int), hipMemcpyDeviceToHost));
    }
    
    void set_mgcg_data(const float* x, size_t nx, const float* b, size_t nb, float rtol_, size_t maxiter_)
    {
        set_outer_x(x, nx);
        set_outer_b(b, nb);
        rtol = rtol_;
        maxiter = maxiter_;
        residuals.resize(maxiter+1);
    }

    size_t get_mgcg_data(float* x_, float* r_)
    {
        fetch_cg_final_x(x_);
        fetch_cg_final_r(r_);
        return niter;
    }

    void mgcg_solve()
    {
        float bnrm2 = init_cg_iter0(residuals.data());
        float atol = bnrm2 * rtol;
        for (size_t iteration=0; iteration<maxiter; iteration++)
        {   
            if (residuals[iteration] < atol)
            {
                niter = iteration; //number of iter to break
                break;
            }
            copy_outer2init_x();  //reset x to x0
            vcycle();
            do_cg_itern(residuals.data(), iteration); //first r is r[0], then r[iter+1]
        }
    }


};

struct AssembleMatrix : Kernels {
    CSR<float> A;
    CSR<float> G;
    CSR<float> M;
    CSR<float> ALPHA;
    float alpha;
    int NE;

    void fetch_A(float *data, int *indices, int *indptr) {
        CHECK_CUDA(hipMemcpy(data, A.data.data(), A.data.size() * sizeof(float), hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(indices, A.indices.data(), A.indices.size() * sizeof(int), hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(indptr, A.indptr.data(), A.indptr.size() * sizeof(int), hipMemcpyDeviceToHost));
    }

    void set_G(float const *datap, int const *indicesp, int const *indptrp, int rows, int cols, int nnz) {
        G.assign(datap, nnz, indicesp, nnz, indptrp, rows + 1, rows, cols, nnz);
    }

    void set_M(float const *datap, int const *indicesp, int const *indptrp, int rows, int cols, int nnz) {
        M.assign(datap, nnz, indicesp, nnz, indptrp, rows + 1, rows, cols, nnz);
    }

    void set_ALPHA(float const *datap, int const *indicesp, int const *indptrp, int rows, int cols, int nnz) {
        ALPHA.assign(datap, nnz, indicesp, nnz, indptrp, rows + 1, rows, cols, nnz);
    }

    void compute_GMG() {
        CSR<float> GM;
        spgemm(G, M, GM);
        CSR<float> GT;
        GT.resize(G.ncols, G.nrows, G.numnonz);
        transpose(G, GT);
        spgemm(GM, GT, A);
    }

};

} // namespace

static VCycle *fastmg = nullptr;
static AssembleMatrix *fastA = nullptr;

#if _WIN32
#define DLLEXPORT __declspec(dllexport)
#else
#define DLLEXPORT
#endif

extern "C" DLLEXPORT void fastmg_setup(size_t numlvs) {
    if (!fastmg)
        fastmg = new VCycle{};
    fastmg->setup(numlvs);
}

extern "C" DLLEXPORT void fastmg_set_coeff(float const *coeff, size_t ncoeffs) {
    fastmg->set_coeff(coeff, ncoeffs);
}

extern "C" DLLEXPORT void fastmg_set_lv_csrmat(size_t lv, size_t which, float const *datap, size_t ndat, int const *indicesp, size_t nind, int const *indptrp, size_t nptr, size_t rows, size_t cols, size_t nnz) {
    fastmg->set_lv_csrmat(lv, which, datap, ndat, indicesp, nind, indptrp, nptr, rows, cols, nnz);
}

extern "C" DLLEXPORT void fastmg_RAP(size_t lv) {
    fastmg->compute_RAP(lv);
}

extern "C" DLLEXPORT void fastmg_fetch_A(size_t lv, float* data, int* indices, int* indptr) {
    fastmg->fetch_A(lv, data, indices, indptr);
}

extern "C" DLLEXPORT void fastmg_vcycle() {
    fastmg->vcycle();
}

extern "C" DLLEXPORT void fastmg_mgcg_solve() {
    fastmg->mgcg_solve();
}

extern "C" DLLEXPORT void fastmg_set_mgcg_data(const float* x, size_t nx, const float* b, size_t nb, float rtol, size_t maxiter) {
    fastmg->set_mgcg_data(x, nx, b, nb, rtol, maxiter);
}

extern "C" DLLEXPORT size_t fastmg_get_mgcg_data(float *x, float *r) {
    size_t niter = fastmg->get_mgcg_data(x, r);
    return niter;
}

// ------------------------------------------------------------------------------
extern "C" DLLEXPORT void fastA_setup() {
    if (!fastA)
        fastA = new AssembleMatrix{};
}

extern "C" DLLEXPORT void fastA_set_G(float* data, int* indices, int* indptr, int rows, int cols, int nnz)
{
    fastA->set_G(data, indices, indptr, rows, cols, nnz);
}

extern "C" DLLEXPORT void fastA_set_M(float* data, int* indices, int* indptr, int rows, int cols, int nnz)
{
    fastA->set_M(data, indices, indptr, rows, cols, nnz);
}

extern "C" DLLEXPORT void fastA_set_ALPHA(float* data, int* indices, int* indptr, int rows, int cols, int nnz)
{
    fastA->set_ALPHA(data, indices, indptr, rows, cols, nnz);
}

extern "C" DLLEXPORT void fastA_compute_GMG() {
    fastA->compute_GMG();
}

extern "C" DLLEXPORT void fastA_fetch_A(float* data, int* indices, int* indptr) {
    fastA->fetch_A(data, indices, indptr);
}