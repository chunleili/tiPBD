#include "hip/hip_runtime.h"
#include <algorithm>
#include <cassert>
#include <cstring>
#include <fstream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsparse.h>
#include <hipsolver.h>
#include <iostream>
#include <string>
#include <sstream>
#include <cstdio>
#include <cmath>
#include <chrono>
#include <array>
#include <unordered_set>
#include <unordered_map>
#include <map>
#include <set>
#include <numeric>

#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/inner_product.h>
#include <thrust/random.h>

// Terminal output color (just for cosmetic purpose)
#define RST  "\x1B[37m"  // Reset color to white
#define KGRN  "\033[0;32m"   // Define green color
#define RD "\x1B[31m"  // Define red color
#define FGRN(x) KGRN x RST  // Define compiler function for green color
#define FRD(x) RD x RST  // Define compiler function for red color

using namespace std;

#if __GNUC__ && __linux__
#include <sys/ptrace.h>

[[noreturn]] static void cuerr() {
    if (ptrace(PTRACE_TRACEME, 0, NULL, NULL) != 0)
        __builtin_trap();
    exit(EXIT_FAILURE);
}
#elif _WIN32 && _MSC_VER
#include <windows.h>

[[noreturn]] static void cuerr() {
    int debugger_present = 0;
    HANDLE process = GetCurrentProcess();
    CheckRemoteDebuggerPresent(process, &debugger_present);
    if (debugger_present) {
        __debugbreak();
    }
    exit(EXIT_FAILURE);
}
#else
[[noreturn]] static void cuerr() {
    exit(EXIT_FAILURE);
}
#endif

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("%s:%d: %s (%d): %s\n", __FILE__, __LINE__,                     \
               hipGetErrorString(status), status, #func);                     \
        cuerr();                                                               \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("%s:%d: %s (%d): %s\n", __FILE__, __LINE__,                     \
               hipsparseGetErrorString(status), status, #func);                 \
        cuerr();                                                               \
    }                                                                          \
}

#define CHECK_CUBLAS(func)                                                     \
{                                                                              \
    hipblasStatus_t status = (func);                                            \
    if (status != HIPBLAS_STATUS_SUCCESS) {                                     \
        printf("%s:%d: %s (%d): %s\n", __FILE__, __LINE__,                     \
               cublasGetStatusString(status), status, #func);                  \
        cuerr();                                                               \
    }                                                                          \
}


// https://github.com/NVIDIA/CUDALibrarySamples/blob/ed19a07b6dd0900b7547b274a6ed9d7c22a6d431/cuSOLVER/utils/cusolver_utils.h#L55
#define CHECK_CUSOLVER(err)                                                                        \
    do {                                                                                           \
        hipsolverStatus_t err_ = (err);                                                             \
        if (err_ != HIPSOLVER_STATUS_SUCCESS) {                                                     \
            printf("cusolver error %d at %s:%d\n", err_, __FILE__, __LINE__);                      \
            throw std::runtime_error("cusolver error");                                            \
        }                                                                                          \
    } while (0)



// Generate random number in the range [0, 1)
struct genRandomNumber {
    __host__ __device__
    float operator()(const int n) const {
        thrust::default_random_engine rng(n);
        thrust::uniform_real_distribution<float> dist(0.0f, 1.0f);
        return dist(rng);
    }
};



/// @brief Usage: Timer t("timer_name");
///               t.start();
///               //do something
///               t.end();
/// You need to include <chrono> and <string> for this to work
class Timer
{
public:
    std::chrono::time_point<std::chrono::steady_clock> m_start;
    std::chrono::time_point<std::chrono::steady_clock> m_end;
    std::chrono::duration<double, std::milli> elapsed_ms;
    std::chrono::duration<double> elapsed_s;
    double elapsed=0.0;
    std::string name = "";

    Timer(std::string name = "") : name(name){};
    inline void start()
    {
        m_start = std::chrono::steady_clock::now();
    };
    inline void end(std::string msg = "", std::string unit = "ms", bool verbose=true, std::string endsep = "\n")
    {
        m_end = std::chrono::steady_clock::now();
        if (unit == "s")
        {
            elapsed_s = m_end - m_start;
            if(verbose)
                printf("%s(%s): %.0f(s)", msg.c_str(), name.c_str(), elapsed_s.count());
            else
                printf("%.0f(s)", elapsed_s.count());
        }
        else //else if(unit == "ms")
        {
            elapsed_ms = m_end - m_start;
            if(verbose)
                printf("%s(%s): %.0f(ms)", msg.c_str(), name.c_str(), elapsed_ms.count());
            else
                printf("%.0f(ms)", elapsed_ms.count());
        }
        printf("%s", endsep.c_str());
    }
    inline void reset()
    {
        m_start = std::chrono::steady_clock::now();
        m_end = std::chrono::steady_clock::now();
        elapsed = 0.0;
    };
    inline void accumulate()
    {
        m_end = std::chrono::steady_clock::now();
        elapsed += std::chrono::duration<double, std::milli>(m_end - m_start).count();
    };
    inline void report()
    {
        std::cout << name << ": " << elapsed << " ms" << std::endl;
    };
    
};


// https://stackoverflow.com/a/41154786/19253199
// https://github.com/aramadia/udacity-cs344/blob/master/Unit2%20Code%20Snippets/gputimer.h
// https://developer.nvidia.com/blog/how-implement-performance-metrics-cuda-cc/
/// @brief Usage: 
///     GpuTimer  timer;
///     timer.start(); 
///     do something
///     timer.stop(); 
///     float elapsedTime = timer.elapsed(); 
///     printf("Elapsed time : %.2f ms\n" ,elapsedTime);
struct GpuTimer
{
      hipEvent_t m_start;
      hipEvent_t m_stop;

      GpuTimer()
      {
            hipEventCreate(&m_start);
            hipEventCreate(&m_stop);
      }

      ~GpuTimer()
      {
            hipEventDestroy(m_start);
            hipEventDestroy(m_stop);
      }

      void start()
      {
            hipEventRecord(m_start, 0);
      }

      void stop()
      {
            hipEventRecord(m_stop, 0);
      }

      float elapsed()
      {
            float elapsed_ms;
            hipEventSynchronize(m_stop);
            hipEventElapsedTime(&elapsed_ms, m_start, m_stop);
            return elapsed_ms;
      }
};



namespace {

struct Buffer {
    void *m_data;
    size_t m_cap;

    Buffer() noexcept : m_data(nullptr), m_cap(0) {
    }

    Buffer(Buffer &&that) noexcept : m_data(that.m_data), m_cap(that.m_cap) {
        that.m_data = nullptr;
        that.m_cap = 0;
    }

    Buffer &operator=(Buffer &&that) noexcept {
        if (this == &that) return *this;
        if (m_data)
            CHECK_CUDA(hipFree(m_data));
        m_data = nullptr;
        m_data = that.m_data;
        m_cap = that.m_cap;
        that.m_data = nullptr;
        that.m_cap = 0;
        return *this;
    }

    ~Buffer() noexcept {
        if (m_data)
            CHECK_CUDA(hipFree(m_data));
        m_data = nullptr;
    }

    void reserve(size_t new_cap) {
        if (m_cap < new_cap) {
            if (m_data)
                CHECK_CUDA(hipFree(m_data));
            m_data = nullptr;
            CHECK_CUDA(hipMalloc(&m_data, new_cap));
            m_cap = new_cap;
        }
    }

    size_t capacity() const noexcept {
        return m_cap;
    }

    void const *data() const noexcept {
        return m_data;
    }

    void *data() noexcept {
        return m_data;
    }
};

template <class T>
hipDataType cudaDataTypeFor();

template <>
hipDataType cudaDataTypeFor<int8_t>() {
    return HIP_R_8I;
}

template <>
hipDataType cudaDataTypeFor<uint8_t>() {
    return HIP_R_8U;
}

template <>
hipDataType cudaDataTypeFor<int16_t>() {
    return HIP_R_16I;
}

template <>
hipDataType cudaDataTypeFor<uint16_t>() {
    return HIP_R_16U;
}

template <>
hipDataType cudaDataTypeFor<int32_t>() {
    return HIP_R_32I;
}

template <>
hipDataType cudaDataTypeFor<uint32_t>() {
    return HIP_R_32U;
}

template <>
hipDataType cudaDataTypeFor<int64_t>() {
    return HIP_R_64I;
}

template <>
hipDataType cudaDataTypeFor<uint64_t>() {
    return HIP_R_64U;
}

template <>
hipDataType cudaDataTypeFor<nv_half>() {
    return HIP_R_16F;
}

template <>
hipDataType cudaDataTypeFor<hip_bfloat16>() {
    return HIP_R_16BF;
}

template <>
hipDataType cudaDataTypeFor<float>() {
    return HIP_R_32F;
}

template <>
hipDataType cudaDataTypeFor<double>() {
    return HIP_R_64F;
}



/* -------------------------------------------------------------------------- */
/*                                   kernels                                  */
/* -------------------------------------------------------------------------- */



__device__ float3 inline d_normalize_diff(std::array<float,3> &v1,  std::array<float,3> &v2)
{
    std::array<float,3> diff = {v1[0]-v2[0], v1[1]-v2[1], v1[2]-v2[2]};
    float norm = sqrt(diff[0]*diff[0] + diff[1]*diff[1] + diff[2]*diff[2]);
    return {diff[0]/norm, diff[1]/norm, diff[2]/norm};
}

__device__ float inline d_dot(std::array<float,3> a, std::array<float,3> b)
{
    return a[0]*b[0] + a[1]*b[1] + a[2]*b[2];
}



// def fill_A_CSR_kernel(data:ti.types.ndarray(dtype=ti.f32), 
//                               indptr:ti.types.ndarray(dtype=ti.i32), 
//                               ii:ti.types.ndarray(dtype=ti.i32), 
//                               jj:ti.types.ndarray(dtype=ti.i32),
//                               adjacent_edge_abc:ti.types.ndarray(dtype=ti.i32),
//                               num_nonz:ti.i32,
//                               alpha:ti.f32):
//     for cnt in range(num_nonz):
//         i = ii[cnt] # row index
//         j = jj[cnt] # col index
//         k = cnt - indptr[i] #k-th non-zero element of i-th row. 
//         # Because the diag is the final element of each row, 
//         # it is also the k-th adjacent edge of i-th edge.
//         if i == j: # diag
//             data[cnt] = inv_mass[edge[i][0]] + inv_mass[edge[i][1]] + alpha
//             continue
//         a = adjacent_edge_abc[i, k * 3]
//         b = adjacent_edge_abc[i, k * 3 + 1]
//         c = adjacent_edge_abc[i, k * 3 + 2]
//         g_ab = (pos[a] - pos[b]).normalized()
//         g_ac = (pos[a] - pos[c]).normalized()
//         offdiag = inv_mass[a] * g_ab.dot(g_ac)
//         data[cnt] = offdiag
// __global__ void fill_A_CSR_kernel(thrust::device_vector<float> &data, 
//                                   thrust::device_vector<int> indptr, 
//                                   thrust::device_vector<int> ii, 
//                                   thrust::device_vector<int> jj,
//                                   thrust::device_vector<thrust::device_vector<int>> adjacent_edge_abc,
//                                   int num_nonz,
//                                   float alpha,
//                                   thrust::device_vector<float3> pos,
//                                   thrust::device_vector<float> inv_mass) {
//     size_t cnt = blockIdx.x * blockDim.x + threadIdx.x;
//     if (cnt < num_nonz) {
//         int i = ii[cnt]; // row index
//         int j = jj[cnt]; // col index
//         int k = cnt - indptr[i]; //k-th non-zero element of i-th row. 
//         // Because the diag is the final element of each row, 
//         // it is also the k-th adjacent edge of i-th edge.
//         if (i == j) { // diag
//             data[cnt] = inv_mass[i] + inv_mass[i] + alpha;
//             return;
//         }
//         int a = adjacent_edge_abc[i][k * 3];
//         int b = adjacent_edge_abc[i][k * 3 + 1];
//         int c = adjacent_edge_abc[i][k * 3 + 2];
//         float3 g_ab = d_normalize_diff(pos[a], pos[b]);
//         float3 g_ac = d_normalize_diff(pos[a], pos[c]);
//         float offdiag = inv_mass[a] * d_dot(g_ab, g_ac);
//         data[cnt] = offdiag;
//     }
// }




// weighted Jacobi for csr matrix
// https://en.wikipedia.org/wiki/Jacobi_method#Weighted_Jacobi_method
// https://stackoverflow.com/questions/78057439/jacobi-algorithm-using-cuda
// https://github.com/pyamg/pyamg/blob/5a51432782c8f96f796d7ae35ecc48f81b194433/pyamg/amg_core/relaxation.h#L232
// i: row index, j: col index, n: data/indices index
// rsum: sum of off-diagonal elements
__global__ void weighted_jacobi_kernel(float *x, float *x_old, const float *b, float *data, int *indices, int *indptr, int nrows, float omega) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < nrows) {
        float rsum = 0.0;
        float diag = 0.0;
        for (size_t n = indptr[i]; n < indptr[i + 1]; ++n) {
            size_t j = indices[n];
            if (j != i) {
                rsum += data[n] * x_old[j];
            }
            else {
                diag = data[n];
            }
        }
        // FIXME: should use x_new to avoid race condition
        if (diag != 0.0)
        {
            x[i] =  omega / diag * (b[i] - rsum)  + (1.0 - omega) * x_old[i];
        }
    }
}

__global__ void copy_field(float *dst, const float *src, int size) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        dst[i] = src[i];
    }
}


void jacobi_serial(const int Ap[], const int Ap_size,
            const int Aj[], const int Aj_size,
            const float Ax[], const int Ax_size,
                  float  x[], const int  x_size,
            const float  b[], const int  b_size,
                  float temp[], const int temp_size,
            const int row_start,
            const int row_stop,
            const int row_step,
            const float omega)
{
    float one = 1.0;

    for(int i = row_start; i != row_stop; i += row_step) {
        temp[i] = x[i];
    }

    for(int i = row_start; i != row_stop; i += row_step) {
        int start = Ap[i];
        int end   = Ap[i+1];
        float rsum = 0;
        float diag = 0;

        for(int jj = start; jj < end; jj++){
            int j = Aj[jj];
            if (i == j)
                diag  = Ax[jj];
            else
                rsum += Ax[jj]*temp[j];
        }

        if (diag != (float) 0.0){
            x[i] = (one - omega) * temp[i] + omega * ((b[i] - rsum)/diag);
        }
    }
}



// https://github.com/pyamg/pyamg/blob/5a51432782c8f96f796d7ae35ecc48f81b194433/pyamg/amg_core/relaxation.h#L45
void gauss_seidel_serial(const int Ap[], const int Ap_size,
                  const int Aj[], const int Aj_size,
                  const float Ax[], const int Ax_size,
                        float  x[], const int  x_size,
                  const float  b[], const int  b_size,
                  const int row_start,
                  const int row_stop,
                  const int row_step)
{
    for(int i = row_start; i != row_stop; i += row_step) {
        int start = Ap[i];
        int end   = Ap[i+1];
        float rsum = 0;
        float diag = 0;

        for(int jj = start; jj < end; jj++){
            int j = Aj[jj];
            if (i == j)
                diag  = Ax[jj];
            else
                rsum += Ax[jj]*x[j];
        }

        if (diag != (float) 0.0){
            x[i] = (b[i] - rsum)/diag;
        }
    }
}

template <class T>
struct Vec {
    T *m_data;
    size_t m_size;
    size_t m_cap;

    Vec() noexcept : m_data(nullptr), m_size(0), m_cap(0) {
    }

    Vec(Vec &&that) noexcept : m_data(that.m_data), m_size(that.m_size), m_cap(that.m_cap) {
        that.m_data = nullptr;
        that.m_size = 0;
        that.m_cap = 0;
    }

    Vec &operator=(Vec &&that) noexcept {
        if (this == &that) return *this;
        if (m_data)
            CHECK_CUDA(hipFree(m_data));
        m_data = nullptr;
        m_data = that.m_data;
        m_size = that.m_size;
        m_cap = that.m_cap;
        that.m_data = nullptr;
        that.m_size = 0;
        that.m_cap = 0;
        return *this;
    }

    void swap(Vec &that) noexcept {
        std::swap(m_data, that.m_data);
        std::swap(m_size, that.m_size);
        std::swap(m_cap, that.m_cap);
    }

    ~Vec() noexcept {
        if (m_data)
            CHECK_CUDA(hipFree(m_data));
        m_data = nullptr;
    }

    void resize(size_t new_size) {
        bool change = m_cap < new_size;
        if (change) {
            if (m_data)
                CHECK_CUDA(hipFree(m_data));
            m_data = nullptr;
            CHECK_CUDA(hipMalloc(&m_data, sizeof(T) * new_size));
            m_cap = new_size;
        }
        if (m_size != new_size || change) {
            m_size = new_size;
        }
    }

    // host to device
    void assign(T const *data, size_t size) {
        resize(size);
        CHECK_CUDA(hipMemcpy(m_data, data, sizeof(T) * size, hipMemcpyHostToDevice));
    }

    // device to host
    void tohost(std::vector<T> &data_host) const{
        data_host.resize(size());
        CHECK_CUDA(hipMemcpy(data_host.data(), m_data, sizeof(T) * size(), hipMemcpyDeviceToHost));
    }

    size_t size() const noexcept {
        return m_size;
    }

    T const *data() const noexcept {
        return m_data;
    }

    T *data() noexcept {
        return m_data;
    }
};

struct DnVec {
    hipsparseDnVecDescr_t handle;

    operator hipsparseDnVecDescr_t() const noexcept {
        return handle;
    }

    DnVec() noexcept : handle(0) {}

    template <class T>
    DnVec(Vec<T> &v) {
        CHECK_CUSPARSE(hipsparseCreateDnVec(&handle, v.size(), v.data(), cudaDataTypeFor<T>()));
    }

    DnVec(DnVec &&that) noexcept : handle(that.handle) {
        that.handle = nullptr;
    }

    DnVec &operator=(DnVec &&that) noexcept {
        if (this == &that) return *this;
        if (handle)
            CHECK_CUSPARSE(hipsparseDestroyDnVec(handle));
        handle = that.handle;
        that.handle = nullptr;
        return *this;
    }

    ~DnVec() {
        if (handle)
            CHECK_CUSPARSE(hipsparseDestroyDnVec(handle));
    }
};

struct ConstDnVec {
    hipsparseConstDnVecDescr_t handle;

    operator hipsparseConstDnVecDescr_t() const noexcept {
        return handle;
    }

    ConstDnVec() noexcept : handle(0) {}

    template <class T>
    ConstDnVec(Vec<T> const &v) {
        CHECK_CUSPARSE(hipsparseCreateConstDnVec(&handle, v.size(), v.data(), cudaDataTypeFor<T>()));
    }

    ConstDnVec(ConstDnVec &&that) noexcept : handle(that.handle) {
        that.handle = nullptr;
    }

    ConstDnVec &operator=(ConstDnVec &&that) noexcept {
        if (this == &that) return *this;
        if (handle)
            CHECK_CUSPARSE(hipsparseDestroyDnVec(handle));
        handle = that.handle;
        that.handle = nullptr;
        return *this;
    }

    ~ConstDnVec() {
        if (handle)
            CHECK_CUSPARSE(hipsparseDestroyDnVec(handle));
    }
};


// Data of csr matrix
template <class T>
struct CSR {
    Vec<T> data;
    Vec<int> indices;
    Vec<int> indptr;
    int64_t nrows;
    int64_t ncols;
    int64_t numnonz;

    CSR() noexcept : nrows(0), ncols(0), numnonz(0) {}

    void assign(T const *datap, size_t ndat, int const *indicesp, size_t nind, int const *indptrp, size_t nptr, size_t rows, size_t cols, size_t nnz) {
        indices.resize(nind);
        indptr.resize(nptr);
        data.resize(ndat);
        CHECK_CUDA(hipMemcpy(data.data(), datap, data.size() * sizeof(T), hipMemcpyHostToDevice));
        CHECK_CUDA(hipMemcpy(indices.data(), indicesp, indices.size() * sizeof(int), hipMemcpyHostToDevice));
        CHECK_CUDA(hipMemcpy(indptr.data(), indptrp, indptr.size() * sizeof(int), hipMemcpyHostToDevice));
        nrows = rows;
        ncols = cols;
        numnonz = nnz;
    }

    void resize(size_t rows, size_t cols, size_t nnz) {
        nrows = rows;
        ncols = cols;
        numnonz = nnz;
        data.resize(nnz);
        indices.resize(nnz);
        indptr.resize(rows + 1);
    }

    void tohost(std::vector<T> &data_host, std::vector<int> &indices_host, std::vector<int> &indptr_host) const {
        data_host.resize(data.size());
        indices_host.resize(indices.size());
        indptr_host.resize(indptr.size());
        CHECK_CUDA(hipMemcpy(data_host.data(), data.data(), data.size() * sizeof(T), hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(indices_host.data(), indices.data(), indices.size() * sizeof(int), hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(indptr_host.data(), indptr.data(), indptr.size() * sizeof(int), hipMemcpyDeviceToHost));
    }
};


// template <class T>
// struct SuperCSR:CSR<T>
// {
//     Vec<T> ii;
//     Vec<T> jj;

//     SuperCSR() noexcept : nrows(0), ncols(0), numnonz(0) {}

//     void assign(T const *datap, size_t ndat, int const *indicesp, size_t nind, int const *indptrp, size_t nptr, size_t rows, size_t cols, size_t nnz, T const *iip, T const *jjp) {
//         indices.resize(nind);
//         indptr.resize(nptr);
//         data.resize(ndat);
//         CHECK_CUDA(hipMemcpy(data.data(), datap, data.size() * sizeof(T), hipMemcpyHostToDevice));
//         CHECK_CUDA(hipMemcpy(indices.data(), indicesp, indices.size() * sizeof(int), hipMemcpyHostToDevice));
//         CHECK_CUDA(hipMemcpy(indptr.data(), indptrp, indptr.size() * sizeof(int), hipMemcpyHostToDevice));
//         nrows = rows;
//         ncols = cols;
//         numnonz = nnz;

//         ii.resize(nnz);
//         jj.resize(nnz);
//         CHECK_CUDA(hipMemcpy(ii.data(), iip, ii.size() * sizeof(T), hipMemcpyHostToDevice));
//         CHECK_CUDA(hipMemcpy(jj.data(), jjp, jj.size() * sizeof(T), hipMemcpyHostToDevice));
//     }

//     void resize(size_t rows, size_t cols, size_t nnz) {
//         nrows = rows;
//         ncols = cols;
//         numnonz = nnz;
//         data.resize(nnz);
//         indices.resize(nnz);
//         indptr.resize(rows + 1);

//         ii.resize(nnz);
//         jj.resize(nnz);
//     }
// };



// container of handle and descriptor
struct SpMat {
    hipsparseSpMatDescr_t handle;

    operator hipsparseSpMatDescr_t() const noexcept {
        return handle;
    }

    SpMat() noexcept : handle(0) {}

    template <class T>
    SpMat(CSR<T> &m) {
        CHECK_CUSPARSE(hipsparseCreateCsr(&handle, m.nrows, m.ncols, m.numnonz,
                                         m.indptr.data(), m.indices.data(), m.data.data(),
                                         HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                         HIPSPARSE_INDEX_BASE_ZERO, cudaDataTypeFor<T>()) );
    }

    SpMat(SpMat &&that) noexcept : handle(that.handle) {
        that.handle = nullptr;
    }

    SpMat &operator=(SpMat &&that) noexcept {
        if (this == &that) return *this;
        if (handle)
            CHECK_CUSPARSE(hipsparseDestroySpMat(handle));
        handle = that.handle;
        that.handle = nullptr;
        return *this;
    }

    ~SpMat() {
        if (handle)
            CHECK_CUSPARSE(hipsparseDestroySpMat(handle));
    }
};

// container of handle and descriptor, const version
struct ConstSpMat {
    hipsparseConstSpMatDescr_t handle;

    operator hipsparseConstSpMatDescr_t() const noexcept {
        return handle;
    }

    ConstSpMat() noexcept : handle(0) {}

    template <class T>
    ConstSpMat(CSR<T> const &m) {
        CHECK_CUSPARSE(hipsparseCreateConstCsr(&handle, m.nrows, m.ncols, m.numnonz,
                                              m.indptr.data(), m.indices.data(), m.data.data(),
                                              HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                              HIPSPARSE_INDEX_BASE_ZERO, cudaDataTypeFor<T>()) );
    }

    ConstSpMat(SpMat &&that) noexcept : handle(that.handle) {
        that.handle = nullptr;
    }

    ConstSpMat &operator=(ConstSpMat &&that) noexcept {
        if (this == &that) return *this;
        if (handle)
            CHECK_CUSPARSE(hipsparseDestroySpMat(handle));
        handle = that.handle;
        that.handle = nullptr;
        return *this;
    }

    ~ConstSpMat() {
        if (handle)
            CHECK_CUSPARSE(hipsparseDestroySpMat(handle));
    }
};

struct Kernels {
    hipblasHandle_t cublas;
    hipsparseHandle_t cusparse;
    hipsolverSpHandle_t cusolverH;

    Kernels() {
        CHECK_CUSPARSE(hipsparseCreate(&cusparse));
        CHECK_CUBLAS(hipblasCreate(&cublas));
        CHECK_CUSOLVER(hipsolverSpCreate(&cusolverH));
    }

    Kernels(Kernels &&) = delete;

    ~Kernels() {
        CHECK_CUSPARSE(hipsparseDestroy(cusparse));
        CHECK_CUBLAS(hipblasDestroy(cublas));
        CHECK_CUSOLVER(hipsolverSpDestroy(cusolverH));
    }

    // out = alpha * A@x + beta * out
    void spmv(Vec<float> &out, float const &alpha, CSR<float> const &A, Vec<float> const &x, float const &beta, Buffer &buffer) {
        assert(out.size() == A.nrows);
        size_t bufSize = 0;
        ConstSpMat dA(A);
        ConstDnVec dx(x);
        DnVec dout(out);
        CHECK_CUSPARSE(hipsparseSpMV_bufferSize(cusparse, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                               &alpha, dA, dx, &beta,
                                               dout, cudaDataTypeFor<float>(),
                                               HIPSPARSE_SPMV_ALG_DEFAULT, &bufSize));
        buffer.reserve(bufSize);
        CHECK_CUSPARSE(hipsparseSpMV(cusparse, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    &alpha, dA, dx, &beta,
                                    dout, cudaDataTypeFor<float>(),
                                    HIPSPARSE_SPMV_ALG_DEFAULT, buffer.data()));
    }

    // C = A * B
    void spgemm(CSR<float> const &matA_,  CSR<float> const &matB_, CSR<float> &matC_) 
    {
        ConstSpMat descA(matA_); //descriptor for A
        ConstSpMat descB(matB_);
        matC_.resize(matA_.nrows, matB_.ncols, 0);
        SpMat descC(matC_);
        // https://github.com/NVIDIA/CUDALibrarySamples/blob/ade391a17672d26e55429035450bc44afd277d34/cuSPARSE/spgemm/spgemm_example.c#L161
        // https://docs.nvidia.com/cuda/cusparse/#cusparsespgemm
        //--------------------------------------------------------------------------
        float               alpha       = 1.0f;
        float               beta        = 0.0f;
        hipsparseOperation_t opA         = HIPSPARSE_OPERATION_NON_TRANSPOSE;
        hipsparseOperation_t opB         = HIPSPARSE_OPERATION_NON_TRANSPOSE;
        hipDataType        computeType = HIP_R_32F;
        //--------------------------------------------------------------------------
        // buffers
        size_t bufferSize1 = 0,    bufferSize2 = 0;
        Buffer dBuffer1, dBuffer2;
        //--------------------------------------------------------------------------
        // SpGEMM Computation
        hipsparseSpGEMMDescr_t spgemmDesc;
        CHECK_CUSPARSE( hipsparseSpGEMM_createDescr(&spgemmDesc) )

        // ask bufferSize1 bytes for external memory
        CHECK_CUSPARSE(
            hipsparseSpGEMM_workEstimation(cusparse, opA, opB,
                                        &alpha, descA, descB, &beta, descC,
                                        computeType, HIPSPARSE_SPGEMM_DEFAULT,
                                        spgemmDesc, &bufferSize1, NULL) )
        // CHECK_CUDA( hipMalloc((void**) &dBuffer1, bufferSize1) )
        dBuffer1.reserve(bufferSize1);

        // inspect the matrices A and B to understand the memory requirement for
        // the next step
        CHECK_CUSPARSE(
            hipsparseSpGEMM_workEstimation(cusparse, opA, opB,
                                        &alpha, descA, descB, &beta, descC,
                                        computeType, HIPSPARSE_SPGEMM_DEFAULT,
                                        spgemmDesc, &bufferSize1, dBuffer1.data()) )

        // ask bufferSize2 bytes for external memory
        CHECK_CUSPARSE(
            hipsparseSpGEMM_compute(cusparse, opA, opB,
                                &alpha, descA, descB, &beta, descC,
                                computeType, HIPSPARSE_SPGEMM_DEFAULT,
                                spgemmDesc, &bufferSize2, NULL) )
        dBuffer2.reserve(bufferSize2);

        // compute the intermediate product of A * B
        CHECK_CUSPARSE( hipsparseSpGEMM_compute(cusparse, opA, opB,
                                            &alpha, descA, descB, &beta, descC,
                                            computeType, HIPSPARSE_SPGEMM_DEFAULT,
                                            spgemmDesc, &bufferSize2, dBuffer2.data()) )
        // --------------------------------------------------------------------------
        // get matrix C non-zero entries C_nnz1
        CHECK_CUSPARSE( hipsparseSpMatGetSize(descC, &matC_.nrows, &matC_.ncols, &matC_.numnonz) )
        // allocate matrix C
        matC_.resize(matC_.nrows, matC_.ncols, matC_.numnonz);
        // update matC with the new pointers
        CHECK_CUSPARSE(hipsparseCsrSetPointers(descC, matC_.indptr.data(), matC_.indices.data(), matC_.data.data()) )

        // copy the final products to the matrix C
        CHECK_CUSPARSE(
            hipsparseSpGEMM_copy(cusparse, opA, opB,
                                &alpha, descA, descB, &beta, descC,
                                computeType, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc) )
    }


    // dst = src + alpha * dst
    void axpy(Vec<float> &dst, float const &alpha, Vec<float> const &src) {
        assert(dst.size() == src.size());
        CHECK_CUBLAS(hipblasSaxpy(cublas, dst.size(), &alpha, src.data(), 1, dst.data(), 1));
    }

    void zero(Vec<float> &dst) {
        CHECK_CUDA(hipMemset(dst.data(), 0, dst.size() * sizeof(float)));
    }

    void copy(Vec<float> &dst, Vec<float> const &src) {
        dst.resize(src.size());
        CHECK_CUDA(hipMemcpy(dst.data(), src.data(), src.size() * sizeof(float), hipMemcpyDeviceToDevice));
    }

    // dst = alpha * x
    void scal2(Vec<float> &dst, float const &alpha, Vec<float> const &x) {
        copy(dst, x);
        CHECK_CUBLAS(hipblasSscal(cublas, dst.size(), &alpha, dst.data(), 1));
    }

    // dst = alpha * alpha
    void scal(Vec<float> &dst, float const &alpha) {
        CHECK_CUBLAS(hipblasSscal(cublas, dst.size(), &alpha, dst.data(), 1));
    }

    // x = A^{-1} b by cusolver cholesky
    // https://docs.nvidia.com/cuda/cusolver/index.html#cusolversp-t-csrlsvchol
    void spsolve(Vec<float> &x, CSR<float> const &A, Vec<float> &b) {
        hipsparseMatDescr_t descrA = NULL;
        CHECK_CUSPARSE(hipsparseCreateMatDescr(&descrA));
        CHECK_CUSPARSE(hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL));
        CHECK_CUSPARSE(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO)); 
        int singularity;

        CHECK_CUSOLVER( hipsolverSpScsrlsvchol(cusolverH, A.nrows, A.numnonz, descrA, A.data.data(), A.indptr.data(), A.indices.data(), b.data(), 1e-10, 0, x.data(), &singularity) );
    }

    float vdot(Vec<float> const &x, Vec<float> const &y) {
        float result;
        CHECK_CUBLAS(hipblasSdot(cublas, x.size(), x.data(), 1, y.data(), 1, &result));
        return result;
    }

    float vnorm(Vec<float> const &x) {
        float result;
        CHECK_CUBLAS(hipblasSnrm2(cublas, x.size(), x.data(), 1, &result));
        return result;
    }

    // transpose csr matrix A to AT
    // https://docs.nvidia.com/cuda/cusparse/index.html?highlight=hipsparseCsr2cscEx2#cusparsecsr2cscex2
    void transpose(CSR<float> const & A, CSR<float>& AT)
    {
        int m = A.nrows;
        int n = A.ncols;
        int nnz = A.numnonz;
        const float *csrVal  = A.data.data();
        const int *csrRowPtr = A.indptr.data();
        const int *csrColInd = A.indices.data();
        float *cscVal  = AT.data.data();
        int *cscColPtr = AT.indptr.data();
        int *cscRowInd = AT.indices.data();
        hipDataType  valType = HIP_R_32F;
        hipsparseAction_t copyValues = HIPSPARSE_ACTION_NUMERIC;
        hipsparseIndexBase_t idxBase = HIPSPARSE_INDEX_BASE_ZERO;
        hipsparseCsr2CscAlg_t    alg = HIPSPARSE_CSR2CSC_ALG_DEFAULT;
        hipsparseStatus_t status;
        size_t bufferSize = 0;
        Buffer buffer;

        CHECK_CUSPARSE( hipsparseCsr2cscEx2_bufferSize(cusparse, m, n, nnz, csrVal, csrRowPtr, csrColInd, cscVal, cscColPtr, cscRowInd, valType, copyValues, idxBase, alg, &bufferSize));
        buffer.reserve(bufferSize);
        CHECK_CUSPARSE( hipsparseCsr2cscEx2(           cusparse, m, n, nnz, csrVal, csrRowPtr, csrColInd, cscVal, cscColPtr, cscRowInd, valType, copyValues, idxBase, alg, buffer.data()));                
    }



//Calculate the largest eigenvalue of a symmetric matrix using the power method!
// https://docs.nvidia.com/cuda/cusolver/index.html#cusolversp-t-csreigvsi  (cusolverSpScsreigvsi is not used here, but it is another option, so I just keep the note. It use the shift inverse method to solve this equation Ax=lam x)
// Reference code: https://github.com/physicslog/maxEigenValueGPU/blob/25e0aa3d6c9bbeb03be6249d0ab8cfaafd32188c/maxeigenvaluepower.cu#L255
float computeMaxEigenvaluePowerMethodOptimized(CSR<float>& M, int max_iter) {
  assert(M.nrows == M.ncols);

  // Initialize two vectors x_i and x_k
  thrust::device_vector<float> x_i(M.nrows), x_k(M.nrows, 0.0f);

  // Set x_i := the random vector
    thrust::transform(thrust::make_counting_iterator<int>(0),
    thrust::make_counting_iterator<int>(M.nrows),
    x_i.begin(),
    genRandomNumber());

  // CUSPARSE APIs
  hipsparseHandle_t handle = NULL;
  hipsparseSpMatDescr_t matM;
  hipsparseDnVecDescr_t xi, xk;
  void *dBuffer = NULL;
  size_t bufferSize = 0;
  float alpha = 1.0f;
  float beta = 0.0f;

  CHECK_CUSPARSE( hipsparseCreate(&handle) )

  CHECK_CUSPARSE( hipsparseCreateCsr(&matM, M.nrows, M.ncols, M.numnonz,
                                   thrust::raw_pointer_cast(M.indptr.data()),
                                   thrust::raw_pointer_cast(M.indices.data()),
                                   thrust::raw_pointer_cast(M.data.data()),
                                   HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                   HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )

  CHECK_CUSPARSE( hipsparseCreateDnVec(&xi, M.nrows, thrust::raw_pointer_cast(x_i.data()), HIP_R_32F) )
  CHECK_CUSPARSE( hipsparseCreateDnVec(&xk, M.nrows, thrust::raw_pointer_cast(x_k.data()), HIP_R_32F) )

  CHECK_CUSPARSE( hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                          &alpha, matM, xi, &beta, xk, HIP_R_32F,
                                          HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize) )

  CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

  float max_eigenvalue(0.0f), max_eigenvalue_prev(0.0f);
  float tol = 1e-6;  // tolerance for convergence
  int itr = 0;
  // Power iteration method
  while (itr < max_iter) {
    // Compute x_k = A * x_i; generates Krylov subspace
    CHECK_CUSPARSE( hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matM, xi, &beta, xk, HIP_R_32F,
                                 HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer) )

    // Compute the L2 norm of x_k
    float norm = std::sqrt(thrust::inner_product(x_k.begin(), x_k.end(), x_k.begin(), 0.0f));

    // Normalize x_k and update x_i
    thrust::transform(x_k.begin(), x_k.end(), x_i.begin(), thrust::placeholders::_1 / norm);

    // Compute the maximum eigenvalue
    CHECK_CUSPARSE( hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                &alpha, matM, xi, &beta, xk, HIP_R_32F,
                                HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer) )

    max_eigenvalue = thrust::inner_product(x_i.begin(), x_i.end(), x_k.begin(), 0.0f);

    if (std::abs(max_eigenvalue - max_eigenvalue_prev) < tol) {
      std::cout << FGRN("[NOTE]: ") << "Converged at iterations: " << itr << std::endl;
      return max_eigenvalue;
    }

    max_eigenvalue_prev = max_eigenvalue;
    itr++;
  }

  // Destroy the handle and descriptors
  CHECK_CUSPARSE( hipsparseDestroySpMat(matM) )
  CHECK_CUSPARSE( hipsparseDestroyDnVec(xi) )
  CHECK_CUSPARSE( hipsparseDestroyDnVec(xk) )
  CHECK_CUSPARSE( hipsparseDestroy(handle) )
  CHECK_CUDA( hipFree(dBuffer) )

  std::cout << FRD("[NOTE]: ") << "Maximum number of iterations reached." << std::endl;  // no convergence
  return max_eigenvalue;
}
};

struct MGLevel {
    CSR<float> A;
    CSR<float> R;
    CSR<float> P;
    Vec<float> residual;
    Vec<float> b;
    Vec<float> x;
    Vec<float> h;
    Vec<float> outh;
};


using thrust::device_vector;

__global__ void fill_A_CSR_kernel(float *a, int size) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        a[i] = 100.0;
    }
}


struct FastFill : Kernels {
    CSR<float> A;
    float alpha;
    int NE;
    int NV;
    std::vector<std::array<int,2>> edges;
    std::vector<float> inv_mass;
    std::vector<std::array<float,3>> pos;
    std::vector<std::vector<int>> adjacent_edges;
    std::vector<int> num_adjacent_edge;
    std::vector<std::vector<int>> adjacent_edge_abc;
    std::vector<int> ii, jj;
    std::vector<int> indptr;
    std::vector<int> indices;
    std::vector<float> data;
    int num_nonz;
    int nrows, ncols;
    Vec<float> d_inv_mass;

    void fetch_A(float *data_in, int *indices_in, int *indptr_in) {
        std::copy(data.begin(), data.end(), data_in);
        std::copy(indices.begin(), indices.end(), indices_in);
        std::copy(indptr.begin(), indptr.end(), indptr_in);
    }

    void set_data(int* edges_in, int NE_in, float* inv_mass_in, int NV_in, float* pos_in, float alpha_in)
    {
        NE = NE_in;
        NV = NV_in;
        nrows = NE;
        ncols = NE;

        edges.resize(NE);
        for(int i=0; i<NE; i++)
        {
            edges[i][0] = edges_in[i*2];
            edges[i][1] = edges_in[i*2+1];
        }

        inv_mass.resize(NV_in);
        for(int i=0; i<NV_in; i++)
        {
            inv_mass[i] = inv_mass_in[i];
        }

        pos.resize(NV);
        for(int i=0; i<NV; i++)
        {
            pos[i][0] = pos_in[i*3];
            pos[i][1] = pos_in[i*3+1];
            pos[i][2] = pos_in[i*3+2];
        }

        alpha = alpha_in;
    }

    void update_pos(float* pos_in)
    {
        for(int i=0; i<NV; i++)
        {
            pos[i][0] = pos_in[i*3];
            pos[i][1] = pos_in[i*3+1];
            pos[i][2] = pos_in[i*3+2];
        }
    }

    void host_to_device()
    {
        d_inv_mass.assign(inv_mass.data(), inv_mass.size());
        cout<<"copy data to device"<<endl;
    }

    void device_to_host()
    {
        d_inv_mass.tohost(inv_mass);
    }


    // init_direct_fill_A
    int init()
    {
        init_adj_edge(edges);
        init_adjacent_edge_abc();
        calc_num_nonz();
        init_A_CSR_pattern();
        csr_index_to_coo_index();

        // transfer data to device
        host_to_device();

        return num_nonz;
    }

    void run(float* pos_in)
    {
        Timer t;
        t.start();
        update_pos(pos_in);
        t.end("update_pos");
        t.start();
        fill_A_CSR_gpu();
        t.end("fill_A_CSR");
    }


    std::array<float,3> inline normalize(std::array<float,3> v)
    {
        float norm = sqrt(v[0]*v[0] + v[1]*v[1] + v[2]*v[2]);
        return {v[0]/norm, v[1]/norm, v[2]/norm};
    }

    std::array<float,3> inline normalize_diff(std::array<float,3> &v1,  std::array<float,3> &v2)
    {
        std::array<float,3> diff = {v1[0]-v2[0], v1[1]-v2[1], v1[2]-v2[2]};
        float norm = sqrt(diff[0]*diff[0] + diff[1]*diff[1] + diff[2]*diff[2]);
        return {diff[0]/norm, diff[1]/norm, diff[2]/norm};
    }

    float inline dot(std::array<float,3> a, std::array<float,3> b)
    {
        return a[0]*b[0] + a[1]*b[1] + a[2]*b[2];
    }

    void launch_check()
    {
        hipError_t varCudaError1 = hipGetLastError();
        if (varCudaError1 != hipSuccess)
        {
            std::cout << "Failed to launch kernel (error code: " << hipGetErrorString(varCudaError1) << ")!" << std::endl;
            exit(EXIT_FAILURE);
        }
    }

    void fill_A_CSR_gpu()
    {
        TODO: finish fill A CSR
        fill_A_CSR_kernel<<<NV/128+1, 128>>>(d_inv_mass.data(), NV);
        hipDeviceSynchronize();
        launch_check();

        cout<<"111"<<endl;
        device_to_host();
        cout<<"inv_mass[0]: "<<inv_mass[0]<<endl;
        exit(0);
    }


    void fill_A_CSR()
    {
        for(int cnt=0; cnt<num_nonz; cnt++)
        {
            int i = ii[cnt]; // row index
            int j = jj[cnt]; // col index
            int k = cnt - indptr[i]; //k-th non-zero element of i-th row. 
            // Because the diag is the final element of each row, 
            // it is also the k-th adjacent edge of i-th edge.
            if (i == j) // diag
            {
                data[cnt] = inv_mass[edges[i][0]] + inv_mass[edges[i][1]] + alpha;
                continue;
            }
            int a = adjacent_edge_abc[i][k*3];
            int b = adjacent_edge_abc[i][k*3+1];
            int c = adjacent_edge_abc[i][k*3+2];
            auto g_ab = normalize_diff(pos[a], pos[b]);
            auto g_ac = normalize_diff(pos[a], pos[c]);
            auto offdiag = inv_mass[a] * dot(g_ab, g_ac);
            data[cnt] = offdiag;
        }
    }


    void init_A_CSR_pattern()
    {
        indptr.resize(NE+1);
        indices.resize(num_nonz);
        data.resize(num_nonz);

        indptr[0] = 0;
        for(int i=0; i<NE; i++)
        {
            int num_adj_i = num_adjacent_edge[i];
            indptr[i+1] = indptr[i] + num_adj_i + 1;
            for(int j=0; j<num_adj_i; j++)
            {
                indices[indptr[i]+j] = adjacent_edges[i][j];
            }
            indices[indptr[i+1]-1] = i;
        }
    }


    void csr_index_to_coo_index()
    {
        ii.resize(num_nonz);
        jj.resize(num_nonz);
        for(int i=0; i<NE; i++)
        {
            for(int j=indptr[i]; j<indptr[i+1]; j++)
            {
                ii[j] = i;
                jj[j] = indices[j];
            }
        }
    }


    void init_adj_edge(std::vector<std::array<int,2>> &edges)
    {
        std::unordered_map<int, std::set<int>> vertex_to_edges;
        for(int edge_index=0; edge_index<edges.size(); edge_index++)
        {
            int v1 = edges[edge_index][0];
            int v2 = edges[edge_index][1];
            if (vertex_to_edges.find(v1) == vertex_to_edges.end())
                vertex_to_edges[v1] = std::set<int>();
            if (vertex_to_edges.find(v2) == vertex_to_edges.end())
                vertex_to_edges[v2] = std::set<int>();
            vertex_to_edges[v1].insert(edge_index);
            vertex_to_edges[v2].insert(edge_index);
        }

        adjacent_edges.resize(edges.size());
        for(int edge_index=0; edge_index<edges.size(); edge_index++)
        {
            int v1 = edges[edge_index][0];
            int v2 = edges[edge_index][1];
            std::set<int> adj; //adjacent edges of one edge
            std::set_union(vertex_to_edges[v1].begin(), vertex_to_edges[v1].end(), vertex_to_edges[v2].begin(), vertex_to_edges[v2].end(), std::inserter(adj, adj.begin()));
            adj.erase(edge_index);
            adjacent_edges[edge_index] = std::vector<int>(adj.begin(), adj.end());
        }

        //calc num_adjacent_edge
        for(auto adj:adjacent_edges)
        {
            num_adjacent_edge.push_back(adj.size());
        }

        NE = edges.size();

        adjacent_edge_abc.resize(NE);
        for(int i=0; i<NE; i++)
        {
            adjacent_edge_abc[i].resize(num_adjacent_edge[i]*3);
        }
    }

    void calc_num_nonz()
    {
        num_nonz = 0;
        for(auto num_adj:num_adjacent_edge)
        {
            num_nonz += num_adj;
        }
        num_nonz += num_adjacent_edge.size();

        A.numnonz = num_nonz;
    }


    void init_adjacent_edge_abc()
    {
        for(int i=0; i<edges.size(); i++)
        {
            auto ii0 = edges[i][0];
            auto ii1 = edges[i][1];

            auto num_adj = num_adjacent_edge[i];
            for(int j=0; j<num_adj; j++)
            {
                auto ia = adjacent_edges[i][j];
                if (ia == i)
                    continue;
                auto jj0 = edges[ia][0];
                auto jj1 = edges[ia][1];
                auto a = -1;
                auto b = -1;
                auto c = -1;
                if (ii0 == jj0)
                {
                    a = ii0;
                    b = ii1;
                    c = jj1;
                }
                else if (ii0 == jj1)
                {
                    a = ii0;
                    b = ii1;
                    c = jj0;
                }
                else if (ii1 == jj0)
                {
                    a = ii1;
                    b = ii0;
                    c = jj1;
                }
                else if (ii1 == jj1)
                {
                    a = ii1;
                    b = ii0;
                    c = jj0;
                }
                adjacent_edge_abc[i][j*3] = a;
                adjacent_edge_abc[i][j*3+1] = b;
                adjacent_edge_abc[i][j*3+2] = c;
            }
        }
    }

}; //FastFill struct


struct VCycle : Kernels {
    std::vector<MGLevel> levels;
    size_t nlvs;
    std::vector<float> chebyshev_coeff;
    size_t smoother_type = 1; //1:chebyshev, 2:jacobi, 3:gauss_seidel
    float jacobi_omega;
    size_t jacobi_niter;
    Vec<float> init_x;
    Vec<float> init_b;
    Vec<float> outer_x;
    Vec<float> final_x;
    Vec<float> outer_b;
    float save_rho_prev;
    Vec<float> save_p;
    Vec<float> save_q;
    Buffer buff;
    float rtol;
    size_t maxiter;
    std::vector<float> residuals;
    size_t niter; //final number of iterations to break the loop
    float max_eig;


    void setup_smoothers_cuda(int type) {
        if(smoother_type == 1)
        {
            setup_chebyshev_cuda(levels[0].A);
        }
        else if (smoother_type == 2)
        {
            //TODO:setup jacobi
        }
    }


    void setup_chebyshev_cuda(CSR<float> &A) {
        float lower_bound=1.0/30.0;
        float upper_bound=1.1;
        float rho = computeMaxEigenvaluePowerMethodOptimized(A, 100);
        float a = rho * lower_bound;
        float b = rho * upper_bound;
        chebyshev_polynomial_coefficients(a, b);
        
        max_eig = rho;
        cout<<"max eigenvalue: "<<max_eig<<endl;
    }


    void chebyshev_polynomial_coefficients(float a, float b)
    {
        int degree=3;
        const float PI = 3.14159265358979323846;

        if(a >= b || a <= 0)
            assert(false && "Invalid input for Chebyshev polynomial coefficients");

        // Chebyshev roots for the interval [-1,1]
        std::vector<float> std_roots(degree);
        for(int i=0; i<degree; i++)
        {
            std_roots[i] = std::cos(PI * (i + 0.5) / (float)degree);
        }

        // Chebyshev roots for the interval [a,b]
        std::vector<float> scaled_roots(degree);
        for(int i=0; i<degree; i++)
        {
            scaled_roots[i] = 0.5 * (b-a) * (1 + std_roots[i]) + a;
        }

        // Compute monic polynomial coefficients of polynomial with scaled roots
        std::vector<float> scaled_poly(4);
        // np.poly for 3 roots. This will calc the coefficients of the polynomial from roots.
        // i.e., (x - root1) * (x - root2) * (x - root3) = x^3 - (root1 + root2 + root3)x^2 + (root1*root2 + root2*root3 + root3*root1)x - root1*root2*root3
        scaled_poly[0] = 1.0;
        scaled_poly[1] = -(scaled_roots[0] + scaled_roots[1] + scaled_roots[2]);
        scaled_poly[2] = scaled_roots[0]*scaled_roots[1] + scaled_roots[1]*scaled_roots[2] + scaled_roots[2]*scaled_roots[0];
        scaled_poly[3] = -scaled_roots[0]*scaled_roots[1]*scaled_roots[2];

        // Scale coefficients to enforce C(0) = 1.0
        float c0 = scaled_poly[3];
        for(int i=0; i<degree; i++)
        {
            scaled_poly[i] /= c0; 
        }


        chebyshev_coeff.resize(degree);
        //CAUTION:setup_chebyshev has "-" at the end
        for(int i=0; i<degree; i++)
        {
            chebyshev_coeff[i] = -scaled_poly[i];
        }

        cout<<"Chebyshev polynomial coefficients: ";
        for(int i=0; i<degree; i++)
        {
            cout<<chebyshev_coeff[i]<<" ";
        }
    }


    float calc_rnorm(Vec<float> const &b, Vec<float> const &x, CSR<float> const &A) {
        float rnorm = 0.0;
        Vec<float> r;
        r.resize(b.size());
        copy(r, b);
        spmv(r, -1, A, x, 1, buff);
        rnorm = vnorm(r);
        return rnorm;
    }


    void setup(size_t numlvs) {
        if (levels.size() < numlvs) {
            levels.resize(numlvs);
        }
        nlvs = numlvs;
        chebyshev_coeff.clear();
        jacobi_omega = 0.0;
    }

    // DEPRECATED
    // void set_lv_csrmat(size_t lv, size_t which, float const *datap, size_t ndat, int const *indicesp, size_t nind, int const *indptrp, size_t nptr, size_t rows, size_t cols, size_t nnz) {
    //     CSR<float> *mat = nullptr;
    //     if (which == 1) mat = &levels.at(lv).A;
    //     if (which == 2) mat = &levels.at(lv).R;
    //     if (which == 3) mat = &levels.at(lv).P;
    //     if (mat) {
    //         mat->assign(datap, ndat, indicesp, nind, indptrp, nptr, rows, cols, nnz);
    //     }
    // }

    void set_P(size_t lv, float const *datap, size_t ndat, int const *indicesp, size_t nind, int const *indptrp, size_t nptr, size_t rows, size_t cols, size_t nnz) {
        levels.at(lv).P.assign(datap, ndat, indicesp, nind, indptrp, nptr, rows, cols, nnz);
    }

    void set_A0(float const *datap, size_t ndat, int const *indicesp, size_t nind, int const *indptrp, size_t nptr, size_t rows, size_t cols, size_t nnz) {
        levels.at(0).A.assign(datap, ndat, indicesp, nind, indptrp, nptr, rows, cols, nnz);
    }


    // only update the data of A0
    void update_A0(float const *datap) {
        CHECK_CUDA(hipMemcpy(levels.at(0).A.data.data(), datap, levels.at(0).A.data.size() * sizeof(float), hipMemcpyHostToDevice));
    }


    void set_A0_from_fastFill(FastFill *ff) {
        levels.at(0).A.assign(ff->data.data(), ff->data.size(), ff->indices.data(), ff->indices.size(), ff->indptr.data(), ff->indptr.size(), ff->nrows, ff->ncols, ff->num_nonz);
    }

    // DEPRECATED
    // void setup_chebyshev(float const *coeff, size_t ncoeffs) {
    //     smoother_type = 1;
    //     chebyshev_coeff.assign(coeff, coeff + ncoeffs);
    // }

    void chebyshev(int lv, Vec<float> &x, Vec<float> const &b) {
        copy(levels.at(lv).residual, b);
        spmv(levels.at(lv).residual, -1, levels.at(lv).A, x, 1, buff); // residual = b - A@x
        scal2(levels.at(lv).h, chebyshev_coeff.at(0), levels.at(lv).residual); // h = c0 * residual


        for (int i = 1; i < chebyshev_coeff.size(); ++i) {
            // h' = ci * residual + A@h
            copy(levels.at(lv).outh, levels.at(lv).residual);
            spmv(levels.at(lv).outh, 1, levels.at(lv).A, levels.at(lv).h, chebyshev_coeff.at(i), buff);

            // copy(levels.at(lv).h, levels.at(lv).outh);
            levels.at(lv).h.swap(levels.at(lv).outh);
        }

        axpy(x, 1, levels.at(lv).h); // x += h
    }

    void setup_jacobi(float const omega, size_t const n) {
        smoother_type = 2;
        jacobi_omega = omega;
        jacobi_niter = n;
    }

    void jacobi(int lv, Vec<float> &x, Vec<float> const &b) {
        Vec<float> x_old;
        x_old.resize(x.size());
        copy(x_old, x);
        for (int i = 0; i < jacobi_niter; ++i) {
            weighted_jacobi_kernel<<<(levels.at(lv).A.nrows + 255) / 256, 256>>>(x.data(), x_old.data(), b.data(), levels.at(lv).A.data.data(), levels.at(lv).A.indices.data(), levels.at(lv).A.indptr.data(), levels.at(lv).A.nrows, jacobi_omega);
            x.swap(x_old);
        }
    }

    void jacobi_cpu(int lv, Vec<float> &x, Vec<float> const &b) {
        // serial jacobi
        std::vector<float> x_host(x.size());
        std::vector<float> b_host(b.size());
        x.tohost(x_host);
        b.tohost(b_host);
        std::vector<float> data_host;
        std::vector<int> indices_host, indptr_host;
        levels.at(lv).A.tohost(data_host, indices_host, indptr_host);
        // cout<<"omega: "<<jacobi_omega<<endl;
        jacobi_serial(
            indptr_host.data(), indptr_host.size(),
            indices_host.data(), indices_host.size(),
            data_host.data(), data_host.size(),
            x_host.data(), x_host.size(),
            b_host.data(), b_host.size(),
            x_host.data(), x_host.size(),
            0, levels.at(lv).A.nrows, 1, jacobi_omega);
        x.assign(x_host.data(), x_host.size());
        // auto r = calc_rnorm(b, x, levels.at(lv).A);
        // cout<<"lv"<<lv<<"   rnorm: "<<r<<endl;
    }


    void setup_gauss_seidel() {
        smoother_type = 3;
    }

    void gauss_seidel_cpu(int lv, Vec<float> &x, Vec<float> const &b) {
        // serial gauss seidel
        std::vector<float> x_host(x.size());
        std::vector<float> b_host(b.size());
        x.tohost(x_host);
        b.tohost(b_host);
        std::vector<float> data_host;
        std::vector<int> indices_host, indptr_host;
        levels.at(lv).A.tohost(data_host, indices_host, indptr_host);
        gauss_seidel_serial(
            indptr_host.data(), indptr_host.size(),
            indices_host.data(), indices_host.size(),
            data_host.data(), data_host.size(),
            x_host.data(), x_host.size(),
            b_host.data(), b_host.size(),
            0, levels.at(lv).A.nrows, 1);
        x.assign(x_host.data(), x_host.size());
        // auto r = calc_rnorm(b, x, levels.at(lv).A);
        // cout<<"lv"<<lv<<"   rnorm: "<<r<<endl;
    }

    void _smooth(int lv, Vec<float> &x, Vec<float> const &b) {
        if(smoother_type == 1)
            chebyshev(lv, x, b);
        else if (smoother_type == 2)
        {
            jacobi(lv, x, b);
        }
        else if (smoother_type == 3)
        {
            gauss_seidel_cpu(lv, x, b);
        }
    }

    GpuTimer ttt1, ttt2, ttt3, ttt;
    std::vector<float> ttt1_elapsed, ttt2_elapsed, ttt3_elapsed;
    std::vector<std::vector<float>> ttt_elapsed;


    void calc_residual(int lv, Vec<float> &x, Vec<float> const &b) {
        copy(levels.at(lv).residual, b);
        spmv(levels.at(lv).residual, -1, levels.at(lv).A, x, 1, buff); // residual = b - A@x
    }


    void vcycle_down() {
        ttt_elapsed.resize(nlvs-1);
        for (int lv = 0; lv < nlvs-1; ++lv) {
            ttt.start();

            ttt1.start();
            Vec<float> &x = lv != 0 ? levels.at(lv - 1).x : init_x;
            Vec<float> &b = lv != 0 ? levels.at(lv - 1).b : init_b;
            ttt1.stop();
            ttt1_elapsed.push_back(ttt1.elapsed());

            ttt2.start();
            _smooth(lv, x, b);
            ttt2.stop();
            ttt2_elapsed.push_back(ttt2.elapsed());

            ttt3.start();
            copy(levels.at(lv).residual, b);
            spmv(levels.at(lv).residual, -1, levels.at(lv).A, x, 1, buff); // residual = b - A@x

            levels.at(lv).b.resize(levels.at(lv).R.nrows);
            spmv(levels.at(lv).b, 1, levels.at(lv).R, levels.at(lv).residual, 0, buff); // coarse_b = R@residual

            levels.at(lv).x.resize(levels.at(lv).b.size());
            zero(levels.at(lv).x);
            ttt3.stop();
            ttt3_elapsed.push_back(ttt3.elapsed());

            ttt.stop();
            ttt_elapsed[lv].push_back(ttt.elapsed());
        }
    }

    void vcycle_up() {
        for (int lv = nlvs-2; lv >= 0; --lv) {
            Vec<float> &x = lv != 0 ? levels.at(lv - 1).x : init_x;
            Vec<float> &b = lv != 0 ? levels.at(lv - 1).b : init_b;
            spmv(x, 1, levels.at(lv).P, levels.at(lv).x, 1, buff); // x += P@coarse_x
            _smooth(lv, x, b);
        }
    }

    GpuTimer tt1, tt2, tt3;
    std::vector<float> tt1_elapsed, tt2_elapsed, tt3_elapsed;

    void vcycle() {
        
        tt1.start();
        vcycle_down();
        tt1.stop();
        tt1_elapsed.push_back(tt1.elapsed());

        tt2.start();
        coarse_solve();
        tt2.stop();
        tt2_elapsed.push_back(tt2.elapsed());


        tt3.start();
        vcycle_up();
        tt3.stop();
        tt3_elapsed.push_back(tt3.elapsed());
    }


    void coarse_solve() {
        auto const &A = levels.at(nlvs - 1).A;
        auto &x = levels.at(nlvs - 2).x;
        auto &b = levels.at(nlvs - 2).b;
        spsolve(x, A, b);
    }

    void copy_outer2init_x() {
        copy(init_x, outer_x);
    }

    void set_outer_x(float const *x, size_t n) {
        outer_x.resize(n);
        CHECK_CUDA(hipMemcpy(outer_x.data(), x, n * sizeof(float), hipMemcpyHostToDevice));
        copy(final_x, outer_x);
    }

    void set_outer_b(float const *b, size_t n) {
        outer_b.resize(n);
        CHECK_CUDA(hipMemcpy(outer_b.data(), b, n * sizeof(float), hipMemcpyHostToDevice));
    }

    float init_cg_iter0(float *residuals) {
        float bnrm2 = vnorm(outer_b);
        // r = b - A@(x)
        copy(init_b, outer_b);
        spmv(outer_b, -1, levels.at(0).A, outer_x, 1, buff);
        float normr = vnorm(init_b);
        residuals[0] = normr;
        return bnrm2;
    }

    void do_cg_itern(float *residuals, size_t iteration) {
        float rho_cur = vdot(init_b, init_x);
        if (iteration > 0) {
            float beta = rho_cur / save_rho_prev;
            // p *= beta
            // p += z
            scal(save_p, beta);
            axpy(save_p, 1, init_x);
        } else {
            // p = move(z)
            save_p.swap(init_x);
        }
        // q = A@(p)
        save_q.resize(levels.at(0).A.nrows);
        spmv(save_q, 1, levels.at(0).A, save_p, 0, buff);
        save_rho_prev = rho_cur;
        float alpha = rho_cur / vdot(save_p, save_q);
        // x += alpha*p
        axpy(final_x, alpha, save_p);
        // r -= alpha*q
        axpy(init_b, -alpha, save_q);
        float normr = vnorm(init_b);
        residuals[iteration + 1] = normr;
    }

    void fetch_cg_final_x(float *x) {
        CHECK_CUDA(hipMemcpy(x, final_x.data(), final_x.size() * sizeof(float), hipMemcpyDeviceToHost));
    }

    void fetch_cg_final_r(float *r) {
        // CHECK_CUDA(hipMemcpy(r, residuals.data(), residuals.size() * sizeof(float), hipMemcpyDeviceToHost));
        std::copy(residuals.begin(), residuals.end(), r);
    }

    void compute_RAP(size_t lv) {
            CSR<float> &A = levels.at(lv).A;
            CSR<float> &R = levels.at(lv).R;
            CSR<float> &P = levels.at(lv).P;
            CSR<float> AP;
            CSR<float> &RAP = levels.at(lv+1).A;
            R.resize(P.ncols, P.nrows, P.numnonz);
            transpose(P, R);            
            spgemm(A, P, AP) ;
            spgemm(R, AP, RAP);
    }

    void fetch_A(size_t lv, float *data, int *indices, int *indptr) {
        CSR<float> &A = levels.at(lv).A;
        CHECK_CUDA(hipMemcpy(data, A.data.data(), A.data.size() * sizeof(float), hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(indices, A.indices.data(), A.indices.size() * sizeof(int), hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(indptr, A.indptr.data(), A.indptr.size() * sizeof(int), hipMemcpyDeviceToHost));
    }
    
    void set_data(const float* x, size_t nx, const float* b, size_t nb, float rtol_, size_t maxiter_)
    {
        set_outer_x(x, nx);
        set_outer_b(b, nb);
        rtol = rtol_;
        maxiter = maxiter_;
        residuals.resize(maxiter+1);
    }

    float get_max_eig()
    {
        Timer t("eigenvalue");
        t.start();
        max_eig = computeMaxEigenvaluePowerMethodOptimized(levels.at(0).A, 100);
        t.end();
        cout<<"max eigenvalue: "<<max_eig<<endl;
        return  max_eig;
    }

    size_t get_data(float* x_, float* r_)
    {
        fetch_cg_final_x(x_);
        fetch_cg_final_r(r_);
        return niter;
    }

    float sum(std::vector<float> &v)
    {
        return std::accumulate(v.begin(), v.end(), 0.0);
    }

    float avg(std::vector<float> &v)
    {
        return std::accumulate(v.begin(), v.end(), 0.0) / v.size();
    }

    void solve()
    {
        GpuTimer t1, t2, t3, t4, t5;
        std::vector<float> t2_elapsed, t3_elapsed, t4_elapsed, t5_elapsed;

        t1.start();
        float bnrm2 = init_cg_iter0(residuals.data());
        float atol = bnrm2 * rtol;
        for (size_t iter=0; iter<maxiter; iter++)
        {   
            t2.start();
            t3.start();

            if (residuals[iter] < atol)
            {
                niter = iter;
                break;
            }
            copy_outer2init_x();  //reset x to x0
            t3.stop();
            t3_elapsed.push_back(t3.elapsed());

            t4.start();
            vcycle();
            t4.stop();
            t4_elapsed.push_back(t4.elapsed());

            t5.start();
            do_cg_itern(residuals.data(), iter); //first r is r[0], then r[iter+1]
            t5.stop();
            t5_elapsed.push_back(t5.elapsed());

            niter = iter;
            t2.stop();
            t2_elapsed.push_back(t2.elapsed());
        }

        bool report_time = false;
        if(report_time)
        {
            float avg_t2 = avg(t2_elapsed);
            float avg_t3 = avg(t3_elapsed);
            float avg_t4 = avg(t4_elapsed);
            float avg_t5 = avg(t5_elapsed);
            float avg_tt1 = avg(tt1_elapsed);
            float avg_tt2 = avg(tt2_elapsed);
            float avg_tt3 = avg(tt3_elapsed);
            float avg_ttt1 = avg(ttt1_elapsed);
            float avg_ttt2 = avg(ttt2_elapsed);
            float avg_ttt3 = avg(ttt3_elapsed);
            

            cout<<"     avg time one iteration: "<<avg_t2<<" ms"<<endl;
            cout<<"     avg time before vcycle: "<<avg_t3<<" ms"<<endl;
            cout<<"     avg time vcycle: "<<avg_t4<<" ms"<<endl;
            cout<<"     avg time after vcycle: "<<avg_t5<<" ms"<<endl;

            cout<<"     avg time vcycle_down: "<<avg_tt1<<" ms"<<endl;
            cout<<"     avg time coarse_solve: "<<avg_tt2<<" ms"<<endl;
            cout<<"     avg time vcycle_up: "<<avg_tt3<<" ms"<<endl;

            cout<<"     avg time vcycle_down before smooth: "<<avg_ttt1<<" ms"<<endl;
            cout<<"     avg time vcycle_down smooth: "<<avg_ttt2<<" ms"<<endl;
            cout<<"     avg time vcycle_down after smooth: "<<avg_ttt3<<" ms"<<endl;

            // print ttt elaspse
            for(int lv=0; lv<nlvs-1; lv++)
            {
                cout<<"     level "<<lv;
                cout<<" avg ttt time: "<< avg(ttt_elapsed[lv])<<" ms"<<endl;
            }

            t1.stop();
            cout<<"     time of solve: "<<t1.elapsed()<<" ms"<<endl;
        }
    }


};

// struct AssembleMatrix : Kernels {
//     CSR<float> A;
//     CSR<float> G;
//     CSR<float> M;
//     CSR<float> ALPHA;
//     float alpha;
//     int NE;

//     void fetch_A(float *data, int *indices, int *indptr) {
//         CHECK_CUDA(hipMemcpy(data, A.data.data(), A.data.size() * sizeof(float), hipMemcpyDeviceToHost));
//         CHECK_CUDA(hipMemcpy(indices, A.indices.data(), A.indices.size() * sizeof(int), hipMemcpyDeviceToHost));
//         CHECK_CUDA(hipMemcpy(indptr, A.indptr.data(), A.indptr.size() * sizeof(int), hipMemcpyDeviceToHost));
//     }

//     void set_G(float const *datap, int const *indicesp, int const *indptrp, int rows, int cols, int nnz) {
//         G.assign(datap, nnz, indicesp, nnz, indptrp, rows + 1, rows, cols, nnz);
//     }

//     void set_M(float const *datap, int const *indicesp, int const *indptrp, int rows, int cols, int nnz) {
//         M.assign(datap, nnz, indicesp, nnz, indptrp, rows + 1, rows, cols, nnz);
//     }

//     void set_ALPHA(float const *datap, int const *indicesp, int const *indptrp, int rows, int cols, int nnz) {
//         ALPHA.assign(datap, nnz, indicesp, nnz, indptrp, rows + 1, rows, cols, nnz);
//     }

//     void compute_GMG() {
//         CSR<float> GM;
//         spgemm(G, M, GM);
//         CSR<float> GT;
//         GT.resize(G.ncols, G.nrows, G.numnonz);
//         transpose(G, GT);
//         spgemm(GM, GT, A);
//     }

// };




} // namespace

static VCycle *fastmg = nullptr;
// static AssembleMatrix *fastA = nullptr;
static FastFill *fastFill = nullptr;

#if _WIN32
#define DLLEXPORT __declspec(dllexport)
#else
#define DLLEXPORT
#endif

extern "C" DLLEXPORT void fastmg_new(size_t numlvs) {
    if (!fastmg)
        fastmg = new VCycle{};
}

extern "C" DLLEXPORT void fastmg_setup_nl(size_t numlvs) {
    fastmg->setup(numlvs);
}

// extern "C" DLLEXPORT void fastmg_setup_chebyshev(float const *coeff, size_t ncoeffs) {
//     fastmg->setup_chebyshev(coeff, ncoeffs);
// }

extern "C" DLLEXPORT void fastmg_setup_jacobi(float const omega, size_t const niter_jacobi) {
    fastmg->setup_jacobi(omega, niter_jacobi);
}

extern "C" DLLEXPORT void fastmg_setup_gauss_seidel() {
    fastmg->setup_gauss_seidel();
}

// extern "C" DLLEXPORT void fastmg_set_lv_csrmat(size_t lv, size_t which, float const *datap, size_t ndat, int const *indicesp, size_t nind, int const *indptrp, size_t nptr, size_t rows, size_t cols, size_t nnz) {
//     fastmg->set_lv_csrmat(lv, which, datap, ndat, indicesp, nind, indptrp, nptr, rows, cols, nnz);
// }

extern "C" DLLEXPORT void fastmg_RAP(size_t lv) {
    fastmg->compute_RAP(lv);
}

extern "C" DLLEXPORT void fastmg_fetch_A(size_t lv, float* data, int* indices, int* indptr) {
    fastmg->fetch_A(lv, data, indices, indptr);
}

// extern "C" DLLEXPORT void fastmg_vcycle() {
//     fastmg->vcycle();
// }

extern "C" DLLEXPORT void fastmg_solve() {
    fastmg->solve();
}

extern "C" DLLEXPORT void fastmg_set_data(const float* x, size_t nx, const float* b, size_t nb, float rtol, size_t maxiter) {
    fastmg->set_data(x, nx, b, nb, rtol, maxiter);
}

extern "C" DLLEXPORT size_t fastmg_get_data(float *x, float *r) {
    size_t niter = fastmg->get_data(x, r);
    return niter;
}

extern "C" DLLEXPORT void fastmg_set_A0(float* data, int* indices, int* indptr, int rows, int cols, int nnz)
{
                // data, ndat, indicesp, nind, indptrp, nptr, rows, cols, nnz
    fastmg->set_A0(data, nnz, indices, nnz, indptr, rows + 1, rows, cols, nnz);
}

// only update the data of A0
extern "C" DLLEXPORT void fastmg_update_A0(const float* data_in)
{
    fastmg->update_A0(data_in);
}

extern "C" DLLEXPORT void fastmg_set_P(int lv, float* data, int* indices, int* indptr, int rows, int cols, int nnz)
{
                //lv, data, ndat, indicesp, nind, indptrp, nptr, rows, cols, nnz
    fastmg->set_P(lv, data, nnz, indices, nnz, indptr, rows + 1, rows, cols, nnz);
}

extern "C" DLLEXPORT float fastmg_get_max_eig() {
    return fastmg->get_max_eig();
}

// extern "C" DLLEXPORT void fastmg_cheby_poly(float a, float b) {
//     fastmg->chebyshev_polynomial_coefficients(a, b);
// }

extern "C" DLLEXPORT void fastmg_setup_smoothers(int type) {
    fastmg->setup_smoothers_cuda(type);
}


extern "C" DLLEXPORT void fastmg_set_A0_from_fastFill() {
    fastmg->set_A0_from_fastFill(fastFill);
}

// // ------------------------------------------------------------------------------
// extern "C" DLLEXPORT void fastA_new() {
//     if (!fastA)
//         fastA = new AssembleMatrix{};
// }

// extern "C" DLLEXPORT void fastA_set_G(float* data, int* indices, int* indptr, int rows, int cols, int nnz)
// {
//     fastA->set_G(data, indices, indptr, rows, cols, nnz);
// }

// extern "C" DLLEXPORT void fastA_set_M(float* data, int* indices, int* indptr, int rows, int cols, int nnz)
// {
//     fastA->set_M(data, indices, indptr, rows, cols, nnz);
// }

// extern "C" DLLEXPORT void fastA_set_ALPHA(float* data, int* indices, int* indptr, int rows, int cols, int nnz)
// {
//     fastA->set_ALPHA(data, indices, indptr, rows, cols, nnz);
// }

// extern "C" DLLEXPORT void fastA_compute_GMG() {
//     fastA->compute_GMG();
// }

// extern "C" DLLEXPORT void fastA_fetch_A(float* data, int* indices, int* indptr) {
//     fastA->fetch_A(data, indices, indptr);
// }

// ------------------------------------------------------------------------------
extern "C" DLLEXPORT void fastFill_new() {
    if (!fastFill)
        fastFill = new FastFill{};
}

extern "C" DLLEXPORT void fastFill_set_data(int* edges_in, int NE_in, float* inv_mass_in, int NV_in, float* pos_in, float alpha_in)
{
    fastFill->set_data(edges_in, NE_in, inv_mass_in, NV_in, pos_in, alpha_in);
}

// init_direct_fill_A
extern "C" DLLEXPORT int fastFill_init() {
    int nnz = fastFill->init();
    return nnz;
}


extern "C" DLLEXPORT void fastFill_run(float* pos_in) {
    fastFill->run(pos_in);
}

extern "C" DLLEXPORT void fastFill_fetch_A(float* data, int* indices, int* indptr) {
    fastFill->fetch_A(data, indices, indptr);
}