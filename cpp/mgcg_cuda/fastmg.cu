#include "hip/hip_runtime.h"
#include <algorithm>
#include <cassert>
#include <cstring>
#include <fstream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsparse.h>
#include <hipsolver.h>
#include <iostream>
#include <string>
#include <sstream>
#include <cstdio>
#include <cmath>
#include <chrono>
#include <array>
#include <unordered_set>
#include <unordered_map>
#include <map>
#include <set>
#include <numeric>

#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/inner_product.h>
#include <thrust/random.h>

#include "kernels.cuh"
#include "utils.cuh"

using std::cout;
using std::endl;

#define USE_LESSMEM 1


namespace {

struct Buffer {
    void *m_data;
    size_t m_cap;

    Buffer() noexcept : m_data(nullptr), m_cap(0) {
    }

    Buffer(Buffer &&that) noexcept : m_data(that.m_data), m_cap(that.m_cap) {
        that.m_data = nullptr;
        that.m_cap = 0;
    }

    Buffer &operator=(Buffer &&that) noexcept {
        if (this == &that) return *this;
        if (m_data)
            CHECK_CUDA(hipFree(m_data));
        m_data = nullptr;
        m_data = that.m_data;
        m_cap = that.m_cap;
        that.m_data = nullptr;
        that.m_cap = 0;
        return *this;
    }

    ~Buffer() noexcept {
        if (m_data)
            CHECK_CUDA(hipFree(m_data));
        m_data = nullptr;
    }

    void reserve(size_t new_cap) {
        if (m_cap < new_cap) {
            if (m_data)
                CHECK_CUDA(hipFree(m_data));
            m_data = nullptr;
            CHECK_CUDA(hipMalloc(&m_data, new_cap));
            m_cap = new_cap;
        }
    }

    size_t capacity() const noexcept {
        return m_cap;
    }

    void const *data() const noexcept {
        return m_data;
    }

    void *data() noexcept {
        return m_data;
    }
};

template <class T>
hipDataType cudaDataTypeFor();

template <>
hipDataType cudaDataTypeFor<int8_t>() {
    return HIP_R_8I;
}

template <>
hipDataType cudaDataTypeFor<uint8_t>() {
    return HIP_R_8U;
}

template <>
hipDataType cudaDataTypeFor<int16_t>() {
    return HIP_R_16I;
}

template <>
hipDataType cudaDataTypeFor<uint16_t>() {
    return HIP_R_16U;
}

template <>
hipDataType cudaDataTypeFor<int32_t>() {
    return HIP_R_32I;
}

template <>
hipDataType cudaDataTypeFor<uint32_t>() {
    return HIP_R_32U;
}

template <>
hipDataType cudaDataTypeFor<int64_t>() {
    return HIP_R_64I;
}

template <>
hipDataType cudaDataTypeFor<uint64_t>() {
    return HIP_R_64U;
}

template <>
hipDataType cudaDataTypeFor<nv_half>() {
    return HIP_R_16F;
}

template <>
hipDataType cudaDataTypeFor<hip_bfloat16>() {
    return HIP_R_16BF;
}

template <>
hipDataType cudaDataTypeFor<float>() {
    return HIP_R_32F;
}

template <>
hipDataType cudaDataTypeFor<double>() {
    return HIP_R_64F;
}





template <class T>
struct Vec {
    T *m_data;
    size_t m_size;
    size_t m_cap;

    Vec() noexcept : m_data(nullptr), m_size(0), m_cap(0) {
    }

    Vec(Vec &&that) noexcept : m_data(that.m_data), m_size(that.m_size), m_cap(that.m_cap) {
        that.m_data = nullptr;
        that.m_size = 0;
        that.m_cap = 0;
    }

    Vec &operator=(Vec &&that) noexcept {
        if (this == &that) return *this;
        if (m_data)
            CHECK_CUDA(hipFree(m_data));
        m_data = nullptr;
        m_data = that.m_data;
        m_size = that.m_size;
        m_cap = that.m_cap;
        that.m_data = nullptr;
        that.m_size = 0;
        that.m_cap = 0;
        return *this;
    }

    void swap(Vec &that) noexcept {
        std::swap(m_data, that.m_data);
        std::swap(m_size, that.m_size);
        std::swap(m_cap, that.m_cap);
    }

    ~Vec() noexcept {
        if (m_data)
            CHECK_CUDA(hipFree(m_data));
        m_data = nullptr;
    }

    void resize(size_t new_size) {
        bool change = m_cap < new_size;
        if (change) {
            if (m_data)
                CHECK_CUDA(hipFree(m_data));
            m_data = nullptr;
            CHECK_CUDA(hipMalloc(&m_data, sizeof(T) * new_size));
            m_cap = new_size;
        }
        if (m_size != new_size || change) {
            m_size = new_size;
        }
    }

    // host to device
    void assign(T const *data, size_t size) {
        resize(size);
        CHECK_CUDA(hipMemcpy(m_data, data, sizeof(T) * size, hipMemcpyHostToDevice));
    }

    // device to host
    void tohost(std::vector<T> &data_host) const{
        data_host.resize(size());
        CHECK_CUDA(hipMemcpy(data_host.data(), m_data, sizeof(T) * size(), hipMemcpyDeviceToHost));
    }

    size_t size() const noexcept {
        return m_size;
    }

    T const *data() const noexcept {
        return m_data;
    }

    T *data() noexcept {
        return m_data;
    }
};


template <class T>
struct HostVec {
    T *m_data;
    size_t m_size;
    size_t m_cap;

    HostVec() noexcept : m_data(nullptr), m_size(0), m_cap(0) {
    }

    HostVec(HostVec &&that) noexcept : m_data(that.m_data), m_size(that.m_size), m_cap(that.m_cap) {
        that.m_data = nullptr;
        that.m_size = 0;
        that.m_cap = 0;
    }

    HostVec &operator=(HostVec &&that) noexcept {
        if (this == &that) return *this;
        if (m_data)
            CHECK_CUDA(hipFree(m_data));
        m_data = nullptr;
        m_data = that.m_data;
        m_size = that.m_size;
        m_cap = that.m_cap;
        that.m_data = nullptr;
        that.m_size = 0;
        that.m_cap = 0;
        return *this;
    }

    void swap(HostVec &that) noexcept {
        std::swap(m_data, that.m_data);
        std::swap(m_size, that.m_size);
        std::swap(m_cap, that.m_cap);
    }

    ~HostVec() noexcept {
        if (m_data)
            CHECK_CUDA(hipHostFree(m_data));
        m_data = nullptr;
    }

    void resize(size_t new_size) {
        bool change = m_cap < new_size;
        if (change) {
            if (m_data)
                CHECK_CUDA(hipHostFree(m_data));
            m_data = nullptr;
            CHECK_CUDA(hipHostMalloc(&m_data, sizeof(T) * new_size));
            m_cap = new_size;
        }
        if (m_size != new_size || change) {
            m_size = new_size;
        }
    }

    // host to device
    void assign(T const *data, size_t size) {
        resize(size);
        CHECK_CUDA(hipMemcpy(m_data, data, sizeof(T) * size, hipMemcpyHostToDevice));
    }

    // device to host
    void tohost(std::vector<T> &data_host) const{
        data_host.resize(size());
        CHECK_CUDA(hipMemcpy(data_host.data(), m_data, sizeof(T) * size(), hipMemcpyDeviceToHost));
    }

    size_t size() const noexcept {
        return m_size;
    }

    T const *data() const noexcept {
        return m_data;
    }

    T *data() noexcept {
        return m_data;
    }
};


template <typename T=float>
std::vector<T> debug_cuda_vec(Vec<T> &v, std::string name) {
    std::vector<T> v_host(v.size());
    v.tohost(v_host);
    cout<<name<<"("<<v.size()<<") :";
    int k=0;
    for(auto i:v_host)
    {
        if(k>10)
            break;
        std::cout<<i<<" ";
        k++;
    }
    std::cout<<endl;
    return v_host;
}

struct DnVec {
    hipsparseDnVecDescr_t handle;

    operator hipsparseDnVecDescr_t() const noexcept {
        return handle;
    }

    DnVec() noexcept : handle(0) {}

    template <class T>
    DnVec(Vec<T> &v) {
        CHECK_CUSPARSE(hipsparseCreateDnVec(&handle, v.size(), v.data(), cudaDataTypeFor<T>()));
    }

    DnVec(DnVec &&that) noexcept : handle(that.handle) {
        that.handle = nullptr;
    }

    DnVec &operator=(DnVec &&that) noexcept {
        if (this == &that) return *this;
        if (handle)
            CHECK_CUSPARSE(hipsparseDestroyDnVec(handle));
        handle = that.handle;
        that.handle = nullptr;
        return *this;
    }

    ~DnVec() {
        if (handle)
            CHECK_CUSPARSE(hipsparseDestroyDnVec(handle));
    }
};

struct ConstDnVec {
    hipsparseConstDnVecDescr_t handle;

    operator hipsparseConstDnVecDescr_t() const noexcept {
        return handle;
    }

    ConstDnVec() noexcept : handle(0) {}

    template <class T>
    ConstDnVec(Vec<T> const &v) {
        CHECK_CUSPARSE(hipsparseCreateConstDnVec(&handle, v.size(), v.data(), cudaDataTypeFor<T>()));
    }

    ConstDnVec(ConstDnVec &&that) noexcept : handle(that.handle) {
        that.handle = nullptr;
    }

    ConstDnVec &operator=(ConstDnVec &&that) noexcept {
        if (this == &that) return *this;
        if (handle)
            CHECK_CUSPARSE(hipsparseDestroyDnVec(handle));
        handle = that.handle;
        that.handle = nullptr;
        return *this;
    }

    ~ConstDnVec() {
        if (handle)
            CHECK_CUSPARSE(hipsparseDestroyDnVec(handle));
    }
};


// Data of csr matrix
template <class T>
struct CSR {
    Vec<int> indices;
    Vec<T> data;
    Vec<int> indptr;
    int64_t nrows;
    int64_t ncols;
    int64_t numnonz;

    CSR() noexcept : nrows(0), ncols(0), numnonz(0) {}

    void assign(T const *datap, size_t ndat, int const *indicesp, size_t nind, int const *indptrp, size_t nptr, size_t rows, size_t cols, size_t nnz) {
        indices.resize(nind);
        indptr.resize(nptr);
        data.resize(ndat);
        CHECK_CUDA(hipMemcpy(data.data(), datap, data.size() * sizeof(T), hipMemcpyHostToDevice));
        CHECK_CUDA(hipMemcpy(indices.data(), indicesp, indices.size() * sizeof(int), hipMemcpyHostToDevice));
        CHECK_CUDA(hipMemcpy(indptr.data(), indptrp, indptr.size() * sizeof(int), hipMemcpyHostToDevice));
        nrows = rows;
        ncols = cols;
        numnonz = nnz;
    }

    void assign_v2(T const *datap,  int const *indicesp,  int const *indptrp, size_t rows, size_t cols, size_t nnz) {
        int ndat = nnz;
        int nind = nnz;
        int nptr = rows + 1;
        indices.resize(nind);
        indptr.resize(nptr);
        data.resize(ndat);
        CHECK_CUDA(hipMemcpy(data.data(), datap, data.size() * sizeof(T), hipMemcpyHostToDevice));
        CHECK_CUDA(hipMemcpy(indices.data(), indicesp, indices.size() * sizeof(int), hipMemcpyHostToDevice));
        CHECK_CUDA(hipMemcpy(indptr.data(), indptrp, indptr.size() * sizeof(int), hipMemcpyHostToDevice));
        nrows = rows;
        ncols = cols;
        numnonz = nnz;
    }

    void resize(size_t rows, size_t cols, size_t nnz) {
        nrows = rows;
        ncols = cols;
        numnonz = nnz;
        data.resize(nnz);
        indices.resize(nnz);
        indptr.resize(rows + 1);
    }

    void tohost(std::vector<T> &data_host, std::vector<int> &indices_host, std::vector<int> &indptr_host) const {
        data_host.resize(data.size());
        indices_host.resize(indices.size());
        indptr_host.resize(indptr.size());
        CHECK_CUDA(hipMemcpy(data_host.data(), data.data(), data.size() * sizeof(T), hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(indices_host.data(), indices.data(), indices.size() * sizeof(int), hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(indptr_host.data(), indptr.data(), indptr.size() * sizeof(int), hipMemcpyDeviceToHost));
    }
};


// template <class T>
// struct SuperCSR:CSR<T>
// {
//     Vec<T> ii;
//     Vec<T> jj;

//     SuperCSR() noexcept : nrows(0), ncols(0), numnonz(0) {}

//     void assign(T const *datap, size_t ndat, int const *indicesp, size_t nind, int const *indptrp, size_t nptr, size_t rows, size_t cols, size_t nnz, T const *iip, T const *jjp) {
//         indices.resize(nind);
//         indptr.resize(nptr);
//         data.resize(ndat);
//         CHECK_CUDA(hipMemcpy(data.data(), datap, data.size() * sizeof(T), hipMemcpyHostToDevice));
//         CHECK_CUDA(hipMemcpy(indices.data(), indicesp, indices.size() * sizeof(int), hipMemcpyHostToDevice));
//         CHECK_CUDA(hipMemcpy(indptr.data(), indptrp, indptr.size() * sizeof(int), hipMemcpyHostToDevice));
//         nrows = rows;
//         ncols = cols;
//         numnonz = nnz;

//         ii.resize(nnz);
//         jj.resize(nnz);
//         CHECK_CUDA(hipMemcpy(ii.data(), iip, ii.size() * sizeof(T), hipMemcpyHostToDevice));
//         CHECK_CUDA(hipMemcpy(jj.data(), jjp, jj.size() * sizeof(T), hipMemcpyHostToDevice));
//     }

//     void resize(size_t rows, size_t cols, size_t nnz) {
//         nrows = rows;
//         ncols = cols;
//         numnonz = nnz;
//         data.resize(nnz);
//         indices.resize(nnz);
//         indptr.resize(rows + 1);

//         ii.resize(nnz);
//         jj.resize(nnz);
//     }
// };



// container of handle and descriptor
struct SpMat {
    hipsparseSpMatDescr_t handle;

    operator hipsparseSpMatDescr_t() const noexcept {
        return handle;
    }

    SpMat() noexcept : handle(0) {}

    template <class T>
    SpMat(CSR<T> &m) {
        CHECK_CUSPARSE(hipsparseCreateCsr(&handle, m.nrows, m.ncols, m.numnonz,
                                         m.indptr.data(), m.indices.data(), m.data.data(),
                                         HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                         HIPSPARSE_INDEX_BASE_ZERO, cudaDataTypeFor<T>()) );
    }

    SpMat(SpMat &&that) noexcept : handle(that.handle) {
        that.handle = nullptr;
    }

    SpMat &operator=(SpMat &&that) noexcept {
        if (this == &that) return *this;
        if (handle)
            CHECK_CUSPARSE(hipsparseDestroySpMat(handle));
        handle = that.handle;
        that.handle = nullptr;
        return *this;
    }

    ~SpMat() {
        if (handle)
            CHECK_CUSPARSE(hipsparseDestroySpMat(handle));
    }
};

// container of handle and descriptor, const version
struct ConstSpMat {
    hipsparseConstSpMatDescr_t handle;

    operator hipsparseConstSpMatDescr_t() const noexcept {
        return handle;
    }

    ConstSpMat() noexcept : handle(0) {}

    template <class T>
    ConstSpMat(CSR<T> const &m) {
        CHECK_CUSPARSE(hipsparseCreateConstCsr(&handle, m.nrows, m.ncols, m.numnonz,
                                              m.indptr.data(), m.indices.data(), m.data.data(),
                                              HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                              HIPSPARSE_INDEX_BASE_ZERO, cudaDataTypeFor<T>()) );
    }

    ConstSpMat(SpMat &&that) noexcept : handle(that.handle) {
        that.handle = nullptr;
    }

    ConstSpMat &operator=(ConstSpMat &&that) noexcept {
        if (this == &that) return *this;
        if (handle)
            CHECK_CUSPARSE(hipsparseDestroySpMat(handle));
        handle = that.handle;
        that.handle = nullptr;
        return *this;
    }

    ~ConstSpMat() {
        if (handle)
            CHECK_CUSPARSE(hipsparseDestroySpMat(handle));
    }
};

struct Kernels {
    hipblasHandle_t cublas;
    hipsparseHandle_t cusparse;
    hipsolverSpHandle_t cusolverH;

    Kernels() {
        CHECK_CUSPARSE(hipsparseCreate(&cusparse));
        CHECK_CUBLAS(hipblasCreate(&cublas));
        CHECK_CUSOLVER(hipsolverSpCreate(&cusolverH));
    }

    Kernels(Kernels &&) = delete;

    ~Kernels() {
        CHECK_CUSPARSE(hipsparseDestroy(cusparse));
        CHECK_CUBLAS(hipblasDestroy(cublas));
        CHECK_CUSOLVER(hipsolverSpDestroy(cusolverH));
    }

    // out = alpha * A@x + beta * out
    void spmv(Vec<float> &out, float const &alpha, CSR<float> const &A, Vec<float> const &x, float const &beta, Buffer &buffer) {
        assert(out.size() == A.nrows);
        size_t bufSize = 0;
        ConstSpMat dA(A);
        ConstDnVec dx(x);
        DnVec dout(out);
        CHECK_CUSPARSE(hipsparseSpMV_bufferSize(cusparse, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                               &alpha, dA, dx, &beta,
                                               dout, cudaDataTypeFor<float>(),
                                               HIPSPARSE_SPMV_ALG_DEFAULT, &bufSize));
        buffer.reserve(bufSize);
        CHECK_CUSPARSE(hipsparseSpMV(cusparse, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    &alpha, dA, dx, &beta,
                                    dout, cudaDataTypeFor<float>(),
                                    HIPSPARSE_SPMV_ALG_DEFAULT, buffer.data()));
    }

    // C = A * B
    void spgemm(CSR<float> const &matA_,  CSR<float> const &matB_, CSR<float> &matC_) 
    {
        ConstSpMat descA(matA_); //descriptor for A
        ConstSpMat descB(matB_);
        matC_.resize(matA_.nrows, matB_.ncols, 0);
        SpMat descC(matC_);
        // https://github.com/NVIDIA/CUDALibrarySamples/blob/ade391a17672d26e55429035450bc44afd277d34/cuSPARSE/spgemm/spgemm_example.c#L161
        // https://docs.nvidia.com/cuda/cusparse/#cusparsespgemm
        //--------------------------------------------------------------------------
        float               alpha       = 1.0f;
        float               beta        = 0.0f;
        hipsparseOperation_t opA         = HIPSPARSE_OPERATION_NON_TRANSPOSE;
        hipsparseOperation_t opB         = HIPSPARSE_OPERATION_NON_TRANSPOSE;
        hipDataType        computeType = HIP_R_32F;
        //--------------------------------------------------------------------------
        // buffers
        size_t bufferSize1 = 0,    bufferSize2 = 0;
        Buffer dBuffer1, dBuffer2;
        //--------------------------------------------------------------------------
        // SpGEMM Computation
        hipsparseSpGEMMDescr_t spgemmDesc;
        CHECK_CUSPARSE( hipsparseSpGEMM_createDescr(&spgemmDesc) )

        // ask bufferSize1 bytes for external memory
        CHECK_CUSPARSE(
            hipsparseSpGEMM_workEstimation(cusparse, opA, opB,
                                        &alpha, descA, descB, &beta, descC,
                                        computeType, HIPSPARSE_SPGEMM_DEFAULT,
                                        spgemmDesc, &bufferSize1, NULL) )
        // CHECK_CUDA( hipMalloc((void**) &dBuffer1, bufferSize1) )
        dBuffer1.reserve(bufferSize1);

        // inspect the matrices A and B to understand the memory requirement for
        // the next step
        CHECK_CUSPARSE(
            hipsparseSpGEMM_workEstimation(cusparse, opA, opB,
                                        &alpha, descA, descB, &beta, descC,
                                        computeType, HIPSPARSE_SPGEMM_DEFAULT,
                                        spgemmDesc, &bufferSize1, dBuffer1.data()) )

        // ask bufferSize2 bytes for external memory
        CHECK_CUSPARSE(
            hipsparseSpGEMM_compute(cusparse, opA, opB,
                                &alpha, descA, descB, &beta, descC,
                                computeType, HIPSPARSE_SPGEMM_DEFAULT,
                                spgemmDesc, &bufferSize2, NULL) )
        dBuffer2.reserve(bufferSize2);

        // compute the intermediate product of A * B
        CHECK_CUSPARSE( hipsparseSpGEMM_compute(cusparse, opA, opB,
                                            &alpha, descA, descB, &beta, descC,
                                            computeType, HIPSPARSE_SPGEMM_DEFAULT,
                                            spgemmDesc, &bufferSize2, dBuffer2.data()) )
        // --------------------------------------------------------------------------
        // get matrix C non-zero entries C_nnz1
        CHECK_CUSPARSE( hipsparseSpMatGetSize(descC, &matC_.nrows, &matC_.ncols, &matC_.numnonz) )
        // allocate matrix C
        matC_.resize(matC_.nrows, matC_.ncols, matC_.numnonz);
        // update matC with the new pointers
        CHECK_CUSPARSE(hipsparseCsrSetPointers(descC, matC_.indptr.data(), matC_.indices.data(), matC_.data.data()) )

        // copy the final products to the matrix C
        CHECK_CUSPARSE(
            hipsparseSpGEMM_copy(cusparse, opA, opB,
                                &alpha, descA, descB, &beta, descC,
                                computeType, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc) )
    }


    // dst = src + alpha * dst
    void axpy(Vec<float> &dst, float const &alpha, Vec<float> const &src) {
        assert(dst.size() == src.size());
        CHECK_CUBLAS(hipblasSaxpy(cublas, dst.size(), &alpha, src.data(), 1, dst.data(), 1));
    }

    void zero(Vec<float> &dst) {
        CHECK_CUDA(hipMemset(dst.data(), 0, dst.size() * sizeof(float)));
    }

    void copy(Vec<float> &dst, Vec<float> const &src) {
        dst.resize(src.size());
        CHECK_CUDA(hipMemcpy(dst.data(), src.data(), src.size() * sizeof(float), hipMemcpyDeviceToDevice));
    }

    // dst = alpha * x
    void scal2(Vec<float> &dst, float const &alpha, Vec<float> const &x) {
        copy(dst, x);
        CHECK_CUBLAS(hipblasSscal(cublas, dst.size(), &alpha, dst.data(), 1));
    }

    // dst = alpha * dst
    void scal(Vec<float> &dst, float const &alpha) {
        CHECK_CUBLAS(hipblasSscal(cublas, dst.size(), &alpha, dst.data(), 1));
    }

    // x = A^{-1} b by cusolver cholesky
    // https://docs.nvidia.com/cuda/cusolver/index.html#cusolversp-t-csrlsvchol
    void spsolve(Vec<float> &x, CSR<float> const &A, Vec<float> &b) {
        hipsparseMatDescr_t descrA = NULL;
        CHECK_CUSPARSE(hipsparseCreateMatDescr(&descrA));
        CHECK_CUSPARSE(hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL));
        CHECK_CUSPARSE(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO)); 
        int singularity;

        CHECK_CUSOLVER( hipsolverSpScsrlsvchol(cusolverH, A.nrows, A.numnonz, descrA, A.data.data(), A.indptr.data(), A.indices.data(), b.data(), 1e-10, 0, x.data(), &singularity) );
    }

    float vdot(Vec<float> const &x, Vec<float> const &y) {
        float result;
        CHECK_CUBLAS(hipblasSdot(cublas, x.size(), x.data(), 1, y.data(), 1, &result));
        return result;
    }

    float vnorm(Vec<float> const &x) {
        float result;
        CHECK_CUBLAS(hipblasSnrm2(cublas, x.size(), x.data(), 1, &result));
        return result;
    }

    // transpose csr matrix A to AT
    // https://docs.nvidia.com/cuda/cusparse/index.html?highlight=hipsparseCsr2cscEx2#cusparsecsr2cscex2
    void transpose(CSR<float> const & A, CSR<float>& AT)
    {
        int m = A.nrows;
        int n = A.ncols;
        int nnz = A.numnonz;
        const float *csrVal  = A.data.data();
        const int *csrRowPtr = A.indptr.data();
        const int *csrColInd = A.indices.data();
        float *cscVal  = AT.data.data();
        int *cscColPtr = AT.indptr.data();
        int *cscRowInd = AT.indices.data();
        hipDataType  valType = HIP_R_32F;
        hipsparseAction_t copyValues = HIPSPARSE_ACTION_NUMERIC;
        hipsparseIndexBase_t idxBase = HIPSPARSE_INDEX_BASE_ZERO;
        hipsparseCsr2CscAlg_t    alg = HIPSPARSE_CSR2CSC_ALG_DEFAULT;
        hipsparseStatus_t status;
        size_t bufferSize = 0;
        Buffer buffer;

        CHECK_CUSPARSE( hipsparseCsr2cscEx2_bufferSize(cusparse, m, n, nnz, csrVal, csrRowPtr, csrColInd, cscVal, cscColPtr, cscRowInd, valType, copyValues, idxBase, alg, &bufferSize));
        buffer.reserve(bufferSize);
        CHECK_CUSPARSE( hipsparseCsr2cscEx2(           cusparse, m, n, nnz, csrVal, csrRowPtr, csrColInd, cscVal, cscColPtr, cscRowInd, valType, copyValues, idxBase, alg, buffer.data()));                
    }



//Calculate the largest eigenvalue of a symmetric matrix using the power method!
// https://docs.nvidia.com/cuda/cusolver/index.html#cusolversp-t-csreigvsi  (cusolverSpScsreigvsi is not used here, but it is another option, so I just keep the note. It use the shift inverse method to solve this equation Ax=lam x)
// Reference code: https://github.com/physicslog/maxEigenValueGPU/blob/25e0aa3d6c9bbeb03be6249d0ab8cfaafd32188c/maxeigenvaluepower.cu#L255
float computeMaxEigenvaluePowerMethodOptimized(CSR<float>& M, int max_iter) {
    // // Terminal output color (just for cosmetic purpose)
    // #define RST  "\x1B[37m"  // Reset color to white
    // #define KGRN  "\033[0;32m"   // Define green color
    // #define RD "\x1B[31m"  // Define red color
    // #define FGRN(x) KGRN x RST  // Define compiler function for green color
    // #define FRD(x) RD x RST  // Define compiler function for red color

  assert(M.nrows == M.ncols);

  // Initialize two vectors x_i and x_k
  thrust::device_vector<float> x_i(M.nrows), x_k(M.nrows, 0.0f);

  // Set x_i := the random vector
    thrust::transform(thrust::make_counting_iterator<int>(0),
    thrust::make_counting_iterator<int>(M.nrows),
    x_i.begin(),
    genRandomNumber());

  // CUSPARSE APIs
  hipsparseHandle_t handle = NULL;
  hipsparseSpMatDescr_t matM;
  hipsparseDnVecDescr_t xi, xk;
  void *dBuffer = NULL;
  size_t bufferSize = 0;
  float alpha = 1.0f;
  float beta = 0.0f;

  CHECK_CUSPARSE( hipsparseCreate(&handle) )

  CHECK_CUSPARSE( hipsparseCreateCsr(&matM, M.nrows, M.ncols, M.numnonz,
                                   thrust::raw_pointer_cast(M.indptr.data()),
                                   thrust::raw_pointer_cast(M.indices.data()),
                                   thrust::raw_pointer_cast(M.data.data()),
                                   HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                   HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )

  CHECK_CUSPARSE( hipsparseCreateDnVec(&xi, M.nrows, thrust::raw_pointer_cast(x_i.data()), HIP_R_32F) )
  CHECK_CUSPARSE( hipsparseCreateDnVec(&xk, M.nrows, thrust::raw_pointer_cast(x_k.data()), HIP_R_32F) )

  CHECK_CUSPARSE( hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                          &alpha, matM, xi, &beta, xk, HIP_R_32F,
                                          HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize) )

  CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

  float max_eigenvalue(0.0f), max_eigenvalue_prev(0.0f);
  float tol = 1e-4;  // tolerance for convergence
  int itr = 0;
  float err = 0.0f;
  // Power iteration method
  while (itr < max_iter) {
    // Compute x_k = A * x_i; generates Krylov subspace
    CHECK_CUSPARSE( hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matM, xi, &beta, xk, HIP_R_32F,
                                 HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer) )

    // Compute the L2 norm of x_k
    float norm = std::sqrt(thrust::inner_product(x_k.begin(), x_k.end(), x_k.begin(), 0.0f));

    // Normalize x_k and update x_i
    thrust::transform(x_k.begin(), x_k.end(), x_i.begin(), thrust::placeholders::_1 / norm);

    // Compute the maximum eigenvalue
    CHECK_CUSPARSE( hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                &alpha, matM, xi, &beta, xk, HIP_R_32F,
                                HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer) )

    max_eigenvalue = thrust::inner_product(x_i.begin(), x_i.end(), x_k.begin(), 0.0f);


    err = std::abs(max_eigenvalue - max_eigenvalue_prev);
    if (err < tol && itr >= 10) {
      std::cout << ("[NOTE]: ") << "Converged at iterations: " << itr << std::endl;
      return max_eigenvalue;
    }

    max_eigenvalue_prev = max_eigenvalue;
    itr++;
  }

  // Destroy the handle and descriptors
  CHECK_CUSPARSE( hipsparseDestroySpMat(matM) )
  CHECK_CUSPARSE( hipsparseDestroyDnVec(xi) )
  CHECK_CUSPARSE( hipsparseDestroyDnVec(xk) )
  CHECK_CUSPARSE( hipsparseDestroy(handle) )
  CHECK_CUDA( hipFree(dBuffer) )

  std::cout << ("\n[NOTE]: ") << "Max_iter("<<max_iter<<") reached when calculating max eig, error=" <<err<< std::endl;  // no convergence
  return max_eigenvalue;
}
};

struct MGLevel {
    CSR<float> A;
    CSR<float> R;
    CSR<float> P;
    Vec<float> residual;
    Vec<float> b;
    Vec<float> x;
    Vec<float> h;
    Vec<float> outh;
    CSR<float> Dinv;
    CSR<float> Aoff;
    float scale_RAP=0.0;
};


struct FastFillCloth : Kernels {
    CSR<float> A;
    float alpha;
    int NE;
    int NV;
    int num_nonz;
    int nrows, ncols;
    Vec<float> d_inv_mass;
    Vec<int> d_ii, d_jj;
    Vec<int> d_edges;
    Vec<float> d_pos;
    Vec<int> d_adjacent_edge_abc;
    Vec<int> d_num_adjacent_edge;

    void fetch_A_data(float *data_in) {
        CHECK_CUDA(hipMemcpy(data_in, A.data.data(), sizeof(float) * A.numnonz, hipMemcpyDeviceToHost));
    }

    void set_data_v2(int* edges_in, int NE_in, float* inv_mass_in, int NV_in, float* pos_in, float alpha_in)
    {
        NE = NE_in;
        NV = NV_in;
        nrows = NE;
        ncols = NE;

        d_edges.assign(edges_in, NE*2);
        d_inv_mass.assign(inv_mass_in, NV);
        d_pos.assign(pos_in, NV*3);

        alpha = alpha_in;
    }

    void update_pos_py2cu(float* pos_in)
    {
        d_pos.assign(pos_in, NV*3);
    }


    void init_from_python_cache_v2(
        int *adjacent_edge_in,
        int *num_adjacent_edge_in,
        int *adjacent_edge_abc_in,
        int num_nonz_in,
        float *spmat_data_in,
        int *spmat_indices_in,
        int *spmat_indptr_in,
        int *spmat_ii_in,
        int *spmat_jj_in,
        int NE_in,
        int NV_in)
    {
        NE = NE_in;
        NV = NV_in;
        num_nonz = num_nonz_in;

        printf("Copying A, ii, jj\n");
        A.assign(spmat_data_in, num_nonz, spmat_indices_in, num_nonz, spmat_indptr_in, NE+1, NE, NE, num_nonz);
        d_ii.assign(spmat_ii_in, num_nonz);
        d_jj.assign(spmat_jj_in, num_nonz);
        cout<<"Finish."<<endl;

        printf("Copying adj\n");
        d_num_adjacent_edge.assign(num_adjacent_edge_in, NE);
        d_adjacent_edge_abc.resize(NE*60);
        CHECK_CUDA(hipMemcpy(d_adjacent_edge_abc.data(), adjacent_edge_abc_in, sizeof(int) * NE * 60, hipMemcpyHostToDevice));
        cout<<"Finish."<<endl;
    }


    void run(float* pos_in)
    {
        update_pos_py2cu(pos_in);
        fill_A_CSR_gpu();
    }


    void fill_A_CSR_gpu()
    {
        fill_A_CSR_cloth_kernel<<<num_nonz / 256 + 1, 256>>>(A.data.data(),
                                                 A.indptr.data(),
                                                 A.indices.data(),
                                                 d_ii.data(),
                                                 d_jj.data(),
                                                 d_adjacent_edge_abc.data(),
                                                 d_num_adjacent_edge.data(),
                                                 num_nonz,
                                                 d_inv_mass.data(),
                                                 alpha,
                                                 NV,
                                                 NE,
                                                 d_edges.data(),
                                                 d_pos.data());
        hipDeviceSynchronize();
        launch_check();
    }
}; //FastFillCloth struct


struct FastFillSoft : Kernels {
    CSR<float> A;
    int NT;
    int NV;
    int MAX_ADJ;
    int num_nonz;
    int nrows, ncols;
    Vec<float> d_inv_mass;
    Vec<int> d_ii;
    Vec<float> d_pos;
    #ifndef USE_LESSMEM
    Vec<int> d_jj; //dont need jj, it is the same as indices
    Vec<int> d_adjacent;   
    Vec<int> d_num_adjacent; 
    Vec<int> d_n_shared_v;
    Vec<int> d_shared_v;
    Vec<int8_t> d_shared_v_order_in_cur;
    Vec<int8_t> d_shared_v_order_in_adj;
    Vec<int> d_nnz_each_row;
    #endif
    Vec<int> d_tet;
    Vec<float> d_gradC;
    Vec<float> d_alpha_tilde;


    void fetch_A_data(float *data_in) {
        CHECK_CUDA(hipMemcpy(data_in, A.data.data(), sizeof(float) * A.numnonz, hipMemcpyDeviceToHost));
    }

    void set_data_v2(int* tet_in, int NT_in, float* inv_mass_in, int NV_in, float* pos_in, float* alpha_tilde_in)
    {
        NT = NT_in;
        NV = NV_in;
        nrows = NT;
        ncols = NT;
        d_alpha_tilde.assign(alpha_tilde_in, NT);
        d_inv_mass.assign(inv_mass_in, NV);
        d_pos.assign(pos_in, NV*3);
        d_tet.assign(tet_in, NT*4);
    }

    void update_pos_and_gradC(float* pos_in, float* gradC_in)
    {
        d_pos.assign(pos_in, NV*3);
        d_gradC.assign(gradC_in, NT*4*3);
    }

    #ifndef USE_LESSMEM
    void init_from_python_cache_v2(
        const int NT_in,
        const int MAX_ADJ_in,
        const int* adjacent_in,
        const int* num_adjacent_in,
        const float* data_in,
        const int* indices_in,
        const int* indptr_in,
        const int* ii_in,
        const int* jj_in,
        const int num_nonz_in,
        const int* nnz_each_row_in,
        const int* n_shared_v_in,
        const int* shared_v_in,
        const int8_t* shared_v_order_in_cur,
        const int8_t* shared_v_order_in_adj
        )
    {
        NT = NT_in;
        MAX_ADJ = MAX_ADJ_in;

        num_nonz = num_nonz_in;
        ncols = NT;
        nrows = NT;
        A.assign_v2(data_in, indices_in, indptr_in, NT, NT, num_nonz);
        d_ii.assign(ii_in, num_nonz_in);
        d_jj.assign(jj_in, num_nonz_in);
        d_nnz_each_row.assign(nnz_each_row_in, NT);

        d_num_adjacent.assign(num_adjacent_in, NT);
        d_adjacent.assign(adjacent_in, NT*MAX_ADJ);
        d_n_shared_v.assign(n_shared_v_in, NT*MAX_ADJ);
        d_shared_v.assign(shared_v_in, NT*MAX_ADJ*3);
        d_shared_v_order_in_cur.assign(shared_v_order_in_cur, NT*MAX_ADJ*3);
        d_shared_v_order_in_adj.assign(shared_v_order_in_adj, NT*MAX_ADJ*3);

        cout<<"Finish load python cache to cuda."<<endl;
    }
    #endif

    void init_from_python_cache_lessmem(
        const int NT_in,
        const int MAX_ADJ_in,
        const float* data_in,
        const int* indices_in,
        const int* indptr_in,
        const int* ii_in,
        const int num_nonz_in
        )
        // const int* jj_in, //jj is the same as indicies
        // const int* nnz_each_row_in
        // const int* adjacent_in,
        // const int* num_adjacent_in,
        // const int* n_shared_v_in,
        // const int* shared_v_in,
        // const int8_t* shared_v_order_in_cur,
        // const int8_t* shared_v_order_in_adj
    {
        NT = NT_in;
        MAX_ADJ = MAX_ADJ_in;

        num_nonz = num_nonz_in;
        ncols = NT;
        nrows = NT;
        A.assign_v2(data_in, indices_in, indptr_in, NT, NT, num_nonz);
        d_ii.assign(ii_in, num_nonz_in);
        // d_jj.assign(jj_in, num_nonz_in);
        // d_nnz_each_row.assign(nnz_each_row_in, NT);
        // d_num_adjacent.assign(num_adjacent_in, NT);
        // d_adjacent.assign(adjacent_in, NT*MAX_ADJ);
        // d_n_shared_v.assign(n_shared_v_in, NT*MAX_ADJ);
        // d_shared_v.assign(shared_v_in, NT*MAX_ADJ*3);
        // d_shared_v_order_in_cur.assign(shared_v_order_in_cur, NT*MAX_ADJ*3);
        // d_shared_v_order_in_adj.assign(shared_v_order_in_adj, NT*MAX_ADJ*3);

        cout<<"Finish load python cache to cuda."<<endl;
    }


    void run(float* pos_in, float* gradC_in)
    {
        update_pos_and_gradC(pos_in, gradC_in);
        fill_A_CSR_gpu();
    }


    void fill_A_CSR_gpu()
    {
        // TODO
        // cout<<"Begin fill_A_CSR_gpu soft."<<endl;

        #ifndef USE_LESSMEM
        fill_A_CSR_soft_kernel<<<num_nonz / 256 + 1, 256>>>(
                A.data.data(),
                A.indptr.data(),
                A.indices.data(),
                d_ii.data(),
                d_jj.data(),
                d_adjacent.data(),
                d_num_adjacent.data(),
                num_nonz,
                d_inv_mass.data(),
                d_alpha_tilde.data(),
                NV,
                NT,
                MAX_ADJ,
                d_tet.data(),
                d_pos.data(),
                d_gradC.data(),
                d_n_shared_v.data(),
                d_shared_v.data(),
                d_shared_v_order_in_cur.data(),
                d_shared_v_order_in_adj.data()
        );
        #else
        fill_A_CSR_soft_lessmem_kernel<<<num_nonz / 256 + 1, 256>>>(
                A.data.data(),
                A.indptr.data(),
                A.indices.data(), //jj is the same as indices
                d_ii.data(),
                num_nonz,
                d_inv_mass.data(),
                d_alpha_tilde.data(),
                NV,
                NT,
                MAX_ADJ,
                d_tet.data(),
                d_pos.data(),
                d_gradC.data()
        );
        #endif
        hipDeviceSynchronize();
        launch_check();
        
        // cout<<"Finish fill_A_CSR_gpu soft."<<endl;

    }
}; //FastFillSoft struct


struct VCycle : Kernels {
    std::vector<MGLevel> levels;
    size_t nlvs;
    std::vector<float> chebyshev_coeff;
    size_t smoother_type = 1; //1:chebyshev, 2:w-jacobi, 3:gauss_seidel(level0)+w-jacobi(other levels)
    size_t coarse_solver_type = 1; //0:direct solver by cusolver (cholesky), 1: one sweep smoother
    float jacobi_omega;
    size_t smoother_niter=2; // TODO: we will replace smoother_niter later
    Vec<float> z;
    Vec<float> r;
    Vec<float> outer_x;
    Vec<float> x_new;
    Vec<float> outer_b;
    float save_rho_prev;
    Vec<float> save_p;
    Vec<float> save_q;
    Buffer buff;
    float rtol;
    size_t maxiter;
    std::vector<float> residuals;
    size_t niter; //final number of iterations to break the loop
    float max_eig;

    void set_scale_RAP(float s, int lv)
    {
        levels.at(lv).scale_RAP = s;
        cout<<"Set scale_RAP: "<<levels.at(lv).scale_RAP<<"  at level "<<lv<<endl;
    }

    void setup_smoothers(int type) {
        cout<<"\nSetting up smoothers..."<<endl;
        smoother_type = type;
        if(smoother_type == 1)
        {
            setup_chebyshev_cuda(levels[0].A);
        }
        else if (smoother_type == 2)
        {
            setup_jacobi_cuda(levels[0].A, smoother_niter);
        }
        else if (smoother_type == 3)
        {
            if (nlvs > 1)
            {
                compute_RAP(0);
                setup_jacobi_cuda(levels[1].A, smoother_niter);
            }
        }
    }


    void setup_chebyshev_cuda(CSR<float> &A) {
        float lower_bound=1.0/30.0;
        float upper_bound=1.1;
        float rho = computeMaxEigenvaluePowerMethodOptimized(A, 100);
        float a = rho * lower_bound;
        float b = rho * upper_bound;
        chebyshev_polynomial_coefficients(a, b);
        
        max_eig = rho;
        cout<<"max eigenvalue: "<<max_eig<<endl;
    }


    void chebyshev_polynomial_coefficients(float a, float b)
    {
        int degree=3;
        const float PI = 3.14159265358979323846;

        if(a >= b || a <= 0)
            assert(false && "Invalid input for Chebyshev polynomial coefficients");

        // Chebyshev roots for the interval [-1,1]
        std::vector<float> std_roots(degree);
        for(int i=0; i<degree; i++)
        {
            std_roots[i] = std::cos(PI * (i + 0.5) / (float)degree);
        }

        // Chebyshev roots for the interval [a,b]
        std::vector<float> scaled_roots(degree);
        for(int i=0; i<degree; i++)
        {
            scaled_roots[i] = 0.5 * (b-a) * (1 + std_roots[i]) + a;
        }

        // Compute monic polynomial coefficients of polynomial with scaled roots
        std::vector<float> scaled_poly(4);
        // np.poly for 3 roots. This will calc the coefficients of the polynomial from roots.
        // i.e., (x - root1) * (x - root2) * (x - root3) = x^3 - (root1 + root2 + root3)x^2 + (root1*root2 + root2*root3 + root3*root1)x - root1*root2*root3
        scaled_poly[0] = 1.0;
        scaled_poly[1] = -(scaled_roots[0] + scaled_roots[1] + scaled_roots[2]);
        scaled_poly[2] = scaled_roots[0]*scaled_roots[1] + scaled_roots[1]*scaled_roots[2] + scaled_roots[2]*scaled_roots[0];
        scaled_poly[3] = -scaled_roots[0]*scaled_roots[1]*scaled_roots[2];

        // Scale coefficients to enforce C(0) = 1.0
        float c0 = scaled_poly[3];
        for(int i=0; i<degree; i++)
        {
            scaled_poly[i] /= c0; 
        }


        chebyshev_coeff.resize(degree);
        //CAUTION:setup_chebyshev has "-" at the end
        for(int i=0; i<degree; i++)
        {
            chebyshev_coeff[i] = -scaled_poly[i];
        }

        cout<<"Chebyshev polynomial coefficients: ";
        for(int i=0; i<degree; i++)
        {
            cout<<chebyshev_coeff[i]<<" ";
        }
        cout<<endl;
    }


    float calc_residual_norm(Vec<float> const &b, Vec<float> const &x, CSR<float> const &A) {
        float rnorm = 0.0;
        Vec<float> r;
        r.resize(b.size());
        copy(r, b);
        spmv(r, -1, A, x, 1, buff);
        rnorm = vnorm(r);
        return rnorm;
    }


    void setup(size_t numlvs) {
        if (levels.size() < numlvs) {
            levels.resize(numlvs);
        }
        nlvs = numlvs;
        chebyshev_coeff.clear();
        jacobi_omega = 0.0;
    }


    void set_P(size_t lv, float const *datap, size_t ndat, int const *indicesp, size_t nind, int const *indptrp, size_t nptr, size_t rows, size_t cols, size_t nnz) {
        levels.at(lv).P.assign(datap, ndat, indicesp, nind, indptrp, nptr, rows, cols, nnz);
    }

    void set_A0(float const *datap, size_t ndat, int const *indicesp, size_t nind, int const *indptrp, size_t nptr, size_t rows, size_t cols, size_t nnz) {
        levels.at(0).A.assign(datap, ndat, indicesp, nind, indptrp, nptr, rows, cols, nnz);
    }


    int get_nnz(int lv) {
        return levels.at(lv).A.numnonz;
    }

    int get_nrows(int lv) {
        return levels.at(lv).A.nrows;
    }

    // only update the data of A0
    void update_A0(float const *datap) {
        CHECK_CUDA(hipMemcpy(levels.at(0).A.data.data(), datap, levels.at(0).A.data.size() * sizeof(float), hipMemcpyHostToDevice));
    }


    void set_A0_from_fastFillCloth(FastFillCloth *ff) {
        levels.at(0).A.data.swap( (ff->A).data);
        levels.at(0).A.indices.swap( (ff->A).indices);
        levels.at(0).A.indptr.swap((ff->A).indptr);
        levels.at(0).A.numnonz = ( ff->num_nonz);
        levels.at(0).A.nrows = ( ff->nrows);

    }

    void set_A0_from_fastFillSoft(FastFillSoft *ff) {

        if (levels.size() < 1) {
            levels.resize(1);
        }

        levels.at(0).A.numnonz = ( ff->num_nonz);
        levels.at(0).A.nrows = ( ff->nrows);

        //FIXME: As in python code, we need copy A, why?

        CHECK_CUDA(hipMemcpy(levels.at(0).A.data.data(), (ff->A).data.data(), levels.at(0).A.data.size() * sizeof(float), hipMemcpyDeviceToDevice));
        CHECK_CUDA(hipMemcpy(levels.at(0).A.indices.data(), (ff->A).indices.data(), levels.at(0).A.indices.size() * sizeof(int), hipMemcpyDeviceToDevice));
        CHECK_CUDA(hipMemcpy(levels.at(0).A.indptr.data(), (ff->A).indptr.data(), levels.at(0).A.indptr.size() * sizeof(int), hipMemcpyDeviceToDevice));

    }


    void chebyshev(int lv, Vec<float> &x, Vec<float> const &b) {
        copy(levels.at(lv).residual, b);
        spmv(levels.at(lv).residual, -1, levels.at(lv).A, x, 1, buff); // residual = b - A@x
        scal2(levels.at(lv).h, chebyshev_coeff.at(0), levels.at(lv).residual); // h = c0 * residual


        for (int i = 1; i < chebyshev_coeff.size(); ++i) {
            // h' = ci * residual + A@h
            copy(levels.at(lv).outh, levels.at(lv).residual);
            spmv(levels.at(lv).outh, 1, levels.at(lv).A, levels.at(lv).h, chebyshev_coeff.at(i), buff);

            // copy(levels.at(lv).h, levels.at(lv).outh);
            levels.at(lv).h.swap(levels.at(lv).outh);
        }

        axpy(x, 1, levels.at(lv).h); // x += h
    }

    void setup_jacobi(float const omega, size_t const n) {
        smoother_type = 2;
        jacobi_omega = omega;
        smoother_niter = n;
    }

    void set_smoother_niter(size_t const n) {
        smoother_niter = n;
    }


    void setup_jacobi_cuda(CSR<float>&A, size_t const n) {
        // smoother_type = 2;
        GpuTimer timer;
        timer.start();
        smoother_niter = n;

        // calc Dinv@A
        // Vec<float> Dinv;
        Vec<float> data_new;
        Vec<float> diag_inv;
        // Dinv.resize(A.nrows);
        data_new.resize(A.data.size());
        diag_inv.resize(A.nrows);
        calc_diag_inv_kernel<<<(A.nrows + 255) / 256, 256>>>(diag_inv.data(),A.data.data(), A.indices.data(), A.indptr.data(), A.nrows);
        hipDeviceSynchronize();

        scale_csr_by_row<<<(A.nrows + 255) / 256, 256>>>(data_new.data(), A.data.data(), A.indices.data(), A.indptr.data(), A.nrows, diag_inv.data());
        hipDeviceSynchronize();
        launch_check();

        CSR<float> DinvA;
        DinvA.assign(data_new.data(), A.numnonz, A.indices.data(), A.numnonz, A.indptr.data(), A.nrows+1, A.nrows, A.ncols, A.numnonz);


        float DinvA_rho = calc_max_eig(DinvA);
        jacobi_omega = 2.0 / (DinvA_rho+0.1);
        cout<<"DinvA_rho: "<<DinvA_rho<<endl;
        cout<<"jacobi_omega: "<<jacobi_omega<<endl; 
        timer.stop();
        cout<<"setup_jacobi_cuda time: "<<timer.elapsed()<<" ms"<<endl;
    }


    void get_Aoff_and_Dinv(CSR<float> &A, CSR<float> &Dinv, CSR<float> &Aoff)
    {
        int n = A.nrows;
        // get diagonal inverse of A, fill into a vector
        Vec<float> d_diag_inv;
        d_diag_inv.resize(n);
        calc_diag_inv_kernel<<<(n + 255) / 256, 256>>>(d_diag_inv.data(),A.data.data(), A.indices.data(), A.indptr.data(), n);
        hipDeviceSynchronize();
        LAUNCH_CHECK();


        // fill diag to a CSR matrix Dinv
        std::vector<int> seqence(n);
        for(int i=0; i<n; i++)
            seqence[i] = i;
        // copy d_diag_inv to host
        std::vector<float> h_diag_inv(n);
        CHECK_CUDA(hipMemcpy(h_diag_inv.data(), d_diag_inv.data(), n*sizeof(float), hipMemcpyDeviceToHost));
        Dinv.assign_v2(h_diag_inv.data(), seqence.data(), seqence.data(), n, n, n);
        hipDeviceSynchronize();
        LAUNCH_CHECK();


        Aoff.resize(n,n,A.numnonz);
        CHECK_CUDA(hipMemcpy(Aoff.data.data(), A.data.data(), A.numnonz*sizeof(float), hipMemcpyDeviceToDevice));
        Aoff.assign(Aoff.data.data(), A.numnonz, A.indices.data(), A.numnonz, A.indptr.data(), n+1, n, n, A.numnonz);
        // get Aoff by set diagonal of A to 0
        get_Aoff_kernel<<<(A.numnonz + 255) / 256, 256>>>(Aoff.data.data(), A.indices.data(), A.indptr.data(), n);
        hipDeviceSynchronize();
        LAUNCH_CHECK();
    }


    void jacobi(int lv, Vec<float> &x, Vec<float> const &b) {
        Vec<float> x_old;
        x_old.resize(x.size());
        copy(x_old, x);
        for (int i = 0; i < smoother_niter; ++i) {
            weighted_jacobi_kernel<<<(levels.at(lv).A.nrows + 255) / 256, 256>>>(x.data(), x_old.data(), b.data(), levels.at(lv).A.data.data(), levels.at(lv).A.indices.data(), levels.at(lv).A.indptr.data(), levels.at(lv).A.nrows, jacobi_omega);
            x.swap(x_old);
        }
    }

    // use cusparse instead of hand-written kernel
    void jacobi_v2(int lv, Vec<float> &x, Vec<float> const &b) {
        Vec<float> x_old;
        x_old.resize(x.size());
        copy(x_old, x);

        Vec<float> b1,b2;
        b1.resize(b.size());
        b2.resize(b.size());
        for (int i = 0; i < smoother_niter; ++i) {
            //x = omega * Dinv * (b - Aoff@x_old) + (1-omega)*x_old

            // 1. b1 = b-Aoff@x_old
            copy(b1, b);
            spmv(b1, -1, levels.at(lv).Aoff, x_old, 1, buff);

            // 2. b2 = omega*Dinv@b1
            spmv(b2, jacobi_omega, levels.at(lv).Dinv, b1, 0, buff);

            // 3. x = b2 + (1-omega)*x_old
            copy(x, x_old);
            axpy(x, 1-jacobi_omega, b2);

            x.swap(x_old);
        }   
    }


    void jacobi_cpu(int lv, Vec<float> &x, Vec<float> const &b) {
        // serial jacobi
        std::vector<float> x_host(x.size());
        std::vector<float> b_host(b.size());
        x.tohost(x_host);
        b.tohost(b_host);
        std::vector<float> data_host;
        std::vector<int> indices_host, indptr_host;
        levels.at(lv).A.tohost(data_host, indices_host, indptr_host);
        // cout<<"omega: "<<jacobi_omega<<endl;
        jacobi_serial(
            indptr_host.data(), indptr_host.size(),
            indices_host.data(), indices_host.size(),
            data_host.data(), data_host.size(),
            x_host.data(), x_host.size(),
            b_host.data(), b_host.size(),
            x_host.data(), x_host.size(),
            0, levels.at(lv).A.nrows, 1, jacobi_omega);
        x.assign(x_host.data(), x_host.size());
        // auto r = calc_residual_norm(b, x, levels.at(lv).A);
        // cout<<"lv"<<lv<<"   rnorm: "<<r<<endl;
    }



    void gauss_seidel_cpu(int lv, Vec<float> &x, Vec<float> const &b) {
        // serial gauss seidel
        std::vector<float> x_host(x.size());
        std::vector<float> b_host(b.size());
        x.tohost(x_host);
        b.tohost(b_host);
        std::vector<float> data_host;
        std::vector<int> indices_host, indptr_host;
        levels.at(lv).A.tohost(data_host, indices_host, indptr_host);
        gauss_seidel_serial(
            indptr_host.data(), indptr_host.size(),
            indices_host.data(), indices_host.size(),
            data_host.data(), data_host.size(),
            x_host.data(), x_host.size(),
            b_host.data(), b_host.size(),
            0, levels.at(lv).A.nrows, 1);
        x.assign(x_host.data(), x_host.size());
        // auto r = calc_residual_norm(b, x, levels.at(lv).A);
        // cout<<"lv"<<lv<<"   rnorm: "<<r<<endl;
    }

    Vec<int> colors; // color index of each node
    int color_num; // number of colors, max(colors)+1
    // parallel gauss seidel
    // https://erkaman.github.io/posts/gauss_seidel_graph_coloring.html
    // https://gist.github.com/Erkaman/b34b3531e209a1db38e259ea53ff0be9#file-gauss_seidel_graph_coloring-cpp-L101
    void set_colors(const int* c, int n, int color_num_in) {
        // get colors from python
        // TODO:
        colors.resize(n);
        CHECK_CUDA(hipMemcpy(colors.data(), c, n*sizeof(int), hipMemcpyHostToDevice));
        color_num = color_num_in;

    }

    void multi_color_gauss_seidel(int lv, Vec<float> &x, Vec<float> const &b) {
        for(int color=0; color<color_num; color++)
        {
            multi_color_gauss_seidel_kernel<<<(levels.at(lv).A.nrows + 255) / 256, 256>>>(x.data(), b.data(), levels.at(lv).A.data.data(), levels.at(lv).A.indices.data(), levels.at(lv).A.indptr.data(), levels.at(lv).A.nrows, colors.data(), color);
        }
    }

    // typedef std::vector<int> Partition;
	// int multi_color_gauss_seidel_impl(Vec& x, const Vec& b, const Mat& m, const std::vector<Partition>& partitions) {
    //     for (Partition partition : partitions) {
    //         // we do a gauss-seidel step for this partition.
    //         // every partition stores a set of variables that will be solved for.
    //         // and these variables can be solved for independently of each other.
    //         // thus, the below loop can easily be parallelized.
    //         // note that this code is very similar to the Gauss-Seidel method implemented
    //         // in the previous article. It's just that the variables are solved for in a different order.
    //         for (int variable : partition) {
    //             float s = 0.0f;
    //             for (int j = 0; j < N; ++j) {
    //                 if (j != variable) {
    //                     s += m.m[variable][j] * x.v[j];
    //                 }
    //             }
    //             x.v[variable] = (1.0f / m.m[variable][variable]) * (b.v[variable] - s);
    //         }
    //     }
	// }



    GpuTimer timer_smoother;
    std::vector<float> elapsed_smoother;

    void _smooth(int lv, Vec<float> &x, Vec<float> const &b) {
        timer_smoother.start();
        if(smoother_type == 1)
        {
            for(int i=0; i<smoother_niter; i++)
                chebyshev(lv, x, b);
        }
        else if (smoother_type == 2)
        {
            // jacobi_cpu(lv, x, b);
            // jacobi(lv, x, b);
            jacobi_v2(lv, x, b);
        }
        else if (smoother_type == 3)
        {
            // gauss_seidel_cpu(lv, x, b);
            if(lv==0)
                for(int i=0; i<smoother_niter; i++)
                    multi_color_gauss_seidel(lv,x,b);
            else{
                jacobi_v2(lv,x,b);
            }
        }
        timer_smoother.stop();
        elapsed_smoother.push_back(timer_smoother.elapsed());
    }


    float calc_residual(int lv, CSR<float> const &A, Vec<float> &x, Vec<float> const &b) {
        copy(r, b);
        spmv(r, -1, A, x, 1, buff); // residual = b - A@x
        return vnorm(r);
    }


    void vcycle_down() {
        for (int lv = 0; lv < nlvs-1; ++lv) {
            Vec<float> &x = lv != 0 ? levels.at(lv - 1).x : z;
            Vec<float> &b = lv != 0 ? levels.at(lv - 1).b : r;

            _smooth(lv, x, b);

            copy(levels.at(lv).residual, b);
            spmv(levels.at(lv).residual, -1, levels.at(lv).A, x, 1, buff); // residual = b - A@x

            levels.at(lv).b.resize(levels.at(lv).R.nrows);
            spmv(levels.at(lv).b, 1, levels.at(lv).R, levels.at(lv).residual, 0, buff); // coarse_b = R@residual

            levels.at(lv).x.resize(levels.at(lv).b.size());
            zero(levels.at(lv).x);
        }
    }

    void vcycle_up() {
        for (int lv = nlvs-2; lv >= 0; --lv) {
            Vec<float> &x = lv != 0 ? levels.at(lv - 1).x : z;
            Vec<float> &b = lv != 0 ? levels.at(lv - 1).b : r;
            spmv(x, 1, levels.at(lv).P, levels.at(lv).x, 1, buff); // x += P@coarse_x
            _smooth(lv, x, b);
        }
    }

    void vcycle() {
        vcycle_down();
        coarse_solve();
        vcycle_up();
    }


    void coarse_solve() {
        auto const &A = levels.at(nlvs - 1).A;
        auto &x = levels.at(nlvs - 2).x;
        auto &b = levels.at(nlvs - 2).b;
        if (coarse_solver_type==0)
        {
            spsolve(x, A, b);
        }
        else if (coarse_solver_type==1)
        {
            _smooth(nlvs-1, x, b);
        }
    }

    void set_outer_x(float const *x, size_t n) {
        outer_x.resize(n);
        CHECK_CUDA(hipMemcpy(outer_x.data(), x, n * sizeof(float), hipMemcpyHostToDevice));
        copy(x_new, outer_x);
    }

    void set_outer_b(float const *b, size_t n) {
        outer_b.resize(n);
        CHECK_CUDA(hipMemcpy(outer_b.data(), b, n * sizeof(float), hipMemcpyHostToDevice));
    }

    float init_cg_iter0(float *residuals) {
        float bnrm2 = vnorm(outer_b);
        // r = b - A@(x)
        copy(r, outer_b);
        spmv(outer_b, -1, levels.at(0).A, outer_x, 1, buff);
        float normr = vnorm(r);
        residuals[0] = normr;
        return bnrm2;
    }

    void do_cg_itern(float *residuals, size_t iteration) {
        float rho_cur = vdot(r, z);
        if (iteration > 0) {
            float beta = rho_cur / save_rho_prev;
            // p *= beta
            // p += z
            scal(save_p, beta);
            axpy(save_p, 1, z);
        } else {
            // p = move(z)
            save_p.swap(z);
        }
        // q = A@(p)
        save_q.resize(levels.at(0).A.nrows);
        spmv(save_q, 1, levels.at(0).A, save_p, 0, buff);
        save_rho_prev = rho_cur;
        float alpha = rho_cur / vdot(save_p, save_q);
        // x += alpha*p
        axpy(x_new, alpha, save_p);
        // r -= alpha*q
        axpy(r, -alpha, save_q);
        float normr = vnorm(r);
        residuals[iteration + 1] = normr;
    }

    void compute_RAP(size_t lv) {
            CSR<float> &A = levels.at(lv).A;
            CSR<float> &R = levels.at(lv).R;
            CSR<float> &P = levels.at(lv).P;
            CSR<float> AP;
            CSR<float> &RAP = levels.at(lv+1).A;
            R.resize(P.ncols, P.nrows, P.numnonz);
            transpose(P, R);            
            spgemm(A, P, AP) ;
            spgemm(R, AP, RAP);

            float s = levels.at(lv).scale_RAP;
            if (s!=0.0){
                // scale RAP by a scalar
                cout<<"scaling RAP by "<<s<<" at lv "<<lv<<endl;
                scal(RAP.data, s);
            }
    }

    void fetch_A_data(float *data) {
        CSR<float> &A = levels.at(0).A;
        CHECK_CUDA(hipMemcpy(data, A.data.data(), A.data.size() * sizeof(float), hipMemcpyDeviceToHost));
    }

    // In python end, before you call fetch A, you should call get_nnz and get_matsize first to determine the size of the csr matrix. 
    void fetch_A(size_t lv, float *data, int *indices, int *indptr) {
        CSR<float> &A = levels.at(lv).A;
        CHECK_CUDA(hipMemcpy(data, A.data.data(), A.data.size() * sizeof(float), hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(indices, A.indices.data(), A.indices.size() * sizeof(int), hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(indptr, A.indptr.data(), A.indptr.size() * sizeof(int), hipMemcpyDeviceToHost));
    }
    
    void set_data(const float* x, size_t nx, const float* b, size_t nb, float rtol_, size_t maxiter_)
    {
        set_outer_x(x, nx);
        set_outer_b(b, nb);
        rtol = rtol_;
        maxiter = maxiter_;
        residuals.resize(maxiter+1);
    }

    float calc_max_eig(CSR<float>& A)
    {
        return  computeMaxEigenvaluePowerMethodOptimized(A, 100);
    }

    size_t get_data(float* x_out, float* r_out)
    {
        CHECK_CUDA(hipMemcpy(x_out, x_new.data(), x_new.size() * sizeof(float), hipMemcpyDeviceToHost));
        std::copy(residuals.begin(), residuals.end(), r_out);
        return niter;
    }

    float sum(std::vector<float> &v)
    {
        return std::accumulate(v.begin(), v.end(), 0.0);
    }

    float avg(std::vector<float> &v)
    {
        return std::accumulate(v.begin(), v.end(), 0.0) / v.size();
    }

    void presolve()
    {
        // TODO: move fillA and RAP from python-end to here as well in the future refactoring
        for(int lv=0; lv<nlvs; lv++)
        {
            // for jacobi_v2 (use cusparse etc.)

            get_Aoff_and_Dinv(levels.at(lv).A, levels.at(lv).Dinv, levels.at(lv).Aoff);
        }
    }

    GpuTimer timer1,timer2;
    std::vector<float> elapsed1, elapsed2;
    void solve()
    {
        timer1.start();
        presolve();

        float bnrm2 = init_cg_iter0(residuals.data());
        float atol = bnrm2 * rtol;
        for (size_t iter=0; iter<maxiter; iter++)
        {   
            if (residuals[iter] < atol)
            {
                niter = iter;
                break;
            }
            copy(z, outer_x);
            
            timer2.start();
            vcycle();
            timer2.stop();
            elapsed2.push_back(timer2.elapsed());

            do_cg_itern(residuals.data(), iter); 
            niter = iter;
        }
        timer1.stop();
        elapsed1.push_back(timer1.elapsed());
        cout<<elapsed1.size()<<" mgpcg time: "<<(elapsed1[0])<<" ms"<<endl;
        cout<<elapsed2.size()<<" vcycle time: "<<sum(elapsed2)<<" ms"<<endl;
        elapsed1.clear();
        elapsed2.clear();

        cout<<elapsed_smoother.size()<<" smoother time: "<<avg(elapsed_smoother)<<" ms"<<" total time: "<<sum(elapsed_smoother)<<" ms"<<endl;
        elapsed_smoother.clear();

        // cout<<"Ax=b residuals: "<<endl;
        // for(int i=0; i<niter;++i)
        // {
        //     cout<<residuals[i]<<endl;
        // }
    }

    void solve_only_smoother()
    {
        timer1.start();
        presolve();
        float bnrm2 = init_cg_iter0(residuals.data());
        float atol = bnrm2 * rtol;
        for (size_t iter=0; iter<maxiter; iter++)
        {   
            _smooth(0, outer_x, outer_b);
            auto r = calc_residual_norm(outer_b, outer_x, levels.at(0).A);
            residuals[iter] = r;
            if (residuals[iter] < atol)
            {
                niter = iter;
                break;
            }
        }
        copy(x_new, outer_x);

        // cout<<"Ax=b residuals: "<<endl;
        // for(int i=0; i<niter;++i)
        // {
        //     cout<<residuals[i]<<endl;
        // }
        timer1.stop();
        elapsed1.push_back(timer1.elapsed());
        cout<<elapsed1.size()<<" only smoother time: "<<(elapsed1[0])<<" ms"<<endl;
        elapsed1.clear();

    }
};

} // namespace


static VCycle *fastmg = nullptr;
static FastFillCloth *fastFillCloth = nullptr;
static FastFillSoft *fastFillSoft = nullptr;

#if _WIN32
#define DLLEXPORT __declspec(dllexport)
#else
#define DLLEXPORT
#endif

extern "C" DLLEXPORT void fastmg_new() {
    if (!fastmg)
        fastmg = new VCycle{};
}

extern "C" DLLEXPORT void fastmg_setup_nl(size_t numlvs) {
    fastmg->setup(numlvs);
}

extern "C" DLLEXPORT void fastmg_setup_jacobi(float const omega, size_t const niter_jacobi) {
    fastmg->setup_jacobi(omega, niter_jacobi);
}


extern "C" DLLEXPORT void fastmg_RAP(size_t lv) {
    fastmg->compute_RAP(lv);
}


extern "C" DLLEXPORT int fastmg_get_nnz(size_t lv) {
    int nnz = fastmg->get_nnz(lv);
    std::cout<<"nnz: "<<nnz<<std::endl;
    return nnz;
}

extern "C" DLLEXPORT int fastmg_get_matsize(size_t lv) {
    int n = fastmg->get_nrows(lv);
    std::cout<<"matsize: "<<n<<std::endl;
    return n;
}

extern "C" DLLEXPORT void fastmg_fetch_A(size_t lv, float* data, int* indices, int* indptr) {
    fastmg->fetch_A(lv, data, indices, indptr);
}

extern "C" DLLEXPORT void fastmg_fetch_A_data(float* data) {
    fastmg->fetch_A_data(data);
}

extern "C" DLLEXPORT void fastmg_solve() {
    fastmg->solve();
}

extern "C" DLLEXPORT void fastmg_set_data(const float* x, size_t nx, const float* b, size_t nb, float rtol, size_t maxiter) {
    fastmg->set_data(x, nx, b, nb, rtol, maxiter);
}

extern "C" DLLEXPORT size_t fastmg_get_data(float *x, float *r) {
    size_t niter = fastmg->get_data(x, r);
    return niter;
}

extern "C" DLLEXPORT void fastmg_set_A0(float* data, int* indices, int* indptr, int rows, int cols, int nnz)
{
    fastmg->set_A0(data, nnz, indices, nnz, indptr, rows + 1, rows, cols, nnz);
}

// only update the data of A0
extern "C" DLLEXPORT void fastmg_update_A0(const float* data_in)
{
    fastmg->update_A0(data_in);
}

extern "C" DLLEXPORT void fastmg_set_P(int lv, float* data, int* indices, int* indptr, int rows, int cols, int nnz)
{
    fastmg->set_P(lv, data, nnz, indices, nnz, indptr, rows + 1, rows, cols, nnz);
}


extern "C" DLLEXPORT void fastmg_setup_smoothers(int type) {
    fastmg->setup_smoothers(type);
}


extern "C" DLLEXPORT void fastmg_set_smoother_niter(const size_t niter) {
    fastmg->set_smoother_niter(niter);
}

extern "C" DLLEXPORT void fastmg_set_A0_from_fastFillCloth() {
    fastmg->set_A0_from_fastFillCloth(fastFillCloth);
}

extern "C" DLLEXPORT void fastmg_set_A0_from_fastFillSoft() {
    fastmg->set_A0_from_fastFillSoft(fastFillSoft);
}

extern "C" DLLEXPORT void fastmg_scale_RAP(float s, int lv) {
    fastmg->set_scale_RAP(s, lv);
}

extern "C" DLLEXPORT void fastmg_set_colors(const int *c, int n, int color_num) {
    fastmg->set_colors(c, n, color_num);
}


extern "C" DLLEXPORT void fastmg_solve_only_smoother() {
    fastmg->solve_only_smoother();
}

// ------------------------------------------------------------------------------
extern "C" DLLEXPORT void fastFillCloth_new() {
    if (!fastFillCloth)
        fastFillCloth = new FastFillCloth{};
}

extern "C" DLLEXPORT void fastFillCloth_set_data(int* edges_in, int NE_in, float* inv_mass_in, int NV_in, float* pos_in, float alpha_in)
{
    fastFillCloth->set_data_v2(edges_in, NE_in, inv_mass_in, NV_in, pos_in, alpha_in);
}


extern "C" DLLEXPORT void fastFillCloth_init_from_python_cache(
    int *adjacent_edge_in,
    int *num_adjacent_edge_in,
    int *adjacent_edge_abc_in,
    int num_nonz_in,
    float *spmat_data_in,
    int *spmat_indices_in,
    int *spmat_indptr_in,
    int *spmat_ii_in,
    int *spmat_jj_in,
    int NE_in,
    int NV_in)
{
    fastFillCloth->init_from_python_cache_v2(adjacent_edge_in,
                                     num_adjacent_edge_in,
                                     adjacent_edge_abc_in,
                                     num_nonz_in,
                                     spmat_data_in,
                                     spmat_indices_in,
                                     spmat_indptr_in,
                                     spmat_ii_in,
                                     spmat_jj_in,
                                     NE_in,
                                     NV_in);
}

extern "C" DLLEXPORT void fastFillCloth_run(float* pos_in) {
    fastFillCloth->run(pos_in);
}

extern "C" DLLEXPORT void fastFillCloth_fetch_A_data(float* data) {
    fastFillCloth->fetch_A_data(data);
}




// ------------------------------------------------------------------------------
extern "C" DLLEXPORT void fastFillSoft_new() {
    if (!fastFillSoft)
        fastFillSoft = new FastFillSoft{};
}

extern "C" DLLEXPORT void fastFillSoft_set_data(int* tet_in, int NT_in, float* inv_mass_in, int NV_in, float* pos_in, float* alpha_tilde_in)
{
    fastFillSoft->set_data_v2(tet_in, NT_in, inv_mass_in, NV_in, pos_in, alpha_tilde_in);
}

#ifndef USE_LESSMEM
extern "C" DLLEXPORT void fastFillSoft_init_from_python_cache(
        const int NT_in,
        const int MAX_ADJ_in,
        const int* adjacent_in,
        const int* num_adjacent_in,
        const float* data_in,
        const int* indices_in,
        const int* indptr_in,
        const int* ii_in,
        const int* jj_in,
        const int num_nonz_in,
        const int* nnz_each_row_in,
        const int* n_shared_v_in,
        const int* shared_v_in,
        const int8_t* shared_v_order_in_cur,
        const int8_t* shared_v_order_in_adj
        )
{
    fastFillSoft->init_from_python_cache_v2(
        NT_in,
        MAX_ADJ_in,
        adjacent_in,
        num_adjacent_in,
        data_in,
        indices_in,
        indptr_in,
        ii_in,
        jj_in,
        num_nonz_in,
        nnz_each_row_in,
        n_shared_v_in,
        shared_v_in,
        shared_v_order_in_cur,
        shared_v_order_in_adj
        );
}
#endif


extern "C" DLLEXPORT void fastFillSoft_init_from_python_cache_lessmem(
        const int NT_in,
        const int MAX_ADJ_in,
        const float* data_in,
        const int* indices_in,
        const int* indptr_in,
        const int* ii_in,
        const int num_nonz_in
        )
{
    fastFillSoft->init_from_python_cache_lessmem(
        NT_in,
        MAX_ADJ_in,
        data_in,
        indices_in,
        indptr_in,
        ii_in,
        num_nonz_in
        );
}

extern "C" DLLEXPORT void fastFillSoft_run(float* pos_in, float* gradC_in) {
    fastFillSoft->run(pos_in, gradC_in);
}

extern "C" DLLEXPORT void fastFillSoft_fetch_A_data(float* data) {
    fastFillSoft->fetch_A_data(data);
}