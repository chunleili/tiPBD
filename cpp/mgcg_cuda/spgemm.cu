// https://github.com/NVIDIA/CUDALibrarySamples/blob/ade391a17672d26e55429035450bc44afd277d34/cuSPARSE/spgemm/spgemm_example.c#L161
// https://docs.nvidia.com/cuda/cusparse/#cusparsespgemm
// https://github.com/NVIDIA/CUDALibrarySamples/tree/ade391a17672d26e55429035450bc44afd277d34/cuSPARSE/spgemm
// C = A * B
//--------------------------------------------------------

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h> // cudaMalloc, cudaMemcpy, etc.
#include <hipsparse.h>         // cusparseSpGEMM
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
// #include "eigen/unsupported/Eigen/SparseExtra" // Eigen::loadMarket

#if __GNUC__ && __linux__
#include <sys/ptrace.h>

[[noreturn]] static void cuerr() {
    if (ptrace(PTRACE_TRACEME, 0, NULL, NULL) != 0)
        __builtin_trap();
    exit(EXIT_FAILURE);
}
#elif _WIN32 && _MSC_VER
#include <windows.h>

[[noreturn]] static void cuerr() {
    int debugger_present = 0;
    HANDLE process = GetCurrentProcess();
    CheckRemoteDebuggerPresent(process, &debugger_present);
    if (debugger_present) {
        __debugbreak();
    }
    exit(EXIT_FAILURE);
}
#else
[[noreturn]] static void cuerr() {
    exit(EXIT_FAILURE);
}
#endif



#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}


// void assign(T const *datap, size_t ndat, int const *indicesp, size_t nind, int const *indptrp, size_t nptr, size_t rows, size_t cols, size_t nnz) {
//     data.resize(ndat);
//     CHECK_CUDA(cudaMemcpy(data.data(), datap, data.size() * sizeof(T), cudaMemcpyHostToDevice));
//     indices.resize(nind);
//     CHECK_CUDA(cudaMemcpy(indices.data(), indicesp, indices.size() * sizeof(int), cudaMemcpyHostToDevice));
//     indptr.resize(nptr);
//     CHECK_CUDA(cudaMemcpy(indptr.data(), indptrp, indptr.size() * sizeof(int), cudaMemcpyHostToDevice));
//     nrows = rows;
//     ncols = cols;
//     numnonz = nnz;
// }

#include <fstream>
#include <iostream>
#include <string>
#include <vector>

template <typename T=int>
std::vector<T> readTxt(std::string filename) {
    std::ifstream file(filename);
    std::vector<T> array;
    T value;

    if (!file.is_open()) {
        std::cerr << "Failed to open file: " << filename << std::endl;
    }

    while (file >> value) {
        array.push_back(value);
    }

    file.close();

    // for (int i : array) {
    //     std::cout << i << std::endl;
    // }
    std::cout<<filename<<" read successfully"<<std::endl;

    return array;
}


void readInfo(int &nrows, int &ncols, int &nnz, std::string filename) {
    std::ifstream file(filename);
    if (!file.is_open()) {
        std::cerr << "Failed to open file: " << filename << std::endl;
    }
    file >> nrows;
    file >> ncols;
    file >> nnz;
    file.close();
    std::cout<<"nrows: "<<nrows<<" ncols: "<<ncols<<" nnz: "<<nnz<<std::endl;
}

void readCSR(std::string filename, std::vector<int>& hA_csrOffsets, std::vector<int>& hA_columns, std::vector<float>& hA_values) {
    // auto indptr = readTxt(filename+"indptr.txt");
    // auto indices = readTxt(filename+"indices.txt");
    // auto data = readTxt<float>(filename+"data.txt");
    hA_csrOffsets = readTxt<int>(filename+"indptr.txt");
    hA_columns = readTxt<int>(filename+"indices.txt");
    hA_values = readTxt<float>(filename+"data.txt");
}

void printArr(int *values, int size) {
    for (int i = 0; i < size; i++) {
        std::cout << values[i] << " ";
    }
    std::cout << std::endl;
}


int main(void) {
    // // Host problem definition
    int A_num_rows ;
    int A_num_cols ;
    int A_nnz      ;
    int B_num_rows ;
    int B_num_cols ;
    int B_nnz      ;
    std::vector<int> hA_csrOffsets, hA_columns, hB_csrOffsets, hB_columns;
    std::vector<float> hA_values, hB_values;

    readInfo(A_num_rows, A_num_cols, A_nnz, "Ainfo.txt");
    readInfo(B_num_rows, B_num_cols, B_nnz, "Binfo.txt");

    readCSR("A", hA_csrOffsets, hA_columns, hA_values);
    readCSR("B", hB_csrOffsets, hB_columns, hB_values);


    float               alpha       = 1.0f;
    float               beta        = 0.0f;
    hipsparseOperation_t opA         = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipsparseOperation_t opB         = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipDataType        computeType = HIP_R_32F;
    //--------------------------------------------------------------------------
    // Device memory management: Allocate and copy A, B
    int   *dA_csrOffsets, *dA_columns, *dB_csrOffsets, *dB_columns,
          *dC_csrOffsets, *dC_columns;
    float *dA_values, *dB_values, *dC_values;
    // allocate A
    CHECK_CUDA( hipMalloc((void**) &dA_csrOffsets,
                           (A_num_rows + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dA_columns, A_nnz * sizeof(int))   )
    CHECK_CUDA( hipMalloc((void**) &dA_values,  A_nnz * sizeof(float)) )
    // allocate B
    CHECK_CUDA( hipMalloc((void**) &dB_csrOffsets,
                           (B_num_rows + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dB_columns, B_nnz * sizeof(int))   )
    CHECK_CUDA( hipMalloc((void**) &dB_values,  B_nnz * sizeof(float)) )
    // allocate C offsets
    CHECK_CUDA( hipMalloc((void**) &dC_csrOffsets,
                           (A_num_rows + 1) * sizeof(int)) )

    // copy A
    CHECK_CUDA( hipMemcpy(dA_csrOffsets, hA_csrOffsets.data(),
                           (A_num_rows + 1) * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA_columns, hA_columns.data(), A_nnz * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA_values, hA_values.data(),
                           A_nnz * sizeof(float), hipMemcpyHostToDevice) )
    // copy B
    CHECK_CUDA( hipMemcpy(dB_csrOffsets, hB_csrOffsets.data(),
                           (B_num_rows + 1) * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dB_columns, hB_columns.data(), B_nnz * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dB_values, hB_values.data(),
                           B_nnz * sizeof(float), hipMemcpyHostToDevice) )
    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA, matB, matC;
    void*  dBuffer1    = NULL, *dBuffer2   = NULL;
    size_t bufferSize1 = 0,    bufferSize2 = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    // Create sparse matrix A in CSR format
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, A_num_rows, A_num_cols, A_nnz,
                                      dA_csrOffsets, dA_columns, dA_values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
    CHECK_CUSPARSE( hipsparseCreateCsr(&matB, B_num_rows, B_num_cols, B_nnz,
                                      dB_csrOffsets, dB_columns, dB_values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
    CHECK_CUSPARSE( hipsparseCreateCsr(&matC, A_num_rows, B_num_cols, 0,
                                      dC_csrOffsets, NULL, NULL,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
    //--------------------------------------------------------------------------
    // SpGEMM Computation
    hipsparseSpGEMMDescr_t spgemmDesc;
    CHECK_CUSPARSE( hipsparseSpGEMM_createDescr(&spgemmDesc) )

    // ask bufferSize1 bytes for external memory
    CHECK_CUSPARSE(
        hipsparseSpGEMM_workEstimation(handle, opA, opB,
                                      &alpha, matA, matB, &beta, matC,
                                      computeType, HIPSPARSE_SPGEMM_DEFAULT,
                                      spgemmDesc, &bufferSize1, NULL) )
    CHECK_CUDA( hipMalloc((void**) &dBuffer1, bufferSize1) )
    // inspect the matrices A and B to understand the memory requirement for
    // the next step
    CHECK_CUSPARSE(
        hipsparseSpGEMM_workEstimation(handle, opA, opB,
                                      &alpha, matA, matB, &beta, matC,
                                      computeType, HIPSPARSE_SPGEMM_DEFAULT,
                                      spgemmDesc, &bufferSize1, dBuffer1) )

    // ask bufferSize2 bytes for external memory
    CHECK_CUSPARSE(
        hipsparseSpGEMM_compute(handle, opA, opB,
                               &alpha, matA, matB, &beta, matC,
                               computeType, HIPSPARSE_SPGEMM_DEFAULT,
                               spgemmDesc, &bufferSize2, NULL) )
    CHECK_CUDA( hipMalloc((void**) &dBuffer2, bufferSize2) )

    // compute the intermediate product of A * B
    CHECK_CUSPARSE( hipsparseSpGEMM_compute(handle, opA, opB,
                                           &alpha, matA, matB, &beta, matC,
                                           computeType, HIPSPARSE_SPGEMM_DEFAULT,
                                           spgemmDesc, &bufferSize2, dBuffer2) )
    // get matrix C non-zero entries C_nnz1
    int64_t C_num_rows1, C_num_cols1, C_nnz1;
    CHECK_CUSPARSE( hipsparseSpMatGetSize(matC, &C_num_rows1, &C_num_cols1,
                                         &C_nnz1) )
    // allocate matrix C
    CHECK_CUDA( hipMalloc((void**) &dC_columns, C_nnz1 * sizeof(int))   )
    CHECK_CUDA( hipMalloc((void**) &dC_values,  C_nnz1 * sizeof(float)) )

    // NOTE: if 'beta' != 0, the values of C must be update after the allocation
    //       of dC_values, and before the call of cusparseSpGEMM_copy

    // update matC with the new pointers
    CHECK_CUSPARSE(
        hipsparseCsrSetPointers(matC, dC_csrOffsets, dC_columns, dC_values) )

    // if beta != 0, cusparseSpGEMM_copy reuses/updates the values of dC_values

    // copy the final products to the matrix C
    CHECK_CUSPARSE(
        hipsparseSpGEMM_copy(handle, opA, opB,
                            &alpha, matA, matB, &beta, matC,
                            computeType, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc) )


    //--------------------------------------------------------------------------
    // // device result check
    std::vector<int> hC_csrOffsets_tmp(C_num_rows1 + 1);
    std::vector<int> hC_columns_tmp(C_nnz1);
    std::vector<float> hC_values_tmp(C_nnz1);

    CHECK_CUDA( hipMemcpy(hC_csrOffsets_tmp.data(), dC_csrOffsets,
                           (A_num_rows + 1) * sizeof(int),
                           hipMemcpyDeviceToHost) )
    CHECK_CUDA( hipMemcpy(hC_columns_tmp.data(), dC_columns, C_nnz1 * sizeof(int),
                           hipMemcpyDeviceToHost) )
    CHECK_CUDA( hipMemcpy(hC_values_tmp.data(), dC_values, C_nnz1 * sizeof(float),
                           hipMemcpyDeviceToHost) )

    std::cout << "spgemm_example test PASSED" << std::endl;
    std::cout << "C_nnz: " << C_nnz1 << std::endl;
    for(int i = 0; i < C_nnz1; i++) {
        std::cout << hC_values_tmp[i] << " ";
    }
    //--------------------------------------------------------------------------
    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseSpGEMM_destroyDescr(spgemmDesc) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matB) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matC) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )
    // device memory deallocation
    CHECK_CUDA( hipFree(dBuffer1) )
    CHECK_CUDA( hipFree(dBuffer2) )
    CHECK_CUDA( hipFree(dA_csrOffsets) )
    CHECK_CUDA( hipFree(dA_columns) )
    CHECK_CUDA( hipFree(dA_values) )
    CHECK_CUDA( hipFree(dB_csrOffsets) )
    CHECK_CUDA( hipFree(dB_columns) )
    CHECK_CUDA( hipFree(dB_values) )
    CHECK_CUDA( hipFree(dC_csrOffsets) )
    CHECK_CUDA( hipFree(dC_columns) )
    CHECK_CUDA( hipFree(dC_values) )
    return EXIT_SUCCESS;
}



#if _WIN32
#define DLLEXPORT __declspec(dllexport)
#else
#define DLLEXPORT
#endif

extern "C" DLLEXPORT void fastmg_GMG(float *G) {
    // GMG(G);
}
